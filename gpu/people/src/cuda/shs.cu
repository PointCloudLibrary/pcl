
#include <hip/hip_runtime.h>

#if 0

/*
* Software License Agreement (BSD License)
*
*  Point Cloud Library (PCL) - www.pointclouds.org
*  Copyright (c) 2010-2012, Willow Garage, Inc.
*
*  All rights reserved.
*
*  Redistribution and use in source and binary forms, with or without
*  modification, are permitted provided that the following conditions
*  are met:
*
*   * Redistributions of source code must retain the above copyright
*     notice, this list of conditions and the following disclaimer.
*   * Redistributions in binary form must reproduce the above
*     copyright notice, this list of conditions and the following
*     disclaimer in the documentation and/or other materials provided
*     with the distribution.
*   * Neither the name of Willow Garage, Inc. nor the names of its
*     contributors may be used to endorse or promote products derived
*     from this software without specific prior written permission.
*
*  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
*  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
*  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
*  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
*  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
*  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
*  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
*  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
*  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
*  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
*  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
*  POSSIBILITY OF SUCH DAMAGE.
*
* $Id: $
* @authors: Anatoly Baskheev
*
*/

#include "internal.h"
#include <pcl/gpu/utils/device/funcattrib.hpp>

namespace pcl
{
    namespace device
    {     
        __device__ __forceinline__ float3 operator*(const Intr& intr, const float3 p)
        {
            float3 r;
            r.x = intr.fx * p.x + intr.cx * p.z;
            r.y = intr.fy * p.y + intr.cy * p.z;
            r.z = p.z;
            return r;
        }

        __device__ __forceinline__
            void getProjectedRadiusSearchBox (int rows, int cols, const device::Intr& intr, const float3& point, float squared_radius, 
            int &minX, int &maxX, int &minY, int &maxY)
        {  
            int min, max;

            float3 q = intr * point;

            // http://www.wolframalpha.com/input/?i=K+%3D+%7B%7Ba%2C+0%2C+b%7D%2C+%7B0%2C+c%2C+d%7D%2C+%7B0%2C+0%2C+1%7D%7D%2C+matrix%5BK+.+transpose%5BK%5D%5D

            float coeff8 = 1;                                   //K_KT_.coeff (8);
            float coeff7 = intr.cy;                             //K_KT_.coeff (7);
            float coeff4 = intr.fy * intr.fy + intr.cy*intr.cy; //K_KT_.coeff (4);

            float coeff6 = intr.cx;                             //K_KT_.coeff (6);
            float coeff0 = intr.fx * intr.fx + intr.cx*intr.cx; //K_KT_.coeff (0);

            float a = squared_radius * coeff8 - q.z * q.z;
            float b = squared_radius * coeff7 - q.y * q.z;
            float c = squared_radius * coeff4 - q.y * q.y;

            // a and c are multiplied by two already => - 4ac -> - ac
            float det = b * b - a * c;

            if (det < 0)
            {
                minY = 0;
                maxY = rows - 1;
            }
            else
            {
                float y1 = (b - sqrt (det)) / a;
                float y2 = (b + sqrt (det)) / a;

                min = min (static_cast<int> (std::floor (y1)), static_cast<int> (std::floor (y2)));
                max = max (static_cast<int> (std::ceil (y1)), static_cast<int> (std::ceil (y2)));
                minY = min (rows - 1, max (0, min));
                maxY = max (min (rows - 1, max), 0);
            }

            b = squared_radius * coeff6 - q.x * q.z;
            c = squared_radius * coeff0 - q.x * q.x;

            det = b * b - a * c;
            if (det < 0)
            {
                minX = 0;
                maxX = cols - 1;
            }
            else
            {
                float x1 = (b - sqrt (det)) / a;
                float x2 = (b + sqrt (det)) / a;

                min = min (static_cast<int> (std::floor (x1)), static_cast<int> (std::floor (x2)));
                max = max (static_cast<int> (std::ceil (x1)), static_cast<int> (std::ceil (x2)));
                minX = min (cols- 1, max (0, min));
                maxX = max (min (cols - 1, max), 0);
            }
        }

        struct Shs
        {
            PtrSz<int> indices;
            PtrStepSz<float8> cloud;
            Intr intr;
            float radius;

            mutable PtrStep<unsigned char> output_mask;
        

            __device__ __forceinline__ void operator()() const
            {
              
            }
        };
    }
}

void pcl::device::shs(const DeviceArray2D<float4> &cloud, float tolerance/*radius*/, const std::vector<int>& indices_in, float delta_hue, Mask& output)
{
    int cols = cloud.cols();
    int rows = cloud.rows();

    output.create(rows, cols);
    device::setZero(output);

    DeviceArray<int> indices_device;
    indices_device.upload(indices_in);
}

#if 0

void optimized_shs5(const PointCloud<PointXYZRGB> &cloud, float tolerance, const PointIndices &indices_in, cv::Mat flowermat, float delta_hue)
{
    int rows = 480;
    int cols = 640;
    device::Intr intr(525, 525, cols/2-0.5f, rows/2-0.5f);

    //FILE *f = fopen("log.txt", "w");

    cv::Mat huebuf(cloud.height, cloud.width, CV_32F);
    float *hue = huebuf.ptr<float>();    

    for(std::size_t i = 0; i < cloud.size(); ++i)
    {
        PointXYZHSV h;
        PointXYZRGB p = cloud[i];
        PointXYZRGBtoXYZHSV(p, h);
        hue[i] = h.h;
    }    
    unsigned char *mask = flowermat.ptr<unsigned char>();


    SearchD search;    
    search.setInputCloud(cloud.makeShared());

    std::vector< std::vector<int> > storage(100);

    //  omp_set_num_threads(1);
    // Process all points in the indices vector
    for (int k = 0; k < static_cast<int> (indices_in.indices.size ()); ++k)
    {
        int i = indices_in.indices[k];
        if (mask[i])
            continue;

        mask[i] = 255;

        //    int id = omp_get_thread_num();
        //std::vector<int>& seed_queue = storage[id];
        std::vector<int> seed_queue;
        seed_queue.clear();
        seed_queue.reserve(cloud.size());
        int sq_idx = 0;
        seed_queue.push_back (i);

        PointXYZRGB p = cloud[i];
        float h = hue[i];

        while (sq_idx < (int)seed_queue.size ())
        {
            int index = seed_queue[sq_idx];
            const PointXYZRGB& q = cloud[index];

            if(!isFinite (q))
                continue;

            // search window
            double squared_radius = tolerance * tolerance;
            //unsigned int left, right, top, bottom;            
            //search.getProjectedRadiusSearchBox (q, squared_radius, left, right, top, bottom);

            int left, right, top, bottom;
            getProjectedRadiusSearchBox(rows, cols, intr, q, squared_radius, left, right, top, bottom);

            //fprintf(f, "%d) %d %d %d %d\n", index, left, right, top, bottom);


            int yEnd  = (bottom + 1) * cloud.width + right + 1;
            int idx  = top * cloud.width + left;
            int skip = cloud.width - right + left - 1;
            int xEnd = idx - left + right + 1;

            for (; xEnd != yEnd; idx += skip, xEnd += cloud.width)
            {
                for (; idx < xEnd; ++idx)
                {
                    if (mask[idx])
                        continue;

                    if (sqnorm(cloud[idx], q) <= squared_radius)
                    {
                        float h_l = hue[idx];

                        if (std::abs(h_l - h) < delta_hue)
                        {
                            if(idx & 1)
                                seed_queue.push_back (idx);
                            mask[idx] = 255;
                        }

                    }
                }
            }
            sq_idx++;

        }        
    }       
}

#endif


#endif