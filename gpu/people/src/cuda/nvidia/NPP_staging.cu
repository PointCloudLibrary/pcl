#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (C) 2009-2010, NVIDIA Corporation, all rights reserved.
 *  Third party copyrights are property of their respective owners.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 * $Id:  $
 * Ported to PCL by Koen Buys : Attention Work in progress!
 */


#include <vector>
#include <hip/hip_runtime.h>
#include "NPP_staging.hpp"


texture<Ncv8u,  1, hipReadModeElementType> tex8u;
texture<Ncv32u, 1, hipReadModeElementType> tex32u;
texture<uint2,  1, hipReadModeElementType> tex64u;


//==============================================================================
//
// CUDA streams handling
//
//==============================================================================


static hipStream_t nppStream = 0;


hipStream_t nppStGetActiveCUDAstream(void)
{
    return nppStream;
}



hipStream_t nppStSetActiveCUDAstream(hipStream_t cudaStream)
{
    hipStream_t tmp = nppStream;
    nppStream = cudaStream;
    return tmp;
}


//==============================================================================
//
// BlockScan.cuh
//
//==============================================================================


NCV_CT_ASSERT(K_WARP_SIZE == 32); //this is required for the manual unroll of the loop in warpScanInclusive


//Almost the same as naive scan1Inclusive, but doesn't need __syncthreads()
//assuming size <= WARP_SIZE and size is power of 2
template <class T>
inline __device__ T warpScanInclusive(T idata, volatile T *s_Data)
{
    Ncv32u pos = 2 * threadIdx.x - (threadIdx.x & (K_WARP_SIZE - 1));
    s_Data[pos] = 0;
    pos += K_WARP_SIZE;
    s_Data[pos] = idata;

    //for(Ncv32u offset = 1; offset < K_WARP_SIZE; offset <<= 1)
    //{
    //    s_Data[pos] += s_Data[pos - offset];
    //}

    s_Data[pos] += s_Data[pos - 1];
    s_Data[pos] += s_Data[pos - 2];
    s_Data[pos] += s_Data[pos - 4];
    s_Data[pos] += s_Data[pos - 8];
    s_Data[pos] += s_Data[pos - 16];

    return s_Data[pos];
}


template <class T>
inline __device__ T warpScanExclusive(T idata, volatile T *s_Data)
{
    return warpScanInclusive(idata, s_Data) - idata;
}


template <class T, Ncv32u tiNumScanThreads>
inline __device__ T blockScanInclusive(T idata, volatile T *s_Data)
{
    if (tiNumScanThreads > K_WARP_SIZE)
    {
        //Bottom-level inclusive warp scan
        T warpResult = warpScanInclusive(idata, s_Data);

        //Save top elements of each warp for exclusive warp scan
        //sync to wait for warp scans to complete (because s_Data is being overwritten)
        __syncthreads();
        if( (threadIdx.x & (K_WARP_SIZE - 1)) == (K_WARP_SIZE - 1) )
        {
            s_Data[threadIdx.x >> K_LOG2_WARP_SIZE] = warpResult;
        }

        //wait for warp scans to complete
        __syncthreads();

        if( threadIdx.x < (tiNumScanThreads / K_WARP_SIZE) )
        {
            //grab top warp elements
            T val = s_Data[threadIdx.x];
            //calculate exclusive scan and write back to shared memory
            s_Data[threadIdx.x] = warpScanExclusive(val, s_Data);
        }

        //return updated warp scans with exclusive scan results
        __syncthreads();
        return warpResult + s_Data[threadIdx.x >> K_LOG2_WARP_SIZE];
    }
    else
    {
        return warpScanInclusive(idata, s_Data);
    }
}


//==============================================================================
//
// IntegralImage.cu
//
//==============================================================================


const Ncv32u NUM_SCAN_THREADS = 256;
const Ncv32u LOG2_NUM_SCAN_THREADS = 8;


template<class T_in, class T_out>
struct _scanElemOp
{
    template<bool tbDoSqr>
    static inline __host__ __device__ T_out scanElemOp(T_in elem)
    {
        return scanElemOp( elem, Int2Type<(int)tbDoSqr>() );
    }

private:

    template <int v> struct Int2Type { enum { value = v }; };

    static inline __host__ __device__ T_out scanElemOp(T_in elem, Int2Type<0>)
    {
        return (T_out)elem;
    }

    static inline __host__ __device__ T_out scanElemOp(T_in elem, Int2Type<1>)
    {
        return (T_out)(elem*elem);
    }
};


template<class T>
inline __device__ T readElem(T *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs);


template<>
inline __device__ Ncv8u readElem<Ncv8u>(Ncv8u *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return tex1Dfetch(tex8u, texOffs + srcStride * blockIdx.x + curElemOffs);
}


template<>
inline __device__ Ncv32u readElem<Ncv32u>(Ncv32u *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return d_src[curElemOffs];
}


template<>
inline __device__ Ncv32f readElem<Ncv32f>(Ncv32f *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return d_src[curElemOffs];
}


/**
* \brief Segmented scan kernel
*
* Calculates per-row prefix scans of the input image.
* Out-of-bounds safe: reads 'size' elements, writes 'size+1' elements
*
* \tparam T_in      Type of input image elements
* \tparam T_out     Type of output image elements
* \tparam T_op      Defines an operation to be performed on the input image pixels
*
* \param d_src      [IN] Source image pointer
* \param srcWidth   [IN] Source image width
* \param srcStride  [IN] Source image stride
* \param d_II       [OUT] Output image pointer
* \param IIstride   [IN] Output image stride
*
* \return None
*/
template <class T_in, class T_out, bool tbDoSqr>
__global__ void scanRows(T_in *d_src, Ncv32u texOffs, Ncv32u srcWidth, Ncv32u srcStride,
                         T_out *d_II, Ncv32u IIstride)
{
    //advance pointers to the current line
    if (sizeof(T_in) != 1)
    {
        d_src += srcStride * blockIdx.x;
    }
    //for initial image 8bit source we use texref tex8u
    d_II += IIstride * blockIdx.x;

    Ncv32u numBuckets = (srcWidth + NUM_SCAN_THREADS - 1) >> LOG2_NUM_SCAN_THREADS;
    Ncv32u offsetX = 0;

    __shared__ T_out shmem[NUM_SCAN_THREADS * 2];
    __shared__ T_out carryElem;
    carryElem = 0;
    __syncthreads();

    while (numBuckets--)
    {
        Ncv32u curElemOffs = offsetX + threadIdx.x;
        T_out curScanElem;

        T_in curElem;
        T_out curElemMod;

        if (curElemOffs < srcWidth)
        {
            //load elements
            curElem = readElem<T_in>(d_src, texOffs, srcStride, curElemOffs);
        }
        curElemMod = _scanElemOp<T_in, T_out>::scanElemOp<tbDoSqr>(curElem);

        //inclusive scan
        curScanElem = blockScanInclusive<T_out, NUM_SCAN_THREADS>(curElemMod, shmem);

        if (curElemOffs <= srcWidth)
        {
            //make scan exclusive and write the bucket to the output buffer
            d_II[curElemOffs] = carryElem + curScanElem - curElemMod;
            offsetX += NUM_SCAN_THREADS;
        }

        //remember last element for subsequent buckets adjustment
        __syncthreads();
        if (threadIdx.x == NUM_SCAN_THREADS-1)
        {
            carryElem += curScanElem;
        }
        __syncthreads();
    }

    if (offsetX == srcWidth && !threadIdx.x)
    {
        d_II[offsetX] = carryElem;
    }
}


template <bool tbDoSqr, class T_in, class T_out>
NCVStatus scanRowsWrapperDevice(T_in *d_src, Ncv32u srcStride,
                                T_out *d_dst, Ncv32u dstStride, NcvSize32u roi)
{
    hipChannelFormatDesc cfdTex;
    size_t alignmentOffset = 0;
    if (sizeof(T_in) == 1)
    {
        cfdTex = hipCreateChannelDesc<Ncv8u>();
        ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, tex8u, d_src, cfdTex, roi.height * srcStride), NPPST_TEXTURE_BIND_ERROR);
        if (alignmentOffset > 0)
        {
            ncvAssertCUDAReturn(hipUnbindTexture(tex8u), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, tex8u, d_src, cfdTex, alignmentOffset + roi.height * srcStride), NPPST_TEXTURE_BIND_ERROR);
        }
    }
    scanRows
        <T_in, T_out, tbDoSqr>
        <<<roi.height, NUM_SCAN_THREADS, 0, nppStGetActiveCUDAstream()>>>
        (d_src, (Ncv32u)alignmentOffset, roi.width, srcStride, d_dst, dstStride);
    
    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


static Ncv32u getPaddedDimension(Ncv32u dim, Ncv32u elemTypeSize, Ncv32u allocatorAlignment)
{
    Ncv32u alignMask = allocatorAlignment-1;
    Ncv32u inverseAlignMask = ~alignMask;
    Ncv32u dimBytes = dim * elemTypeSize;
    Ncv32u pitch = (dimBytes + alignMask) & inverseAlignMask;
    Ncv32u PaddedDim = pitch / elemTypeSize;
    return PaddedDim;
}


template <class T_in, class T_out>
NCVStatus ncvIntegralImage_device(T_in *d_src, Ncv32u srcStep,
                                  T_out *d_dst, Ncv32u dstStep, NcvSize32u roi,
                                  INCVMemAllocator &gpuAllocator)
{
    ncvAssertReturn(sizeof(T_out) == sizeof(Ncv32u), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn(gpuAllocator.memType() == NCVMemoryTypeDevice ||
                      gpuAllocator.memType() == NCVMemoryTypeNone, NPPST_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn((d_src != NULL && d_dst != NULL) || gpuAllocator.isCounting(), NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roi.width * sizeof(T_in) &&
                      dstStep >= (roi.width + 1) * sizeof(T_out) &&
                      srcStep % sizeof(T_in) == 0 &&
                      dstStep % sizeof(T_out) == 0, NPPST_INVALID_STEP);
    srcStep /= sizeof(T_in);
    dstStep /= sizeof(T_out);

    Ncv32u WidthII = roi.width + 1;
    Ncv32u HeightII = roi.height + 1;
    Ncv32u PaddedWidthII32 = getPaddedDimension(WidthII, sizeof(Ncv32u), gpuAllocator.alignment());
    Ncv32u PaddedHeightII32 = getPaddedDimension(HeightII, sizeof(Ncv32u), gpuAllocator.alignment());

    NCVMatrixAlloc<T_out> Tmp32_1(gpuAllocator, PaddedWidthII32, PaddedHeightII32);
    ncvAssertReturn(gpuAllocator.isCounting() || Tmp32_1.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    NCVMatrixAlloc<T_out> Tmp32_2(gpuAllocator, PaddedHeightII32, PaddedWidthII32);
    ncvAssertReturn(gpuAllocator.isCounting() || Tmp32_2.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn(Tmp32_1.pitch() * Tmp32_1.height() == Tmp32_2.pitch() * Tmp32_2.height(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat;
    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

    NCV_SKIP_COND_BEGIN

    ncvStat = scanRowsWrapperDevice
        <false>
        (d_src, srcStep, Tmp32_1.ptr(), PaddedWidthII32, roi);
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_32u_C1R((Ncv32u *)Tmp32_1.ptr(), PaddedWidthII32*sizeof(Ncv32u),
                                      (Ncv32u *)Tmp32_2.ptr(), PaddedHeightII32*sizeof(Ncv32u), NcvSize32u(WidthII, roi.height));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = scanRowsWrapperDevice
        <false>
        (Tmp32_2.ptr(), PaddedHeightII32, Tmp32_1.ptr(), PaddedHeightII32, NcvSize32u(roi.height, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_32u_C1R((Ncv32u *)Tmp32_1.ptr(), PaddedHeightII32*sizeof(Ncv32u),
                                      (Ncv32u *)d_dst, dstStep*sizeof(Ncv32u), NcvSize32u(HeightII, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    NCV_SKIP_COND_END

    return NPPST_SUCCESS;
}


NCVStatus ncvSquaredIntegralImage_device(Ncv8u *d_src, Ncv32u srcStep,
                                         Ncv64u *d_dst, Ncv32u dstStep, NcvSize32u roi,
                                         INCVMemAllocator &gpuAllocator)
{
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn(gpuAllocator.memType() == NCVMemoryTypeDevice ||
                      gpuAllocator.memType() == NCVMemoryTypeNone, NPPST_MEM_RESIDENCE_ERROR);
    ncvAssertReturn((d_src != NULL && d_dst != NULL) || gpuAllocator.isCounting(), NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roi.width &&
                      dstStep >= (roi.width + 1) * sizeof(Ncv64u) &&
                      dstStep % sizeof(Ncv64u) == 0, NPPST_INVALID_STEP);
    dstStep /= sizeof(Ncv64u);

    Ncv32u WidthII = roi.width + 1;
    Ncv32u HeightII = roi.height + 1;
    Ncv32u PaddedWidthII32 = getPaddedDimension(WidthII, sizeof(Ncv32u), gpuAllocator.alignment());
    Ncv32u PaddedHeightII32 = getPaddedDimension(HeightII, sizeof(Ncv32u), gpuAllocator.alignment());
    Ncv32u PaddedWidthII64 = getPaddedDimension(WidthII, sizeof(Ncv64u), gpuAllocator.alignment());
    Ncv32u PaddedHeightII64 = getPaddedDimension(HeightII, sizeof(Ncv64u), gpuAllocator.alignment());
    Ncv32u PaddedWidthMax = PaddedWidthII32 > PaddedWidthII64 ? PaddedWidthII32 : PaddedWidthII64;
    Ncv32u PaddedHeightMax = PaddedHeightII32 > PaddedHeightII64 ? PaddedHeightII32 : PaddedHeightII64;

    NCVMatrixAlloc<Ncv32u> Tmp32_1(gpuAllocator, PaddedWidthII32, PaddedHeightII32);
    ncvAssertReturn(Tmp32_1.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    NCVMatrixAlloc<Ncv64u> Tmp64(gpuAllocator, PaddedWidthMax, PaddedHeightMax);
    ncvAssertReturn(Tmp64.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);

    NCVMatrixReuse<Ncv32u> Tmp32_2(Tmp64.getSegment(), gpuAllocator.alignment(), PaddedWidthII32, PaddedHeightII32);
    ncvAssertReturn(Tmp32_2.isMemReused(), NPPST_MEM_INTERNAL_ERROR);
    NCVMatrixReuse<Ncv64u> Tmp64_2(Tmp64.getSegment(), gpuAllocator.alignment(), PaddedWidthII64, PaddedHeightII64);
    ncvAssertReturn(Tmp64_2.isMemReused(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat;
    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

    NCV_SKIP_COND_BEGIN

    ncvStat = scanRowsWrapperDevice
        <true, Ncv8u, Ncv32u>
        (d_src, srcStep, Tmp32_2.ptr(), PaddedWidthII32, roi);
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_32u_C1R(Tmp32_2.ptr(), PaddedWidthII32*sizeof(Ncv32u),
                                      Tmp32_1.ptr(), PaddedHeightII32*sizeof(Ncv32u), NcvSize32u(WidthII, roi.height));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = scanRowsWrapperDevice
        <false, Ncv32u, Ncv64u>
        (Tmp32_1.ptr(), PaddedHeightII32, Tmp64_2.ptr(), PaddedHeightII64, NcvSize32u(roi.height, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_64u_C1R(Tmp64_2.ptr(), PaddedHeightII64*sizeof(Ncv64u),
                                      d_dst, dstStep*sizeof(Ncv64u), NcvSize32u(HeightII, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    NCV_SKIP_COND_END

    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegralGetSize_8u32u(NcvSize32u roiSize, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    ncvAssertReturn(pBufsize != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);

    NCVMemStackAllocator gpuCounter(static_cast<Ncv32u>(devProp.textureAlignment));
    ncvAssertReturn(gpuCounter.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvIntegralImage_device((Ncv8u*)NULL, roiSize.width,
                                                  (Ncv32u*)NULL, (roiSize.width+1) * sizeof(Ncv32u),
                                                  roiSize, gpuCounter);
    ncvAssertReturnNcvStat(ncvStat);

    *pBufsize = (Ncv32u)gpuCounter.maxSize();
    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegralGetSize_32f32f(NcvSize32u roiSize, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    ncvAssertReturn(pBufsize != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);

    NCVMemStackAllocator gpuCounter(static_cast<Ncv32u>(devProp.textureAlignment));
    ncvAssertReturn(gpuCounter.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvIntegralImage_device((Ncv32f*)NULL, roiSize.width * sizeof(Ncv32f),
                                                  (Ncv32f*)NULL, (roiSize.width+1) * sizeof(Ncv32f),
                                                  roiSize, gpuCounter);
    ncvAssertReturnNcvStat(ncvStat);

    *pBufsize = (Ncv32u)gpuCounter.maxSize();
    return NPPST_SUCCESS;
}


NCVStatus nppiStSqrIntegralGetSize_8u64u(NcvSize32u roiSize, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    ncvAssertReturn(pBufsize != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);

    NCVMemStackAllocator gpuCounter(static_cast<Ncv32u>(devProp.textureAlignment));
    ncvAssertReturn(gpuCounter.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvSquaredIntegralImage_device(NULL, roiSize.width,
                                                         NULL, (roiSize.width+1) * sizeof(Ncv64u),
                                                         roiSize, gpuCounter);
    ncvAssertReturnNcvStat(ncvStat);

    *pBufsize = (Ncv32u)gpuCounter.maxSize();
    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegral_8u32u_C1R(Ncv8u *d_src, Ncv32u srcStep,
                                   Ncv32u *d_dst, Ncv32u dstStep,
                                   NcvSize32u roiSize, Ncv8u *pBuffer,
                                   Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    NCVMemStackAllocator gpuAllocator(NCVMemoryTypeDevice, bufSize, static_cast<Ncv32u>(devProp.textureAlignment), pBuffer);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvIntegralImage_device(d_src, srcStep, d_dst, dstStep, roiSize, gpuAllocator);
    ncvAssertReturnNcvStat(ncvStat);

    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegral_32f32f_C1R(Ncv32f *d_src, Ncv32u srcStep,
                                    Ncv32f *d_dst, Ncv32u dstStep,
                                    NcvSize32u roiSize, Ncv8u *pBuffer,
                                    Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    NCVMemStackAllocator gpuAllocator(NCVMemoryTypeDevice, bufSize, static_cast<Ncv32u>(devProp.textureAlignment), pBuffer);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvIntegralImage_device(d_src, srcStep, d_dst, dstStep, roiSize, gpuAllocator);
    ncvAssertReturnNcvStat(ncvStat);

    return NPPST_SUCCESS;
}


NCVStatus nppiStSqrIntegral_8u64u_C1R(Ncv8u *d_src, Ncv32u srcStep,
                                      Ncv64u *d_dst, Ncv32u dstStep,
                                      NcvSize32u roiSize, Ncv8u *pBuffer,
                                      Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    NCVMemStackAllocator gpuAllocator(NCVMemoryTypeDevice, bufSize, static_cast<Ncv32u>(devProp.textureAlignment), pBuffer);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvSquaredIntegralImage_device(d_src, srcStep, d_dst, dstStep, roiSize, gpuAllocator);
    ncvAssertReturnNcvStat(ncvStat);

    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegral_8u32u_C1R_host(Ncv8u *h_src, Ncv32u srcStep,
                                        Ncv32u *h_dst, Ncv32u dstStep,
                                        NcvSize32u roiSize)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roiSize.width &&
                      dstStep >= (roiSize.width + 1) * sizeof(Ncv32u) &&
                      dstStep % sizeof(Ncv32u) == 0, NPPST_INVALID_STEP);
    dstStep /= sizeof(Ncv32u);

    Ncv32u WidthII = roiSize.width + 1;
    Ncv32u HeightII = roiSize.height + 1;

    memset(h_dst, 0, WidthII * sizeof(Ncv32u));
    for (Ncv32u i=1; i<HeightII; i++)
    {
        h_dst[i * dstStep] = 0;
        for (Ncv32u j=1; j<WidthII; j++)
        {
            Ncv32u top = h_dst[(i-1) * dstStep + j];
            Ncv32u left = h_dst[i * dstStep + (j - 1)];
            Ncv32u topleft = h_dst[(i - 1) * dstStep + (j - 1)];
            Ncv32u elem = h_src[(i - 1) * srcStep + (j - 1)];
            h_dst[i * dstStep + j] = elem + left - topleft + top;
        }
    }

    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegral_32f32f_C1R_host(Ncv32f *h_src, Ncv32u srcStep,
                                         Ncv32f *h_dst, Ncv32u dstStep,
                                         NcvSize32u roiSize)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roiSize.width * sizeof(Ncv32f) &&
                      dstStep >= (roiSize.width + 1) * sizeof(Ncv32f) &&
                      srcStep % sizeof(Ncv32f) == 0 &&
                      dstStep % sizeof(Ncv32f) == 0, NPPST_INVALID_STEP);
    srcStep /= sizeof(Ncv32f);
    dstStep /= sizeof(Ncv32f);

    Ncv32u WidthII = roiSize.width + 1;
    Ncv32u HeightII = roiSize.height + 1;

    memset(h_dst, 0, WidthII * sizeof(Ncv32u));
    for (Ncv32u i=1; i<HeightII; i++)
    {
        h_dst[i * dstStep] = 0.0f;
        for (Ncv32u j=1; j<WidthII; j++)
        {
            Ncv32f top = h_dst[(i-1) * dstStep + j];
            Ncv32f left = h_dst[i * dstStep + (j - 1)];
            Ncv32f topleft = h_dst[(i - 1) * dstStep + (j - 1)];
            Ncv32f elem = h_src[(i - 1) * srcStep + (j - 1)];
            h_dst[i * dstStep + j] = elem + left - topleft + top;
        }
    }

    return NPPST_SUCCESS;
}


NCVStatus nppiStSqrIntegral_8u64u_C1R_host(Ncv8u *h_src, Ncv32u srcStep,
                                           Ncv64u *h_dst, Ncv32u dstStep,
                                           NcvSize32u roiSize)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roiSize.width &&
                      dstStep >= (roiSize.width + 1) * sizeof(Ncv64u) &&
                      dstStep % sizeof(Ncv64u) == 0, NPPST_INVALID_STEP);
    dstStep /= sizeof(Ncv64u);

    Ncv32u WidthII = roiSize.width + 1;
    Ncv32u HeightII = roiSize.height + 1;

    memset(h_dst, 0, WidthII * sizeof(Ncv64u));
    for (Ncv32u i=1; i<HeightII; i++)
    {
        h_dst[i * dstStep] = 0;
        for (Ncv32u j=1; j<WidthII; j++)
        {
            Ncv64u top = h_dst[(i-1) * dstStep + j];
            Ncv64u left = h_dst[i * dstStep + (j - 1)];
            Ncv64u topleft = h_dst[(i - 1) * dstStep + (j - 1)];
            Ncv64u elem = h_src[(i - 1) * srcStep + (j - 1)];
            h_dst[i * dstStep + j] = elem*elem + left - topleft + top;
        }
    }

    return NPPST_SUCCESS;
}


//==============================================================================
//
// Decimate.cu
//
//==============================================================================


const Ncv32u NUM_DOWNSAMPLE_NEAREST_THREADS_X = 32;
const Ncv32u NUM_DOWNSAMPLE_NEAREST_THREADS_Y = 8;


template<class T, NcvBool tbCacheTexture>
__device__ T getElem_Decimate(Ncv32u x, T *d_src);


template<>
__device__ Ncv32u getElem_Decimate<Ncv32u, true>(Ncv32u x, Ncv32u *d_src)
{
    return tex1Dfetch(tex32u, x);
}


template<>
__device__ Ncv32u getElem_Decimate<Ncv32u, false>(Ncv32u x, Ncv32u *d_src)
{
    return d_src[x];
}


template<>
__device__ Ncv64u getElem_Decimate<Ncv64u, true>(Ncv32u x, Ncv64u *d_src)
{
    uint2 tmp = tex1Dfetch(tex64u, x);
    Ncv64u res = (Ncv64u)tmp.y;
    res <<= 32;
    res |= tmp.x;
    return res;
}


template<>
__device__ Ncv64u getElem_Decimate<Ncv64u, false>(Ncv32u x, Ncv64u *d_src)
{
    return d_src[x];
}


template <class T, NcvBool tbCacheTexture>
__global__ void decimate_C1R(T *d_src, Ncv32u srcStep, T *d_dst, Ncv32u dstStep,
                                      NcvSize32u dstRoi, Ncv32u scale)
{
    int curX = blockIdx.x * blockDim.x + threadIdx.x;
    int curY = blockIdx.y * blockDim.y + threadIdx.y;

    if (curX >= dstRoi.width || curY >= dstRoi.height)
    {
        return;
    }

    d_dst[curY * dstStep + curX] = getElem_Decimate<T, tbCacheTexture>((curY * srcStep + curX) * scale, d_src);
}


template <class T>
static NCVStatus decimateWrapperDevice(T *d_src, Ncv32u srcStep,
                                                T *d_dst, Ncv32u dstStep,
                                                NcvSize32u srcRoi, Ncv32u scale,
                                                NcvBool readThruTexture)
{
    ncvAssertReturn(d_src != NULL && d_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(srcRoi.width > 0 && srcRoi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(scale != 0, NPPST_INVALID_SCALE);
    ncvAssertReturn(srcStep >= (Ncv32u)(srcRoi.width) * sizeof(T) &&
                      dstStep >= (Ncv32u)(srcRoi.width * sizeof(T) / scale), NPPST_INVALID_STEP);
    srcStep /= sizeof(T);
    dstStep /= sizeof(T);

    NcvSize32u dstRoi;
    dstRoi.width = srcRoi.width / scale;
    dstRoi.height = srcRoi.height / scale;

    dim3 grid((dstRoi.width + NUM_DOWNSAMPLE_NEAREST_THREADS_X - 1) / NUM_DOWNSAMPLE_NEAREST_THREADS_X,
              (dstRoi.height + NUM_DOWNSAMPLE_NEAREST_THREADS_Y - 1) / NUM_DOWNSAMPLE_NEAREST_THREADS_Y);
    dim3 block(NUM_DOWNSAMPLE_NEAREST_THREADS_X, NUM_DOWNSAMPLE_NEAREST_THREADS_Y);

    if (!readThruTexture)
    {
        decimate_C1R
            <T, false>
            <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
            (d_src, srcStep, d_dst, dstStep, dstRoi, scale);
    }
    else
    {
        hipChannelFormatDesc cfdTexSrc;

        if (sizeof(T) == sizeof(Ncv32u))
        {
            cfdTexSrc = hipCreateChannelDesc<Ncv32u>();

            size_t alignmentOffset;
            ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, tex32u, d_src, cfdTexSrc, srcRoi.height * srcStep * sizeof(T)), NPPST_TEXTURE_BIND_ERROR);
            ncvAssertReturn(alignmentOffset==0, NPPST_TEXTURE_BIND_ERROR);
        }
        else
        {
            cfdTexSrc = hipCreateChannelDesc<uint2>();

            size_t alignmentOffset;
            ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, tex64u, d_src, cfdTexSrc, srcRoi.height * srcStep * sizeof(T)), NPPST_TEXTURE_BIND_ERROR);
            ncvAssertReturn(alignmentOffset==0, NPPST_TEXTURE_BIND_ERROR);
        }

        decimate_C1R
            <T, true>
            <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
            (d_src, srcStep, d_dst, dstStep, dstRoi, scale);
    }

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


template <class T>
static NCVStatus decimateWrapperHost(T *h_src, Ncv32u srcStep,
                                              T *h_dst, Ncv32u dstStep,
                                              NcvSize32u srcRoi, Ncv32u scale)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(srcRoi.width != 0 && srcRoi.height != 0, NPPST_INVALID_ROI);
    ncvAssertReturn(scale != 0, NPPST_INVALID_SCALE);
    ncvAssertReturn(srcStep >= (Ncv32u)(srcRoi.width) * sizeof(T) &&
                      dstStep >= (Ncv32u)(srcRoi.width * sizeof(T) / scale) &&
                      srcStep % sizeof(T) == 0 && dstStep % sizeof(T) == 0, NPPST_INVALID_STEP);
    srcStep /= sizeof(T);
    dstStep /= sizeof(T);

    NcvSize32u dstRoi;
    dstRoi.width = srcRoi.width / scale;
    dstRoi.height = srcRoi.height / scale;

    for (Ncv32u i=0; i<dstRoi.height; i++)
    {
        for (Ncv32u j=0; j<dstRoi.width; j++)
        {
            h_dst[i*dstStep+j] = h_src[i*scale*srcStep + j*scale];
        }
    }

    return NPPST_SUCCESS;
}


#define implementNppDecimate(bit, typ) \
    NCVStatus nppiStDecimate_##bit##typ##_C1R(Ncv##bit##typ *d_src, Ncv32u srcStep, \
                                                     Ncv##bit##typ *d_dst, Ncv32u dstStep, \
                                                     NcvSize32u srcRoi, Ncv32u scale, NcvBool readThruTexture) \
    { \
        return decimateWrapperDevice<Ncv##bit##u>((Ncv##bit##u *)d_src, srcStep, \
                                                           (Ncv##bit##u *)d_dst, dstStep, \
                                                           srcRoi, scale, readThruTexture); \
    }


#define implementNppDecimateHost(bit, typ) \
    NCVStatus nppiStDecimate_##bit##typ##_C1R_host(Ncv##bit##typ *h_src, Ncv32u srcStep, \
                                                          Ncv##bit##typ *h_dst, Ncv32u dstStep, \
                                                          NcvSize32u srcRoi, Ncv32u scale) \
    { \
        return decimateWrapperHost<Ncv##bit##u>((Ncv##bit##u *)h_src, srcStep, \
                                                         (Ncv##bit##u *)h_dst, dstStep, \
                                                         srcRoi, scale); \
    }


implementNppDecimate(32, u)
implementNppDecimate(32, s)
implementNppDecimate(32, f)
implementNppDecimate(64, u)
implementNppDecimate(64, s)
implementNppDecimate(64, f)
implementNppDecimateHost(32, u)
implementNppDecimateHost(32, s)
implementNppDecimateHost(32, f)
implementNppDecimateHost(64, u)
implementNppDecimateHost(64, s)
implementNppDecimateHost(64, f)


//==============================================================================
//
// RectStdDev.cu
//
//==============================================================================


const Ncv32u NUM_RECTSTDDEV_THREADS = 128;


template <NcvBool tbCacheTexture>
__device__ Ncv32u getElemSum(Ncv32u x, Ncv32u *d_sum)
{
    if (tbCacheTexture)
    {
        return tex1Dfetch(tex32u, x);
    }
    else
    {
        return d_sum[x];
    }
}


template <NcvBool tbCacheTexture>
__device__ Ncv64u getElemSqSum(Ncv32u x, Ncv64u *d_sqsum)
{
    if (tbCacheTexture)
    {
        uint2 tmp = tex1Dfetch(tex64u, x);
        Ncv64u res = (Ncv64u)tmp.y;
        res <<= 32;
        res |= tmp.x;
        return res;
    }
    else
    {
        return d_sqsum[x];
    }
}


template <NcvBool tbCacheTexture>
__global__ void rectStdDev_32f_C1R(Ncv32u *d_sum, Ncv32u sumStep,
                                   Ncv64u *d_sqsum, Ncv32u sqsumStep,
                                   Ncv32f *d_norm, Ncv32u normStep,
                                   NcvSize32u roi, NcvRect32u rect, Ncv32f invRectArea)
{
    Ncv32u x_offs = blockIdx.x * NUM_RECTSTDDEV_THREADS + threadIdx.x;
    if (x_offs >= roi.width)
    {
        return;
    }

    Ncv32u sum_offset = blockIdx.y * sumStep + x_offs;
    Ncv32u sqsum_offset = blockIdx.y * sqsumStep + x_offs;

    //OPT: try swapping order (could change cache hit/miss ratio)
    Ncv32u sum_tl = getElemSum<tbCacheTexture>(sum_offset + rect.y * sumStep + rect.x, d_sum);
    Ncv32u sum_bl = getElemSum<tbCacheTexture>(sum_offset + (rect.y + rect.height) * sumStep + rect.x, d_sum);
    Ncv32u sum_tr = getElemSum<tbCacheTexture>(sum_offset + rect.y * sumStep + rect.x + rect.width, d_sum);
    Ncv32u sum_br = getElemSum<tbCacheTexture>(sum_offset + (rect.y + rect.height) * sumStep + rect.x + rect.width, d_sum);
    Ncv32u sum_val = sum_br + sum_tl - sum_tr - sum_bl;

    Ncv64u sqsum_tl, sqsum_bl, sqsum_tr, sqsum_br;
    sqsum_tl = getElemSqSum<tbCacheTexture>(sqsum_offset + rect.y * sqsumStep + rect.x, d_sqsum);
    sqsum_bl = getElemSqSum<tbCacheTexture>(sqsum_offset + (rect.y + rect.height) * sqsumStep + rect.x, d_sqsum);
    sqsum_tr = getElemSqSum<tbCacheTexture>(sqsum_offset + rect.y * sqsumStep + rect.x + rect.width, d_sqsum);
    sqsum_br = getElemSqSum<tbCacheTexture>(sqsum_offset + (rect.y + rect.height) * sqsumStep + rect.x + rect.width, d_sqsum);
    Ncv64u sqsum_val = sqsum_br + sqsum_tl - sqsum_tr - sqsum_bl;

    Ncv32f mean = sum_val * invRectArea;

    //////////////////////////////////////////////////////////////////////////
    // sqsum_val_res = sqsum_val / rectArea
    //////////////////////////////////////////////////////////////////////////

    Ncv32f sqsum_val_1 = __ull2float_rz(sqsum_val);
    Ncv64u sqsum_val_2 = __float2ull_rz(sqsum_val_1);
    Ncv64u sqsum_val_3 = sqsum_val - sqsum_val_2;
    Ncv32f sqsum_val_4 = __ull2float_rn(sqsum_val_3);
    sqsum_val_1 *= invRectArea;
    sqsum_val_4 *= invRectArea;
    Ncv32f sqsum_val_res = sqsum_val_1 + sqsum_val_4;

    //////////////////////////////////////////////////////////////////////////
    // variance = sqsum_val_res - mean * mean
    //////////////////////////////////////////////////////////////////////////

#if defined DISABLE_MAD_SELECTIVELY
    Ncv32f variance = sqsum_val_2 - __fmul_rn(mean, mean);
#else
    Ncv32f variance = sqsum_val_res - mean * mean;
#endif

    //////////////////////////////////////////////////////////////////////////
    // stddev = sqrtf(variance)
    //////////////////////////////////////////////////////////////////////////

    //Ncv32f stddev = sqrtf(variance);
    Ncv32f stddev = __fsqrt_rn(variance);

    d_norm[blockIdx.y * normStep + x_offs] = stddev;
}


NCVStatus nppiStRectStdDev_32f_C1R(Ncv32u *d_sum, Ncv32u sumStep,
                                   Ncv64u *d_sqsum, Ncv32u sqsumStep,
                                   Ncv32f *d_norm, Ncv32u normStep,
                                   NcvSize32u roi, NcvRect32u rect,
                                   Ncv32f scaleArea, NcvBool readThruTexture)
{
    ncvAssertReturn(d_sum != NULL && d_sqsum != NULL && d_norm != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(sumStep >= (Ncv32u)(roi.width + rect.x + rect.width - 1) * sizeof(Ncv32u) &&
                      sqsumStep >= (Ncv32u)(roi.width + rect.x + rect.width - 1) * sizeof(Ncv64u) &&
                      normStep >= (Ncv32u)roi.width * sizeof(Ncv32f) &&
                      sumStep % sizeof(Ncv32u) == 0 &&
                      sqsumStep % sizeof(Ncv64u) == 0 &&
                      normStep % sizeof(Ncv32f) == 0, NPPST_INVALID_STEP);
    ncvAssertReturn(scaleArea >= 1.0f, NPPST_INVALID_SCALE);
    sumStep /= sizeof(Ncv32u);
    sqsumStep /= sizeof(Ncv64u);
    normStep /= sizeof(Ncv32f);

    Ncv32f rectArea = rect.width * rect.height * scaleArea;
    Ncv32f invRectArea = 1.0f / rectArea;

    dim3 grid(((roi.width + NUM_RECTSTDDEV_THREADS - 1) / NUM_RECTSTDDEV_THREADS), roi.height);
    dim3 block(NUM_RECTSTDDEV_THREADS);

    if (!readThruTexture)
    {
        rectStdDev_32f_C1R
            <false>
            <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
            (d_sum, sumStep, d_sqsum, sqsumStep, d_norm, normStep, roi, rect, invRectArea);
    }
    else
    {
        hipChannelFormatDesc cfdTexSrc;
        hipChannelFormatDesc cfdTexSqr;
        cfdTexSrc = hipCreateChannelDesc<Ncv32u>();
        cfdTexSqr = hipCreateChannelDesc<uint2>();

        size_t alignmentOffset;
        ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, tex32u, d_sum, cfdTexSrc, (roi.height + rect.y + rect.height) * sumStep * sizeof(Ncv32u)), NPPST_TEXTURE_BIND_ERROR);
        ncvAssertReturn(alignmentOffset==0, NPPST_TEXTURE_BIND_ERROR);
        ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, tex64u, d_sqsum, cfdTexSqr, (roi.height + rect.y + rect.height) * sqsumStep * sizeof(Ncv64u)), NPPST_TEXTURE_BIND_ERROR);
        ncvAssertReturn(alignmentOffset==0, NPPST_TEXTURE_BIND_ERROR);

        rectStdDev_32f_C1R
            <true>
            <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
            (NULL, sumStep, NULL, sqsumStep, d_norm, normStep, roi, rect, invRectArea);
    }

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


NCVStatus nppiStRectStdDev_32f_C1R_host(Ncv32u *h_sum, Ncv32u sumStep,
                                        Ncv64u *h_sqsum, Ncv32u sqsumStep,
                                        Ncv32f *h_norm, Ncv32u normStep,
                                        NcvSize32u roi, NcvRect32u rect,
                                        Ncv32f scaleArea)
{
    ncvAssertReturn(h_sum != NULL && h_sqsum != NULL && h_norm != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(sumStep >= (Ncv32u)(roi.width + rect.x + rect.width - 1) * sizeof(Ncv32u) &&
                      sqsumStep >= (Ncv32u)(roi.width + rect.x + rect.width - 1) * sizeof(Ncv64u) &&
                      normStep >= (Ncv32u)roi.width * sizeof(Ncv32f) &&
                      sumStep % sizeof(Ncv32u) == 0 &&
                      sqsumStep % sizeof(Ncv64u) == 0 &&
                      normStep % sizeof(Ncv32f) == 0, NPPST_INVALID_STEP);
    ncvAssertReturn(scaleArea >= 1.0f, NPPST_INVALID_SCALE);
    sumStep /= sizeof(Ncv32u);
    sqsumStep /= sizeof(Ncv64u);
    normStep /= sizeof(Ncv32f);

    Ncv32f rectArea = rect.width * rect.height * scaleArea;
    Ncv32f invRectArea = 1.0f / rectArea;

    for (Ncv32u i=0; i<roi.height; i++)
    {
        for (Ncv32u j=0; j<roi.width; j++)
        {
            Ncv32u sum_offset = i * sumStep + j;
            Ncv32u sqsum_offset = i * sqsumStep + j;

            Ncv32u sum_tl = h_sum[sum_offset + rect.y * sumStep + rect.x];
            Ncv32u sum_bl = h_sum[sum_offset + (rect.y + rect.height) * sumStep + rect.x];
            Ncv32u sum_tr = h_sum[sum_offset + rect.y * sumStep + rect.x + rect.width];
            Ncv32u sum_br = h_sum[sum_offset + (rect.y + rect.height) * sumStep + rect.x + rect.width];
            Ncv64f sum_val = sum_br + sum_tl - sum_tr - sum_bl;

            Ncv64u sqsum_tl = h_sqsum[sqsum_offset + rect.y * sqsumStep + rect.x];
            Ncv64u sqsum_bl = h_sqsum[sqsum_offset + (rect.y + rect.height) * sqsumStep + rect.x];
            Ncv64u sqsum_tr = h_sqsum[sqsum_offset + rect.y * sqsumStep + rect.x + rect.width];
            Ncv64u sqsum_br = h_sqsum[sqsum_offset + (rect.y + rect.height) * sqsumStep + rect.x + rect.width];
            Ncv64f sqsum_val = (Ncv64f)(sqsum_br + sqsum_tl - sqsum_tr - sqsum_bl);

            Ncv64f mean = sum_val * invRectArea;
            Ncv64f sqsum_val_2 = sqsum_val / rectArea;
            Ncv64f variance = sqsum_val_2 - mean * mean;

            h_norm[i * normStep + j] = (Ncv32f)sqrt(variance);
        }
    }

    return NPPST_SUCCESS;
}


//==============================================================================
//
// Transpose.cu
//
//==============================================================================


const Ncv32u TRANSPOSE_TILE_DIM   = 16;
const Ncv32u TRANSPOSE_BLOCK_ROWS = 16;


/**
* \brief Matrix transpose kernel
*
* Calculates transpose of the input image
* \see TRANSPOSE_TILE_DIM
*
* \tparam T_in      Type of input image elements
* \tparam T_out     Type of output image elements
*
* \param d_src      [IN] Source image pointer
* \param srcStride  [IN] Source image stride
* \param d_dst      [OUT] Output image pointer
* \param dstStride  [IN] Output image stride
*
* \return None
*/
template <class T>
__global__ void transpose(T *d_src, Ncv32u srcStride,
                          T *d_dst, Ncv32u dstStride, NcvSize32u srcRoi)
{
    __shared__ T tile[TRANSPOSE_TILE_DIM][TRANSPOSE_TILE_DIM+1];

    Ncv32u blockIdx_x, blockIdx_y;

    // do diagonal reordering
    if (gridDim.x == gridDim.y)
    {
        blockIdx_y = blockIdx.x;
        blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
    }
    else
    {
        Ncv32u bid = blockIdx.x + gridDim.x * blockIdx.y;
        blockIdx_y = bid % gridDim.y;
        blockIdx_x = ((bid / gridDim.y) + blockIdx_y) % gridDim.x;
    }

    Ncv32u xIndex = blockIdx_x * TRANSPOSE_TILE_DIM + threadIdx.x;
    Ncv32u yIndex = blockIdx_y * TRANSPOSE_TILE_DIM + threadIdx.y;
    Ncv32u index_gmem = xIndex + yIndex * srcStride;

    if (xIndex < srcRoi.width)
    {
        for (Ncv32u i=0; i<TRANSPOSE_TILE_DIM; i+=TRANSPOSE_BLOCK_ROWS)
        {
            if (yIndex + i < srcRoi.height)
            {
                tile[threadIdx.y+i][threadIdx.x] = d_src[index_gmem+i*srcStride];
            }
        }
    }

    __syncthreads();

    xIndex = blockIdx_y * TRANSPOSE_TILE_DIM + threadIdx.x;
    yIndex = blockIdx_x * TRANSPOSE_TILE_DIM + threadIdx.y;
    index_gmem = xIndex + yIndex * dstStride;

    if (xIndex < srcRoi.height)
    {
        for (Ncv32u i=0; i<TRANSPOSE_TILE_DIM; i+=TRANSPOSE_BLOCK_ROWS)
        {
            if (yIndex + i < srcRoi.width)
            {
                d_dst[index_gmem+i*dstStride] = tile[threadIdx.x][threadIdx.y+i];
            }
        }
    }
}


template <class T>
NCVStatus transposeWrapperDevice(T *d_src, Ncv32u srcStride,
                                   T *d_dst, Ncv32u dstStride, NcvSize32u srcRoi)
{
    ncvAssertReturn(d_src != NULL && d_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(srcRoi.width > 0 && srcRoi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStride >= srcRoi.width * sizeof(T) &&
                      dstStride >= srcRoi.height * sizeof(T) &&
                      srcStride % sizeof(T) == 0 && dstStride % sizeof(T) == 0, NPPST_INVALID_STEP);
    srcStride /= sizeof(T);
    dstStride /= sizeof(T);

    dim3 grid((srcRoi.width + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM,
              (srcRoi.height + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM);
    dim3 block(TRANSPOSE_TILE_DIM, TRANSPOSE_TILE_DIM);
    transpose
        <T>
        <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
        (d_src, srcStride, d_dst, dstStride, srcRoi);
    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


template <class T>
static NCVStatus transposeWrapperHost(T *h_src, Ncv32u srcStride,
                                        T *h_dst, Ncv32u dstStride, NcvSize32u srcRoi)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(srcRoi.width > 0 && srcRoi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStride >= srcRoi.width * sizeof(T) &&
                      dstStride >= srcRoi.height * sizeof(T) &&
                      srcStride % sizeof(T) == 0 && dstStride % sizeof(T) == 0, NPPST_INVALID_STEP);
    srcStride /= sizeof(T);
    dstStride /= sizeof(T);

    for (Ncv32u i=0; i<srcRoi.height; i++)
    {
        for (Ncv32u j=0; j<srcRoi.width; j++)
        {
            h_dst[j*dstStride+i] = h_src[i*srcStride + j];
        }
    }

    return NPPST_SUCCESS;
}


#define implementNppTranspose(bit, typ) \
    NCVStatus nppiStTranspose_##bit##typ##_C1R(Ncv##bit##typ *d_src, Ncv32u srcStep, \
                                             Ncv##bit##typ *d_dst, Ncv32u dstStep, NcvSize32u srcRoi) \
    { \
        return transposeWrapperDevice<Ncv##bit##u>((Ncv##bit##u *)d_src, srcStep, \
                                                   (Ncv##bit##u *)d_dst, dstStep, srcRoi); \
    }


#define implementNppTransposeHost(bit, typ) \
    NCVStatus nppiStTranspose_##bit##typ##_C1R_host(Ncv##bit##typ *h_src, Ncv32u srcStep, \
                                                  Ncv##bit##typ *h_dst, Ncv32u dstStep, \
                                                  NcvSize32u srcRoi) \
    { \
        return transposeWrapperHost<Ncv##bit##u>((Ncv##bit##u *)h_src, srcStep, \
                                                 (Ncv##bit##u *)h_dst, dstStep, srcRoi); \
    }


implementNppTranspose(32,u)
implementNppTranspose(32,s)
implementNppTranspose(32,f)
implementNppTranspose(64,u)
implementNppTranspose(64,s)
implementNppTranspose(64,f)

implementNppTransposeHost(32,u)
implementNppTransposeHost(32,s)
implementNppTransposeHost(32,f)
implementNppTransposeHost(64,u)
implementNppTransposeHost(64,s)
implementNppTransposeHost(64,f)


NCVStatus nppiStTranspose_128_C1R(void *d_src, Ncv32u srcStep,
                                  void *d_dst, Ncv32u dstStep, NcvSize32u srcRoi)
{
    return transposeWrapperDevice<uint4>((uint4 *)d_src, srcStep, (uint4 *)d_dst, dstStep, srcRoi);
}


NCVStatus nppiStTranspose_128_C1R_host(void *d_src, Ncv32u srcStep,
                                       void *d_dst, Ncv32u dstStep, NcvSize32u srcRoi)
{
    return transposeWrapperHost<uint4>((uint4 *)d_src, srcStep, (uint4 *)d_dst, dstStep, srcRoi);
}


//==============================================================================
//
// Compact.cu
//
//==============================================================================


const Ncv32u NUM_REMOVE_THREADS = 256;


template <bool bRemove, bool bWritePartial>
__global__ void removePass1Scan(Ncv32u *d_src, Ncv32u srcLen,
                                Ncv32u *d_offsets, Ncv32u *d_blockSums,
                                Ncv32u elemRemove)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    Ncv32u elemAddrIn = blockId * NUM_REMOVE_THREADS + threadIdx.x;

    if (elemAddrIn > srcLen + blockDim.x)
    {
        return;
    }

    __shared__ Ncv32u shmem[NUM_REMOVE_THREADS * 2];

    Ncv32u scanElem = 0;
    if (elemAddrIn < srcLen)
    {
        if (bRemove)
        {
            scanElem = (d_src[elemAddrIn] != elemRemove) ? 1 : 0;
        }
        else
        {
            scanElem = d_src[elemAddrIn];
        }
    }

    Ncv32u localScanInc = blockScanInclusive<Ncv32u, NUM_REMOVE_THREADS>(scanElem, shmem);
    __syncthreads();

    if (elemAddrIn < srcLen)
    {
        if (threadIdx.x == NUM_REMOVE_THREADS-1 && bWritePartial)
        {
            d_blockSums[blockId] = localScanInc;
        }

        if (bRemove)
        {
            d_offsets[elemAddrIn] = localScanInc - scanElem;
        }
        else
        {
            d_src[elemAddrIn] = localScanInc - scanElem;
        }
    }
}


__global__ void removePass2Adjust(Ncv32u *d_offsets, Ncv32u srcLen, Ncv32u *d_blockSums)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    Ncv32u elemAddrIn = blockId * NUM_REMOVE_THREADS + threadIdx.x;
    if (elemAddrIn >= srcLen)
    {
        return;
    }

    __shared__ Ncv32u valOffs;
    valOffs = d_blockSums[blockId];
    __syncthreads();

    d_offsets[elemAddrIn] += valOffs;
}


__global__ void removePass3Compact(Ncv32u *d_src, Ncv32u srcLen,
                                   Ncv32u *d_offsets, Ncv32u *d_dst,
                                   Ncv32u elemRemove, Ncv32u *dstLenValue)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    Ncv32u elemAddrIn = blockId * NUM_REMOVE_THREADS + threadIdx.x;
    if (elemAddrIn >= srcLen)
    {
        return;
    }

    Ncv32u elem = d_src[elemAddrIn];
    Ncv32u elemAddrOut = d_offsets[elemAddrIn];
    if (elem != elemRemove)
    {
        d_dst[elemAddrOut] = elem;
    }

    if (elemAddrIn == srcLen-1)
    {
        if (elem != elemRemove)
        {
            *dstLenValue = elemAddrOut + 1;
        }
        else
        {
            *dstLenValue = elemAddrOut;
        }
    }
}


NCVStatus compactVector_32u_device(Ncv32u *d_src, Ncv32u srcLen,
                                   Ncv32u *d_dst, Ncv32u *dstLenPinned,
                                   Ncv32u elemRemove,
                                   INCVMemAllocator &gpuAllocator)
{
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn((d_src != NULL && d_dst != NULL) || gpuAllocator.isCounting(), NPPST_NULL_POINTER_ERROR);

    if (srcLen == 0)
    {
        if (dstLenPinned != NULL)
        {
            *dstLenPinned = 0;
        }
        return NPPST_SUCCESS;
    }

    std::vector<Ncv32u> partSumNums;
    std::vector<Ncv32u> partSumOffsets;
    Ncv32u partSumLastNum = srcLen;
    Ncv32u partSumLastOffs = 0;
    do
    {
        partSumNums.push_back(partSumLastNum);
        partSumOffsets.push_back(partSumLastOffs);

        Ncv32u curPartSumAlignedLength = alignUp(partSumLastNum * sizeof(Ncv32u),
                                                 gpuAllocator.alignment()) / sizeof(Ncv32u);
        partSumLastOffs += curPartSumAlignedLength;

        partSumLastNum = (partSumLastNum + NUM_REMOVE_THREADS - 1) / NUM_REMOVE_THREADS;
    }
    while (partSumLastNum>1);
    partSumNums.push_back(partSumLastNum);
    partSumOffsets.push_back(partSumLastOffs);

    NCVVectorAlloc<Ncv32u> d_hierSums(gpuAllocator, partSumLastOffs+1);
    ncvAssertReturn(gpuAllocator.isCounting() || d_hierSums.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    NCVVectorAlloc<Ncv32u> d_numDstElements(gpuAllocator, 1);
    ncvAssertReturn(gpuAllocator.isCounting() || d_numDstElements.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);

    NCV_SET_SKIP_COND(gpuAllocator.isCounting());
    NCV_SKIP_COND_BEGIN

    dim3 block(NUM_REMOVE_THREADS);

    //calculate zero-level partial sums for indices calculation
    if (partSumNums.size() > 2)
    {
        dim3 grid(partSumNums[1]);

        if (grid.x > 65535)
        {
            grid.y = (grid.x + 65534) / 65535;
            grid.x = 65535;
        }
        removePass1Scan
            <true, true>
            <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
            (d_src, srcLen,
             d_hierSums.ptr(),
             d_hierSums.ptr() + partSumOffsets[1],
             elemRemove);

        ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

        //calculate hierarchical partial sums
        for (Ncv32u i=1; i<partSumNums.size()-1; i++)
        {
            dim3 grid(partSumNums[i+1]);
            if (grid.x > 65535)
            {
                grid.y = (grid.x + 65534) / 65535;
                grid.x = 65535;
            }
            if (grid.x != 1)
            {
                removePass1Scan
                    <false, true>
                    <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
                    (d_hierSums.ptr() + partSumOffsets[i],
                     partSumNums[i], NULL,
                     d_hierSums.ptr() + partSumOffsets[i+1],
                     NULL);
            }
            else
            {
                removePass1Scan
                    <false, false>
                    <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
                    (d_hierSums.ptr() + partSumOffsets[i],
                     partSumNums[i], NULL,
                     NULL,
                     NULL);
            }

            ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
        }

        //adjust hierarchical partial sums
        for (Ncv32s i=(Ncv32s)partSumNums.size()-3; i>=0; i--)
        {
            dim3 grid(partSumNums[i+1]);
            if (grid.x > 65535)
            {
                grid.y = (grid.x + 65534) / 65535;
                grid.x = 65535;
            }
            removePass2Adjust
                <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
                (d_hierSums.ptr() + partSumOffsets[i], partSumNums[i],
                 d_hierSums.ptr() + partSumOffsets[i+1]);

            ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
        }
    }
    else
    {
        dim3 grid(partSumNums[1]);
        removePass1Scan
            <true, false>
            <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
            (d_src, srcLen,
             d_hierSums.ptr(),
             NULL, elemRemove);

        ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
    }

    //compact source vector using indices
    dim3 grid(partSumNums[1]);
    if (grid.x > 65535)
    {
        grid.y = (grid.x + 65534) / 65535;
        grid.x = 65535;
    }
    removePass3Compact
        <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
        (d_src, srcLen, d_hierSums.ptr(), d_dst,
         elemRemove, d_numDstElements.ptr());

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    //get number of dst elements
    if (dstLenPinned != NULL)
    {
        ncvAssertCUDAReturn(hipMemcpyAsync(dstLenPinned, d_numDstElements.ptr(), sizeof(Ncv32u),
                                              hipMemcpyDeviceToHost, nppStGetActiveCUDAstream()), NPPST_MEM_RESIDENCE_ERROR);
        ncvAssertCUDAReturn(hipStreamSynchronize(nppStGetActiveCUDAstream()), NPPST_MEM_RESIDENCE_ERROR);
    }

    NCV_SKIP_COND_END

    return NPPST_SUCCESS;
}


NCVStatus nppsStCompactGetSize_32u(Ncv32u srcLen, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    ncvAssertReturn(pBufsize != NULL, NPPST_NULL_POINTER_ERROR);

    if (srcLen == 0)
    {
        *pBufsize = 0;
        return NPPST_SUCCESS;
    }

    NCVMemStackAllocator gpuCounter(static_cast<Ncv32u>(devProp.textureAlignment));
    ncvAssertReturn(gpuCounter.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = compactVector_32u_device(NULL, srcLen, NULL, NULL, 0xC001C0DE,
                                                 gpuCounter);
    ncvAssertReturnNcvStat(ncvStat);

    *pBufsize = (Ncv32u)gpuCounter.maxSize();
    return NPPST_SUCCESS;
}


NCVStatus nppsStCompactGetSize_32s(Ncv32u srcLen, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    return nppsStCompactGetSize_32u(srcLen, pBufsize, devProp);
}


NCVStatus nppsStCompactGetSize_32f(Ncv32u srcLen, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    return nppsStCompactGetSize_32u(srcLen, pBufsize, devProp);
}


NCVStatus nppsStCompact_32u(Ncv32u *d_src, Ncv32u srcLen,
                            Ncv32u *d_dst, Ncv32u *p_dstLen,
                            Ncv32u elemRemove, Ncv8u *pBuffer,
                            Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    NCVMemStackAllocator gpuAllocator(NCVMemoryTypeDevice, bufSize, static_cast<Ncv32u>(devProp.textureAlignment), pBuffer);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = compactVector_32u_device(d_src, srcLen, d_dst, p_dstLen, elemRemove,
                                                 gpuAllocator);
    ncvAssertReturnNcvStat(ncvStat);

    return NPPST_SUCCESS;
}


NCVStatus nppsStCompact_32s(Ncv32s *d_src, Ncv32u srcLen,
                            Ncv32s *d_dst, Ncv32u *p_dstLen,
                            Ncv32s elemRemove, Ncv8u *pBuffer,
                            Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    return nppsStCompact_32u((Ncv32u *)d_src, srcLen, (Ncv32u *)d_dst, p_dstLen,
                             *(Ncv32u *)&elemRemove, pBuffer, bufSize, devProp);
}


NCVStatus nppsStCompact_32f(Ncv32f *d_src, Ncv32u srcLen,
                            Ncv32f *d_dst, Ncv32u *p_dstLen,
                            Ncv32f elemRemove, Ncv8u *pBuffer,
                            Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    return nppsStCompact_32u((Ncv32u *)d_src, srcLen, (Ncv32u *)d_dst, p_dstLen,
                             *(Ncv32u *)&elemRemove, pBuffer, bufSize, devProp);
}


NCVStatus nppsStCompact_32u_host(Ncv32u *h_src, Ncv32u srcLen,
                                 Ncv32u *h_dst, Ncv32u *dstLen, Ncv32u elemRemove)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);

    if (srcLen == 0)
    {
        if (dstLen != NULL)
        {
            *dstLen = 0;
        }
        return NPPST_SUCCESS;
    }

    Ncv32u dstIndex = 0;
    for (Ncv32u srcIndex=0; srcIndex<srcLen; srcIndex++)
    {
        if (h_src[srcIndex] != elemRemove)
        {
            h_dst[dstIndex++] = h_src[srcIndex];
        }
    }

    if (dstLen != NULL)
    {
        *dstLen = dstIndex;
    }

    return NPPST_SUCCESS;
}


NCVStatus nppsStCompact_32s_host(Ncv32s *h_src, Ncv32u srcLen,
                                 Ncv32s *h_dst, Ncv32u *dstLen, Ncv32s elemRemove)
{
    return nppsStCompact_32u_host((Ncv32u *)h_src, srcLen, (Ncv32u *)h_dst, dstLen, *(Ncv32u *)&elemRemove);
}


NCVStatus nppsStCompact_32f_host(Ncv32f *h_src, Ncv32u srcLen,
                                 Ncv32f *h_dst, Ncv32u *dstLen, Ncv32f elemRemove)
{
    return nppsStCompact_32u_host((Ncv32u *)h_src, srcLen, (Ncv32u *)h_dst, dstLen, *(Ncv32u *)&elemRemove);
}


//==============================================================================
//
// Filter.cu
//
//==============================================================================


texture <float, 1, hipReadModeElementType> texSrc;
texture <float, 1, hipReadModeElementType> texKernel;


__forceinline__ __device__ float getValueMirrorRow(const int rowOffset,
                                                   int i,
                                                   int w)
{
    if (i < 0) i = 1 - i;
    if (i >= w) i = w + w - i - 1;
    return tex1Dfetch (texSrc, rowOffset + i);
}


__forceinline__ __device__ float getValueMirrorColumn(const int offset,
                                                      const int rowStep,
                                                      int j,
                                                      int h)
{
    if (j < 0) j = 1 - j;
    if (j >= h) j = h + h - j - 1;
    return tex1Dfetch (texSrc, offset + j * rowStep);
}


__global__ void FilterRowBorderMirror_32f_C1R(Ncv32u srcStep,
                                              Ncv32f *pDst, 
                                              NcvSize32u dstSize,
                                              Ncv32u dstStep,
                                              NcvRect32u roi,
                                              Ncv32s nKernelSize,
                                              Ncv32s nAnchor,
                                              Ncv32f multiplier)
{
    // position within ROI
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix >= roi.width || iy >= roi.height)
    {
        return;
    }

    const int p = nKernelSize - nAnchor - 1;

    const int j = roi.y + iy;

    const int rowOffset = j * srcStep + roi.x;

    float sum = 0.0f;
    for (int m = 0; m < nKernelSize; ++m)
    {
        sum += getValueMirrorRow (rowOffset, ix + m - p, roi.width) 
            * tex1Dfetch (texKernel, m);
    }

    pDst[iy * dstStep + ix] = sum * multiplier;
}


__global__ void FilterColumnBorderMirror_32f_C1R(Ncv32u srcStep,
                                                 Ncv32f *pDst,
                                                 NcvSize32u dstSize,
                                                 Ncv32u dstStep,
                                                 NcvRect32u roi,
                                                 Ncv32s nKernelSize,
                                                 Ncv32s nAnchor,
                                                 Ncv32f multiplier)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix >= roi.width || iy >= roi.height)
    {
        return;
    }

    const int p = nKernelSize - nAnchor - 1;
    const int i = roi.x + ix;
    const int offset = i + roi.y * srcStep;

    float sum = 0.0f;
    for (int m = 0; m < nKernelSize; ++m)
    {
        sum += getValueMirrorColumn (offset, srcStep, iy + m - p, roi.height) 
            * tex1Dfetch (texKernel, m);
    }

    pDst[ix + iy * dstStep] = sum * multiplier;
}


NCVStatus nppiStFilterRowBorder_32f_C1R(const Ncv32f *pSrc,
                                        NcvSize32u srcSize,
                                        Ncv32u nSrcStep,
                                        Ncv32f *pDst,
                                        NcvSize32u dstSize,
                                        Ncv32u nDstStep,
                                        NcvRect32u oROI,
                                        NppStBorderType borderType,
                                        const Ncv32f *pKernel,
                                        Ncv32s nKernelSize,
                                        Ncv32s nAnchor,
                                        Ncv32f multiplier)
{
    ncvAssertReturn (pSrc != NULL &&
        pDst != NULL &&
        pKernel != NULL, NCV_NULL_PTR);

    ncvAssertReturn (oROI.width > 0 && oROI.height > 0, NPPST_INVALID_ROI);

    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep &&
        dstSize.width * sizeof (Ncv32f) <= nDstStep &&
        oROI.width * sizeof (Ncv32f) <= nSrcStep &&
        oROI.width * sizeof (Ncv32f) <= nDstStep &&
        nSrcStep % sizeof (Ncv32f) == 0 &&
        nDstStep % sizeof (Ncv32f) == 0, NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u dstStep = nDstStep / sizeof (Ncv32f);

    // adjust ROI size to be within source image
    if (oROI.x + oROI.width > srcSize.width)
    {
        oROI.width = srcSize.width - oROI.x;
    }

    if (oROI.y + oROI.height > srcSize.height)
    {
        oROI.height = srcSize.height - oROI.y;
    }

    hipChannelFormatDesc floatChannel = hipCreateChannelDesc <float> ();
    texSrc.normalized    = false;
    texKernel.normalized = false;

    hipBindTexture (0, texSrc, pSrc, floatChannel, srcSize.height * nSrcStep);
    hipBindTexture (0, texKernel, pKernel, floatChannel, nKernelSize * sizeof (Ncv32f));

    dim3 ctaSize (32, 6);
    dim3 gridSize ((oROI.width + ctaSize.x - 1) / ctaSize.x,
        (oROI.height + ctaSize.y - 1) / ctaSize.y);

    switch (borderType)
    {
    case nppStBorderNone:
        return NPPST_ERROR;
    case nppStBorderClamp:
        return NPPST_ERROR;
    case nppStBorderWrap:
        return NPPST_ERROR;
    case nppStBorderMirror:
        FilterRowBorderMirror_32f_C1R <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream ()>>>
            (srcStep, pDst, dstSize, dstStep, oROI, nKernelSize, nAnchor, multiplier);
        ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
        break;
    default:
        return NPPST_ERROR;
    }

    return NPPST_SUCCESS;
}


NCVStatus nppiStFilterColumnBorder_32f_C1R(const Ncv32f *pSrc,
                                           NcvSize32u srcSize,
                                           Ncv32u nSrcStep,
                                           Ncv32f *pDst,
                                           NcvSize32u dstSize,
                                           Ncv32u nDstStep,
                                           NcvRect32u oROI,
                                           NppStBorderType borderType,
                                           const Ncv32f *pKernel,
                                           Ncv32s nKernelSize,
                                           Ncv32s nAnchor,
                                           Ncv32f multiplier)
{
    ncvAssertReturn (pSrc != NULL &&
        pDst != NULL &&
        pKernel != NULL, NCV_NULL_PTR);

    ncvAssertReturn (oROI.width > 0 && oROI.height > 0, NPPST_INVALID_ROI);

    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep &&
        dstSize.width * sizeof (Ncv32f) <= nDstStep &&
        oROI.width * sizeof (Ncv32f) <= nSrcStep &&
        oROI.width * sizeof (Ncv32f) <= nDstStep &&
        nSrcStep % sizeof (Ncv32f) == 0 &&
        nDstStep % sizeof (Ncv32f) == 0, NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u dstStep = nDstStep / sizeof (Ncv32f);

    // adjust ROI size to be within source image
    if (oROI.x + oROI.width > srcSize.width)
    {
        oROI.width = srcSize.width - oROI.x;
    }

    if (oROI.y + oROI.height > srcSize.height)
    {
        oROI.height = srcSize.height - oROI.y;
    }

    hipChannelFormatDesc floatChannel = hipCreateChannelDesc <float> ();
    texSrc.normalized    = false;
    texKernel.normalized = false;

    hipBindTexture (0, texSrc, pSrc, floatChannel, srcSize.height * nSrcStep);
    hipBindTexture (0, texKernel, pKernel, floatChannel, nKernelSize * sizeof (Ncv32f));

    dim3 ctaSize (32, 6);
    dim3 gridSize ((oROI.width + ctaSize.x - 1) / ctaSize.x,
        (oROI.height + ctaSize.y - 1) / ctaSize.y);

    switch (borderType)
    {
    case nppStBorderClamp:
        return NPPST_ERROR;
    case nppStBorderWrap:
        return NPPST_ERROR;
    case nppStBorderMirror:
        FilterColumnBorderMirror_32f_C1R <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream ()>>>
            (srcStep, pDst, dstSize, dstStep, oROI, nKernelSize, nAnchor, multiplier);
        ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
        break;
    default:
        return NPPST_ERROR;
    }

    return NPPST_SUCCESS;
}


//==============================================================================
//
// FrameInterpolate.cu
//
//==============================================================================


inline Ncv32u iDivUp(Ncv32u num, Ncv32u denom)
{
    return (num + denom - 1)/denom;
}


texture<float, 2, hipReadModeElementType> tex_src1;
texture<float, 2, hipReadModeElementType> tex_src0;


__global__ void BlendFramesKernel(const float *u, const float *v,   // forward flow
                                  const float *ur, const float *vr, // backward flow
                                  const float *o0, const float *o1, // coverage masks
                                  int w, int h, int s, 
                                  float theta, float *out)
{
    const int ix = threadIdx.x + blockDim.x * blockIdx.x;
    const int iy = threadIdx.y + blockDim.y * blockIdx.y;

    const int pos = ix + s * iy;

    if (ix >= w || iy >= h) return;

    float _u = u[pos];
    float _v = v[pos];

    float _ur = ur[pos];
    float _vr = vr[pos];

    float x = (float)ix + 0.5f;
    float y = (float)iy + 0.5f;
    bool b0 = o0[pos] > 1e-4f;
    bool b1 = o1[pos] > 1e-4f;

    if (b0 && b1)
    {
        // pixel is visible on both frames
        out[pos] = tex2D(tex_src0, x - _u * theta, y - _v * theta) * (1.0f - theta) + 
            tex2D(tex_src1, x + _u * (1.0f - theta), y + _v * (1.0f - theta)) * theta;
    }
    else if (b0)
    {
        // visible on the first frame only
        out[pos] = tex2D(tex_src0, x - _u * theta, y - _v * theta);
    }
    else
    {
        // visible on the second frame only
        out[pos] = tex2D(tex_src1, x - _ur * (1.0f - theta), y - _vr * (1.0f - theta));
    }
}


NCVStatus BlendFrames(const Ncv32f *src0,
                      const Ncv32f *src1,
                      const Ncv32f *ufi,
                      const Ncv32f *vfi,
                      const Ncv32f *ubi,
                      const Ncv32f *vbi,
                      const Ncv32f *o1,
                      const Ncv32f *o2,
                      Ncv32u width,
                      Ncv32u height,
                      Ncv32u stride,
                      Ncv32f theta,
                      Ncv32f *out)
{
    tex_src1.addressMode[0] = hipAddressModeClamp;
    tex_src1.addressMode[1] = hipAddressModeClamp;
    tex_src1.filterMode = hipFilterModeLinear;
    tex_src1.normalized = false;

    tex_src0.addressMode[0] = hipAddressModeClamp;
    tex_src0.addressMode[1] = hipAddressModeClamp;
    tex_src0.filterMode = hipFilterModeLinear;
    tex_src0.normalized = false;

    hipChannelFormatDesc desc = hipCreateChannelDesc <float> ();
    const Ncv32u pitch = stride * sizeof (float);
    ncvAssertCUDAReturn (hipBindTexture2D (0, tex_src1, src1, desc, width, height, pitch), NPPST_TEXTURE_BIND_ERROR);
    ncvAssertCUDAReturn (hipBindTexture2D (0, tex_src0, src0, desc, width, height, pitch), NPPST_TEXTURE_BIND_ERROR);

    dim3 threads (32, 4);
    dim3 blocks (iDivUp (width, threads.x), iDivUp (height, threads.y));

    BlendFramesKernel<<<blocks, threads, 0, nppStGetActiveCUDAstream ()>>>
        (ufi, vfi, ubi, vbi, o1, o2, width, height, stride, theta, out);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


NCVStatus nppiStGetInterpolationBufferSize(NcvSize32u srcSize,
                                           Ncv32u nStep,
                                           Ncv32u *hpSize)
{
    NCVStatus status = NPPST_ERROR;
    status = nppiStVectorWarpGetBufferSize(srcSize, nStep, hpSize);
    return status;
}


NCVStatus nppiStInterpolateFrames(const NppStInterpolationState *pState)
{
    // check state validity
    ncvAssertReturn (pState->pSrcFrame0 != 0 &&
        pState->pSrcFrame1 != 0 &&
        pState->pFU != 0 &&
        pState->pFV != 0 &&
        pState->pBU != 0 &&
        pState->pBV != 0 &&
        pState->pNewFrame != 0 &&
        pState->ppBuffers[0] != 0 &&
        pState->ppBuffers[1] != 0 &&
        pState->ppBuffers[2] != 0 &&
        pState->ppBuffers[3] != 0 &&
        pState->ppBuffers[4] != 0 &&
        pState->ppBuffers[5] != 0, NPPST_NULL_POINTER_ERROR);

    ncvAssertReturn (pState->size.width  > 0 &&
        pState->size.height > 0, NPPST_ERROR);

    ncvAssertReturn (pState->nStep >= pState->size.width * sizeof (Ncv32f) &&
        pState->nStep > 0 &&
        pState->nStep % sizeof (Ncv32f) == 0,
        NPPST_INVALID_STEP);

    // change notation
    Ncv32f *cov0 = pState->ppBuffers[0];
    Ncv32f *cov1 = pState->ppBuffers[1];
    Ncv32f *fwdU = pState->ppBuffers[2]; // forward u
    Ncv32f *fwdV = pState->ppBuffers[3]; // forward v
    Ncv32f *bwdU = pState->ppBuffers[4]; // backward u
    Ncv32f *bwdV = pState->ppBuffers[5]; // backward v
    // warp flow
    ncvAssertReturnNcvStat (
        nppiStVectorWarp_PSF2x2_32f_C1 (pState->pFU, 
        pState->size, 
        pState->nStep,
        pState->pFU,
        pState->pFV,
        pState->nStep,
        cov0,
        pState->pos,
        fwdU) );
    ncvAssertReturnNcvStat (
        nppiStVectorWarp_PSF2x2_32f_C1 (pState->pFV, 
        pState->size, 
        pState->nStep,
        pState->pFU,
        pState->pFV,
        pState->nStep,
        cov0,
        pState->pos,
        fwdV) );
    // warp backward flow
    ncvAssertReturnNcvStat (
        nppiStVectorWarp_PSF2x2_32f_C1 (pState->pBU, 
        pState->size, 
        pState->nStep,
        pState->pBU,
        pState->pBV,
        pState->nStep,
        cov1,
        1.0f - pState->pos,
        bwdU) );
    ncvAssertReturnNcvStat (
        nppiStVectorWarp_PSF2x2_32f_C1 (pState->pBV, 
        pState->size, 
        pState->nStep,
        pState->pBU,
        pState->pBV,
        pState->nStep,
        cov1,
        1.0f - pState->pos,
        bwdU) );
    // interpolate frame
    ncvAssertReturnNcvStat (
        BlendFrames (pState->pSrcFrame0,
        pState->pSrcFrame1,
        fwdU,
        fwdV,
        bwdU,
        bwdV,
        cov0,
        cov1,
        pState->size.width,
        pState->size.height,
        pState->nStep / sizeof (Ncv32f),
        pState->pos,
        pState->pNewFrame) );

    return NPPST_SUCCESS;
}


//==============================================================================
//
// VectorWarpFrame.cu
//
//==============================================================================


#if ((defined __CUDA_ARCH__) && (__CUDA_ARCH__ < 200))

// FP32 atomic add
static __forceinline__ __device__ float _atomicAdd(float *addr, float val)
{
    float old = *addr, assumed;

    do {
        assumed = old;
        old = int_as_float(__iAtomicCAS((int*)addr,
              float_as_int(assumed),
              float_as_int(val+assumed)));
    } while( assumed!=old );

    return old;
}
#else
#define _atomicAdd atomicAdd
#endif


__global__ void ForwardWarpKernel_PSF2x2(const float *u,
                                         const float *v,
                                         const float *src,
                                         const int w,
                                         const int h,
                                         const int flow_stride,
                                         const int image_stride,
                                         const float time_scale,
                                         float *normalization_factor,
                                         float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    //bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    float px;
    float py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);
    // target pixel integer coords
    int tx;
    int ty;
    tx = (int) px;
    ty = (int) py;
    float value = src[image_row_offset + j];
    float weight;
    // fill pixel containing bottom right corner
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        weight = dx * dy;
        _atomicAdd (dst + ty * image_stride + tx, value * weight);
        _atomicAdd (normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing bottom left corner
    tx -= 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        weight = (1.0f - dx) * dy;
        _atomicAdd (dst + ty * image_stride + tx, value * weight);
        _atomicAdd (normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper left corner
    ty -= 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd (dst + ty * image_stride + tx, value * weight);
        _atomicAdd (normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper right corner
    tx += 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        weight = dx * (1.0f - dy);
        _atomicAdd (dst + ty * image_stride + tx, value * weight);
        _atomicAdd (normalization_factor + ty * image_stride + tx, weight);
    }
}


__global__ void ForwardWarpKernel_PSF1x1(const float *u,
                                         const float *v,
                                         const float *src,
                                         const int w,
                                         const int h,
                                         const int flow_stride,
                                         const int image_stride,
                                         const float time_scale,
                                         float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    //bottom left corner of target pixel
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    int tx = __float2int_rn (cx);
    int ty = __float2int_rn (cy);

    float value = src[image_row_offset + j];
    // fill pixel
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        _atomicAdd (dst + ty * image_stride + tx, value);
    }
}


__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];

    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

    image[pos] *= invScale;
}


__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= h || j >= w) return;

    const int pos = i * w + j;

    image[pos] = value;
}


NCVStatus nppiStVectorWarpGetBufferSize (NcvSize32u srcSize, Ncv32u nSrcStep, Ncv32u *hpSize)
{
    ncvAssertReturn (hpSize != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep,
        NPPST_INVALID_STEP);

    *hpSize = nSrcStep * srcSize.height;

    return NPPST_SUCCESS;
}


// does not require normalization
NCVStatus nppiStVectorWarp_PSF1x1_32f_C1(const Ncv32f *pSrc,
                                         NcvSize32u srcSize,
                                         Ncv32u nSrcStep,
                                         const Ncv32f *pU,
                                         const Ncv32f *pV,
                                         Ncv32u nVFStep,
                                         Ncv32f timeScale,
                                         Ncv32f *pDst)
{
    ncvAssertReturn (pSrc != NULL && 
        pU   != NULL &&
        pV   != NULL &&
        pDst != NULL, NPPST_NULL_POINTER_ERROR);

    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep &&
        srcSize.width * sizeof (Ncv32f) <= nVFStep,
        NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u vfStep  = nVFStep / sizeof (Ncv32f);

    dim3 ctaSize (32, 6);
    dim3 gridSize (iDivUp (srcSize.width, ctaSize.x), iDivUp (srcSize.height, ctaSize.y));

    ForwardWarpKernel_PSF1x1 <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream()>>>
        (pU, pV, pSrc, srcSize.width, srcSize.height, vfStep, srcStep, timeScale, pDst);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


NCVStatus nppiStVectorWarp_PSF2x2_32f_C1(const Ncv32f *pSrc,
                                         NcvSize32u srcSize,
                                         Ncv32u nSrcStep,
                                         const Ncv32f *pU,
                                         const Ncv32f *pV,
                                         Ncv32u nVFStep,
                                         Ncv32f *pBuffer,
                                         Ncv32f timeScale,
                                         Ncv32f *pDst)
{
    ncvAssertReturn (pSrc != NULL && 
        pU   != NULL &&
        pV   != NULL &&
        pDst != NULL &&
        pBuffer != NULL, NPPST_NULL_POINTER_ERROR);

    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep &&
        srcSize.width * sizeof (Ncv32f) <= nVFStep, NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u vfStep = nVFStep / sizeof(Ncv32f);

    dim3 ctaSize(32, 6);
    dim3 gridSize (iDivUp (srcSize.width, ctaSize.x), iDivUp (srcSize.height, ctaSize.y));

    MemsetKernel <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream()>>>
        (0, srcSize.width, srcSize.height, pBuffer);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    ForwardWarpKernel_PSF2x2 <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream()>>>
        (pU, pV, pSrc, srcSize.width, srcSize.height, vfStep, srcStep, timeScale, pBuffer, pDst);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    NormalizeKernel <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream()>>>
        (pBuffer, srcSize.width, srcSize.height, srcStep, pDst);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


//==============================================================================
//
// Resize.cu
//
//==============================================================================


texture <float, 2, hipReadModeElementType> texSrc2D;


__forceinline__
__device__ float processLine(int spos,
                             float xmin,
                             float xmax,
                             int ixmin,
                             int ixmax,
                             float fxmin,
                             float cxmax)
{
    // first element
    float wsum = 1.0f - xmin + fxmin;
    float sum = tex1Dfetch(texSrc, spos) * (1.0f - xmin + fxmin);
    spos++;
    for (int ix = ixmin + 1; ix < ixmax; ++ix)
    {
        sum += tex1Dfetch(texSrc, spos);
        spos++;
        wsum += 1.0f;
    }
    sum += tex1Dfetch(texSrc, spos) * (cxmax - xmax);
    wsum += cxmax - xmax;
    return sum / wsum;
}


__global__ void resizeSuperSample_32f(NcvSize32u srcSize,
                                      Ncv32u srcStep,
                                      NcvRect32u srcROI,
                                      Ncv32f *dst,
                                      NcvSize32u dstSize,
                                      Ncv32u dstStep,
                                      NcvRect32u dstROI,
                                      Ncv32f scaleX,
                                      Ncv32f scaleY)
{
    // position within dst ROI
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= dstROI.width || iy >= dstROI.height)
    {
        return;
    }

    float rw = (float) srcROI.width;
    float rh = (float) srcROI.height; 

    // source position
    float x = scaleX * (float) ix;
    float y = scaleY * (float) iy;

    // x sampling range
    float xBegin = fmax (x - scaleX, 0.0f);
    float xEnd   = fmin (x + scaleX, rw - 1.0f);
    // y sampling range
    float yBegin = fmax (y - scaleY, 0.0f);
    float yEnd   = fmin (y + scaleY, rh - 1.0f);
    // x range of source samples
    float floorXBegin = floorf (xBegin);
    float ceilXEnd    = ceilf (xEnd);
    int iXBegin = srcROI.x + (int) floorXBegin;
    int iXEnd   = srcROI.x + (int) ceilXEnd;
    // y range of source samples
    float floorYBegin = floorf (yBegin);
    float ceilYEnd    = ceilf (yEnd);
    int iYBegin = srcROI.y + (int) floorYBegin;
    int iYEnd   = srcROI.y + (int) ceilYEnd;

    // first row
    int pos = iYBegin * srcStep + iXBegin;

    float wsum = 1.0f - yBegin + floorYBegin;

    float sum = processLine (pos, xBegin, xEnd, iXBegin, iXEnd, floorXBegin,
        ceilXEnd) * (1.0f - yBegin + floorYBegin);
    pos += srcStep;
    for (int iy = iYBegin + 1; iy < iYEnd; ++iy)
    {
        sum += processLine (pos, xBegin, xEnd, iXBegin, iXEnd, floorXBegin,
            ceilXEnd);
        pos += srcStep;
        wsum += 1.0f;
    }

    sum += processLine (pos, xBegin, xEnd, iXBegin, iXEnd, floorXBegin,
        ceilXEnd) * (ceilYEnd - yEnd);
    wsum += ceilYEnd - yEnd;
    sum /= wsum;

    dst[(ix + dstROI.x) + (iy + dstROI.y) * dstStep] = sum;
}


// bicubic interpolation
__forceinline__
__device__ float bicubicCoeff(float x_)
{
    float x = fabsf(x_);
    if (x <= 1.0f)
    {
        return x * x * (1.5f * x - 2.5f) + 1.0f;
    }
    else if (x < 2.0f)
    {
        return x * (x * (-0.5f * x + 2.5f) - 4.0f) + 2.0f;
    }
    else
    {
        return 0.0f;
    }
}


__global__ void resizeBicubic(NcvSize32u srcSize,
                              NcvRect32u srcROI,
                              NcvSize32u dstSize,
                              Ncv32u dstStep,
                              Ncv32f *dst,
                              NcvRect32u dstROI,
                              Ncv32f scaleX,
                              Ncv32f scaleY)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= dstROI.width || iy >= dstROI.height)
    {
        return;
    }

    const float dx = 1.0f / srcROI.width;
    const float dy = 1.0f / srcROI.height;

    float rx = (float) srcROI.x;
    float ry = (float) srcROI.y;

    float rw = (float) srcROI.width;
    float rh = (float) srcROI.height;

    float x = scaleX * (float) ix;
    float y = scaleY * (float) iy;

    // sampling range
    // border mode is clamp
    float xmin = fmax (ceilf (x - 2.0f), 0.0f);
    float xmax = fmin (floorf (x + 2.0f), rw - 1.0f);

    float ymin = fmax (ceilf (y - 2.0f), 0.0f);
    float ymax = fmin (floorf (y + 2.0f), rh - 1.0f);

    // shift data window to match ROI
    rx += 0.5f;
    ry += 0.5f;

    x += rx;
    y += ry;

    xmin += rx;
    xmax += rx;
    ymin += ry;
    ymax += ry;

    float sum  = 0.0f;
    float wsum = 0.0f;

    for (float cy = ymin; cy <= ymax; cy += 1.0f)
    {
        for (float cx = xmin; cx <= xmax; cx += 1.0f)
        {
            float xDist = x - cx;
            float yDist = y - cy;
            float wx = bicubicCoeff (xDist);
            float wy = bicubicCoeff (yDist);
            wx *= wy;
            sum += wx * tex2D (texSrc2D, cx * dx, cy * dy);
            wsum += wx;
        }
    }
    dst[(ix + dstROI.x)+ (iy + dstROI.y) * dstStep] = (!wsum)? 0 : sum / wsum;
}


NCVStatus nppiStResize_32f_C1R(const Ncv32f *pSrc,
                               NcvSize32u srcSize,
                               Ncv32u nSrcStep,
                               NcvRect32u srcROI,
                               Ncv32f *pDst,
                               NcvSize32u dstSize,
                               Ncv32u nDstStep,
                               NcvRect32u dstROI,
                               Ncv32f xFactor,
                               Ncv32f yFactor,
                               NppStInterpMode interpolation)
{
    NCVStatus status = NPPST_SUCCESS;

    ncvAssertReturn (pSrc != NULL && pDst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn (xFactor != 0.0 && yFactor != 0.0, NPPST_INVALID_SCALE);

    ncvAssertReturn (nSrcStep >= sizeof (Ncv32f) * (Ncv32u) srcSize.width && 
        nDstStep >= sizeof (Ncv32f) * (Ncv32f) dstSize.width,
        NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u dstStep = nDstStep / sizeof (Ncv32f);

    // TODO: preprocess ROI to prevent out of bounds access

    if (interpolation == nppStSupersample)
    {
        // bind texture
        hipBindTexture (0, texSrc, pSrc, srcSize.height * nSrcStep);
        // invoke kernel
        dim3 ctaSize (32, 6);
        dim3 gridSize ((dstROI.width  + ctaSize.x - 1) / ctaSize.x,
            (dstROI.height + ctaSize.y - 1) / ctaSize.y);

        resizeSuperSample_32f <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream ()>>> 
            (srcSize, srcStep, srcROI, pDst, dstSize, dstStep, dstROI, 1.0f / xFactor, 1.0f / yFactor);
    }
    else if (interpolation == nppStBicubic)
    {
        texSrc2D.addressMode[0] = hipAddressModeMirror;
        texSrc2D.addressMode[1] = hipAddressModeMirror;
        texSrc2D.normalized = true;

        hipChannelFormatDesc desc = hipCreateChannelDesc <float> ();

        hipBindTexture2D (0, texSrc2D, pSrc, desc, srcSize.width, srcSize.height,
            nSrcStep);

        dim3 ctaSize (32, 6);
        dim3 gridSize ((dstSize.width  + ctaSize.x - 1) / ctaSize.x,
            (dstSize.height + ctaSize.y - 1) / ctaSize.y);

        resizeBicubic <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream ()>>>
            (srcSize, srcROI, dstSize, dstStep, pDst, dstROI, 1.0f / xFactor, 1.0f / yFactor);
    }
    else
    {
        status = NPPST_ERROR;
    }

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return status;
}
