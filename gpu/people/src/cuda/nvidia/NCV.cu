#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (C) 2009-2010, NVIDIA Corporation, all rights reserved.
 *  Third party copyrights are property of their respective owners.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 * $Id:  $
 * Ported to PCL by Koen Buys : Attention Work in progress!
 */

#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include "NCV.hpp"


//==============================================================================
//
// Error handling helpers
//
//==============================================================================


static void stdDebugOutput(const std::string &msg)
{
    std::cout << msg;
}


static NCVDebugOutputHandler *debugOutputHandler = stdDebugOutput;


void ncvDebugOutput(const std::string &msg)
{
    debugOutputHandler(msg);
}


void ncvSetDebugOutputHandler(NCVDebugOutputHandler *func)
{
    debugOutputHandler = func;
}


//==============================================================================
//
// Memory wrappers and helpers
//
//==============================================================================


Ncv32u alignUp(Ncv32u what, Ncv32u alignment)
{
    Ncv32u alignMask = alignment-1;
    Ncv32u inverseAlignMask = ~alignMask;
    Ncv32u res = (what + alignMask) & inverseAlignMask;
    return res;
}


void NCVMemPtr::clear()
{
    ptr = NULL;
    memtype = NCVMemoryTypeNone;
}


void NCVMemSegment::clear()
{
    begin.clear();
    size = 0;
}


NCVStatus memSegCopyHelper(void *dst, NCVMemoryType dstType, const void *src, NCVMemoryType srcType, std::size_t sz, hipStream_t cuStream)
{
    NCVStatus ncvStat;
    switch (dstType)
    {
    case NCVMemoryTypeHostPageable:
    case NCVMemoryTypeHostPinned:
        switch (srcType)
        {
        case NCVMemoryTypeHostPageable:
        case NCVMemoryTypeHostPinned:
            memcpy(dst, src, sz);
            ncvStat = NCV_SUCCESS;
            break;
        case NCVMemoryTypeDevice:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpyAsync(dst, src, sz, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy(dst, src, sz, hipMemcpyDeviceToHost), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        default:
            ncvStat = NCV_MEM_RESIDENCE_ERROR;
        }
        break;
    case NCVMemoryTypeDevice:
        switch (srcType)
        {
        case NCVMemoryTypeHostPageable:
        case NCVMemoryTypeHostPinned:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpyAsync(dst, src, sz, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy(dst, src, sz, hipMemcpyHostToDevice), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        case NCVMemoryTypeDevice:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpyAsync(dst, src, sz, hipMemcpyDeviceToDevice, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy(dst, src, sz, hipMemcpyDeviceToDevice), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        default:
            ncvStat = NCV_MEM_RESIDENCE_ERROR;
        }
        break;
    default:
        ncvStat = NCV_MEM_RESIDENCE_ERROR;
    }

    return ncvStat;
}


NCVStatus memSegCopyHelper2D(void *dst, Ncv32u dstPitch, NCVMemoryType dstType,
                             const void *src, Ncv32u srcPitch, NCVMemoryType srcType,
                             Ncv32u widthbytes, Ncv32u height, hipStream_t cuStream)
{
    NCVStatus ncvStat;
    switch (dstType)
    {
    case NCVMemoryTypeHostPageable:
    case NCVMemoryTypeHostPinned:
        switch (srcType)
        {
        case NCVMemoryTypeHostPageable:
        case NCVMemoryTypeHostPinned:
            for (Ncv32u i=0; i<height; i++)
            {
                memcpy((char*)dst + i * dstPitch, (char*)src + i * srcPitch, widthbytes);
            }
            ncvStat = NCV_SUCCESS;
            break;
        case NCVMemoryTypeDevice:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpy2DAsync(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy2D(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyDeviceToHost), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        default:
            ncvStat = NCV_MEM_RESIDENCE_ERROR;
        }
        break;
    case NCVMemoryTypeDevice:
        switch (srcType)
        {
        case NCVMemoryTypeHostPageable:
        case NCVMemoryTypeHostPinned:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpy2DAsync(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy2D(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyHostToDevice), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        case NCVMemoryTypeDevice:
            if (cuStream != 0)
            {
                ncvAssertCUDAReturn(hipMemcpy2DAsync(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyDeviceToDevice, cuStream), NCV_CUDA_ERROR);
            }
            else
            {
                ncvAssertCUDAReturn(hipMemcpy2D(dst, dstPitch, src, srcPitch, widthbytes, height, hipMemcpyDeviceToDevice), NCV_CUDA_ERROR);
            }
            ncvStat = NCV_SUCCESS;
            break;
        default:
            ncvStat = NCV_MEM_RESIDENCE_ERROR;
        }
        break;
    default:
        ncvStat = NCV_MEM_RESIDENCE_ERROR;
    }

    return ncvStat;
}


//===================================================================
//
// NCVMemStackAllocator class members implementation
//
//===================================================================


NCVMemStackAllocator::NCVMemStackAllocator(Ncv32u alignment)
    :
    currentSize(0),
    _maxSize(0),
    allocBegin(NULL),
    begin(NULL),
    end(NULL),
    _memType(NCVMemoryTypeNone),
    _alignment(alignment),
    bReusesMemory(false)
{
    NcvBool bProperAlignment = (alignment & (alignment-1)) == 0;
    ncvAssertPrintCheck(bProperAlignment, "NCVMemStackAllocator ctor:: alignment not power of 2");
}


NCVMemStackAllocator::NCVMemStackAllocator(NCVMemoryType memT, std::size_t capacity, Ncv32u alignment, void *reusePtr)
    :
    currentSize(0),
    _maxSize(0),
    allocBegin(NULL),
    _memType(memT),
    _alignment(alignment)
{
    NcvBool bProperAlignment = (alignment & (alignment-1)) == 0;
    ncvAssertPrintCheck(bProperAlignment, "NCVMemStackAllocator ctor:: _alignment not power of 2");
    ncvAssertPrintCheck(memT != NCVMemoryTypeNone, "NCVMemStackAllocator ctor:: Incorrect allocator type");

    allocBegin = NULL;

    if (reusePtr == NULL && capacity != 0)
    {
        bReusesMemory = false;
        switch (memT)
        {
        case NCVMemoryTypeDevice:
            ncvAssertCUDAReturn(hipMalloc(&allocBegin, capacity), );
            break;
        case NCVMemoryTypeHostPinned:
            ncvAssertCUDAReturn(hipHostMalloc(&allocBegin, capacity), );
            break;
        case NCVMemoryTypeHostPageable:
            allocBegin = (Ncv8u *)malloc(capacity);
            break;
        default:;
        }
    }
    else
    {
        bReusesMemory = true;
        allocBegin = (Ncv8u *)reusePtr;
    }

    if (capacity == 0)
    {
        allocBegin = (Ncv8u *)(0x1);
    }

    if (!isCounting())
    {
        begin = allocBegin;
        end = begin + capacity;
    }
}


NCVMemStackAllocator::~NCVMemStackAllocator()
{
    if (allocBegin != NULL)
    {
        ncvAssertPrintCheck(currentSize == 0, "NCVMemStackAllocator dtor:: not all objects were deallocated properly, forcing destruction");

        if (!bReusesMemory && (allocBegin != (Ncv8u *)(0x1)))
        {
            switch (_memType)
            {
            case NCVMemoryTypeDevice:
                ncvAssertCUDAReturn(hipFree(allocBegin), );
                break;
            case NCVMemoryTypeHostPinned:
                ncvAssertCUDAReturn(hipHostFree(allocBegin), );
                break;
            case NCVMemoryTypeHostPageable:
                free(allocBegin);
                break;
            default:;
            }
        }

        allocBegin = NULL;
    }
}


NCVStatus NCVMemStackAllocator::alloc(NCVMemSegment &seg, std::size_t size)
{
    seg.clear();
    ncvAssertReturn(isInitialized(), NCV_ALLOCATOR_BAD_ALLOC);

    size = alignUp(size, this->_alignment);
    this->currentSize += size;
    this->_maxSize = max(this->_maxSize, this->currentSize);

    if (!isCounting())
    {
        std::size_t availSize = end - begin;
        ncvAssertReturn(size <= availSize, NCV_ALLOCATOR_INSUFFICIENT_CAPACITY);
    }

    seg.begin.ptr = begin;
    seg.begin.memtype = this->_memType;
    seg.size = size;
    begin += size;

    return NCV_SUCCESS;
}


NCVStatus NCVMemStackAllocator::dealloc(NCVMemSegment &seg)
{
    ncvAssertReturn(isInitialized(), NCV_ALLOCATOR_BAD_ALLOC);
    ncvAssertReturn(seg.begin.memtype == this->_memType, NCV_ALLOCATOR_BAD_DEALLOC);
    ncvAssertReturn(seg.begin.ptr != NULL || isCounting(), NCV_ALLOCATOR_BAD_DEALLOC);
    ncvAssertReturn(seg.begin.ptr == begin - seg.size, NCV_ALLOCATOR_DEALLOC_ORDER);

    currentSize -= seg.size;
    begin -= seg.size;

    seg.clear();

    ncvAssertReturn(allocBegin <= begin, NCV_ALLOCATOR_BAD_DEALLOC);

    return NCV_SUCCESS;
}


NcvBool NCVMemStackAllocator::isInitialized(void) const
{
    return ((this->_alignment & (this->_alignment-1)) == 0) && isCounting() || this->allocBegin != NULL;
}


NcvBool NCVMemStackAllocator::isCounting(void) const
{
    return this->_memType == NCVMemoryTypeNone;
}


NCVMemoryType NCVMemStackAllocator::memType(void) const
{
    return this->_memType;
}


Ncv32u NCVMemStackAllocator::alignment(void) const
{
    return this->_alignment;
}


size_t NCVMemStackAllocator::maxSize(void) const
{
    return this->_maxSize;
}


//===================================================================
//
// NCVMemNativeAllocator class members implementation
//
//===================================================================


NCVMemNativeAllocator::NCVMemNativeAllocator(NCVMemoryType memT, Ncv32u alignment)
    :
    currentSize(0),
    _maxSize(0),
    _memType(memT),
    _alignment(alignment)
{
    ncvAssertPrintReturn(memT != NCVMemoryTypeNone, "NCVMemNativeAllocator ctor:: counting not permitted for this allocator type", );
}


NCVMemNativeAllocator::~NCVMemNativeAllocator()
{
    ncvAssertPrintCheck(currentSize == 0, "NCVMemNativeAllocator dtor:: detected memory leak");
}


NCVStatus NCVMemNativeAllocator::alloc(NCVMemSegment &seg, std::size_t size)
{
    seg.clear();
    ncvAssertReturn(isInitialized(), NCV_ALLOCATOR_BAD_ALLOC);

    switch (this->_memType)
    {
    case NCVMemoryTypeDevice:
        ncvAssertCUDAReturn(hipMalloc(&seg.begin.ptr, size), NCV_CUDA_ERROR);
        break;
    case NCVMemoryTypeHostPinned:
        ncvAssertCUDAReturn(hipHostMalloc(&seg.begin.ptr, size), NCV_CUDA_ERROR);
        break;
    case NCVMemoryTypeHostPageable:
        seg.begin.ptr = (Ncv8u *)malloc(size);
        break;
    default:;
    }

    this->currentSize += alignUp(size, this->_alignment);
    this->_maxSize = max(this->_maxSize, this->currentSize);

    seg.begin.memtype = this->_memType;
    seg.size = size;

    return NCV_SUCCESS;
}


NCVStatus NCVMemNativeAllocator::dealloc(NCVMemSegment &seg)
{
    ncvAssertReturn(isInitialized(), NCV_ALLOCATOR_BAD_ALLOC);
    ncvAssertReturn(seg.begin.memtype == this->_memType, NCV_ALLOCATOR_BAD_DEALLOC);
    ncvAssertReturn(seg.begin.ptr != NULL, NCV_ALLOCATOR_BAD_DEALLOC);

    ncvAssertReturn(currentSize >= alignUp(seg.size, this->_alignment), NCV_ALLOCATOR_BAD_DEALLOC);
    currentSize -= alignUp(seg.size, this->_alignment);

    switch (this->_memType)
    {
    case NCVMemoryTypeDevice:
        ncvAssertCUDAReturn(hipFree(seg.begin.ptr), NCV_CUDA_ERROR);
        break;
    case NCVMemoryTypeHostPinned:
        ncvAssertCUDAReturn(hipHostFree(seg.begin.ptr), NCV_CUDA_ERROR);
        break;
    case NCVMemoryTypeHostPageable:
        free(seg.begin.ptr);
        break;
    default:;
    }

    seg.clear();

    return NCV_SUCCESS;
}


NcvBool NCVMemNativeAllocator::isInitialized(void) const
{
    return (this->_alignment != 0);
}


NcvBool NCVMemNativeAllocator::isCounting(void) const
{
    return false;
}


NCVMemoryType NCVMemNativeAllocator::memType(void) const
{
    return this->_memType;
}


Ncv32u NCVMemNativeAllocator::alignment(void) const
{
    return this->_alignment;
}


size_t NCVMemNativeAllocator::maxSize(void) const
{
    return this->_maxSize;
}


//===================================================================
//
// Operations with rectangles
//
//===================================================================


template <class T>
static NCVStatus drawRectsWrapperHost(T *h_dst,
                                      Ncv32u dstStride,
                                      Ncv32u dstWidth,
                                      Ncv32u dstHeight,
                                      NcvRect32u *h_rects,
                                      Ncv32u numRects,
                                      T color)
{
    ncvAssertReturn(h_dst != NULL && h_rects != NULL, NCV_NULL_PTR);
    ncvAssertReturn(dstWidth > 0 && dstHeight > 0, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(dstStride >= dstWidth, NCV_INVALID_STEP);
    ncvAssertReturn(numRects != 0, NCV_SUCCESS);
    ncvAssertReturn(numRects <= dstWidth * dstHeight, NCV_DIMENSIONS_INVALID);

    for (Ncv32u i=0; i<numRects; i++)
    {
        NcvRect32u rect = h_rects[i];

        if (rect.x < dstWidth)
        {
            for (Ncv32u i=rect.y; i<rect.y+rect.height && i<dstHeight; i++)
            {
                h_dst[i*dstStride+rect.x] = color;
            }
        }
        if (rect.x+rect.width-1 < dstWidth)
        {
            for (Ncv32u i=rect.y; i<rect.y+rect.height && i<dstHeight; i++)
            {
                h_dst[i*dstStride+rect.x+rect.width-1] = color;
            }
        }
        if (rect.y < dstHeight)
        {
            for (Ncv32u j=rect.x; j<rect.x+rect.width && j<dstWidth; j++)
            {
                h_dst[rect.y*dstStride+j] = color;
            }
        }
        if (rect.y + rect.height - 1 < dstHeight)
        {
            for (Ncv32u j=rect.x; j<rect.x+rect.width && j<dstWidth; j++)
            {
                h_dst[(rect.y+rect.height-1)*dstStride+j] = color;
            }
        }
    }

    return NCV_SUCCESS;
}


NCVStatus ncvDrawRects_8u_host(Ncv8u *h_dst,
                               Ncv32u dstStride,
                               Ncv32u dstWidth,
                               Ncv32u dstHeight,
                               NcvRect32u *h_rects,
                               Ncv32u numRects,
                               Ncv8u color)
{
    return drawRectsWrapperHost(h_dst, dstStride, dstWidth, dstHeight, h_rects, numRects, color);
}


NCVStatus ncvDrawRects_32u_host(Ncv32u *h_dst,
                                Ncv32u dstStride,
                                Ncv32u dstWidth,
                                Ncv32u dstHeight,
                                NcvRect32u *h_rects,
                                Ncv32u numRects,
                                Ncv32u color)
{
    return drawRectsWrapperHost(h_dst, dstStride, dstWidth, dstHeight, h_rects, numRects, color);
}


const Ncv32u NUMTHREADS_DRAWRECTS = 32;
const Ncv32u NUMTHREADS_DRAWRECTS_LOG2 = 5;


template <class T>
__global__ void drawRects(T *d_dst,
                          Ncv32u dstStride,
                          Ncv32u dstWidth,
                          Ncv32u dstHeight,
                          NcvRect32u *d_rects,
                          Ncv32u numRects,
                          T color)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    if (blockId > numRects * 4)
    {
        return;
    }

    NcvRect32u curRect = d_rects[blockId >> 2];
    NcvBool bVertical = blockId & 0x1;
    NcvBool bTopLeft = blockId & 0x2;

    Ncv32u pt0x, pt0y;
    if (bVertical)
    {
        Ncv32u numChunks = (curRect.height + NUMTHREADS_DRAWRECTS - 1) >> NUMTHREADS_DRAWRECTS_LOG2;

        pt0x = bTopLeft ? curRect.x : curRect.x + curRect.width - 1;
        pt0y = curRect.y;

        if (pt0x < dstWidth)
        {
            for (Ncv32u chunkId = 0; chunkId < numChunks; chunkId++)
            {
                Ncv32u ptY = pt0y + chunkId * NUMTHREADS_DRAWRECTS + threadIdx.x;
                if (ptY < pt0y + curRect.height && ptY < dstHeight)
                {
                    d_dst[ptY * dstStride + pt0x] = color;
                }
            }
        }
    }
    else
    {
        Ncv32u numChunks = (curRect.width + NUMTHREADS_DRAWRECTS - 1) >> NUMTHREADS_DRAWRECTS_LOG2;

        pt0x = curRect.x;
        pt0y = bTopLeft ? curRect.y : curRect.y + curRect.height - 1;

        if (pt0y < dstHeight)
        {
            for (Ncv32u chunkId = 0; chunkId < numChunks; chunkId++)
            {
                Ncv32u ptX = pt0x + chunkId * NUMTHREADS_DRAWRECTS + threadIdx.x;
                if (ptX < pt0x + curRect.width && ptX < dstWidth)
                {
                    d_dst[pt0y * dstStride + ptX] = color;
                }
            }
        }
    }
}


template <class T>
static NCVStatus drawRectsWrapperDevice(T *d_dst,
                                        Ncv32u dstStride,
                                        Ncv32u dstWidth,
                                        Ncv32u dstHeight,
                                        NcvRect32u *d_rects,
                                        Ncv32u numRects,
                                        T color,
                                        hipStream_t cuStream)
{
    ncvAssertReturn(d_dst != NULL && d_rects != NULL, NCV_NULL_PTR);
    ncvAssertReturn(dstWidth > 0 && dstHeight > 0, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(dstStride >= dstWidth, NCV_INVALID_STEP);
    ncvAssertReturn(numRects <= dstWidth * dstHeight, NCV_DIMENSIONS_INVALID);

    if (numRects == 0)
    {
        return NCV_SUCCESS;
    }

    dim3 grid(numRects * 4);
    dim3 block(NUMTHREADS_DRAWRECTS);
    if (grid.x > 65535)
    {
        grid.y = (grid.x + 65534) / 65535;
        grid.x = 65535;
    }

    drawRects<T><<<grid, block>>>(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color);

    ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);

    return NCV_SUCCESS;
}


NCVStatus ncvDrawRects_8u_device(Ncv8u *d_dst,
                                 Ncv32u dstStride,
                                 Ncv32u dstWidth,
                                 Ncv32u dstHeight,
                                 NcvRect32u *d_rects,
                                 Ncv32u numRects,
                                 Ncv8u color,
                                 hipStream_t cuStream)
{
    return drawRectsWrapperDevice(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color, cuStream);
}


NCVStatus ncvDrawRects_32u_device(Ncv32u *d_dst,
                                  Ncv32u dstStride,
                                  Ncv32u dstWidth,
                                  Ncv32u dstHeight,
                                  NcvRect32u *d_rects,
                                  Ncv32u numRects,
                                  Ncv32u color,
                                  hipStream_t cuStream)
{
    return drawRectsWrapperDevice(d_dst, dstStride, dstWidth, dstHeight, d_rects, numRects, color, cuStream);
}
