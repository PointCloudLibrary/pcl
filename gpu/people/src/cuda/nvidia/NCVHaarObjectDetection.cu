#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (C) 2009-2010, NVIDIA Corporation, all rights reserved.
 *  Third party copyrights are property of their respective owners.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 * $Id:  $
 * Ported to PCL by Koen Buys : Attention Work in progress!
 */

////////////////////////////////////////////////////////////////////////////////
//
// NVIDIA CUDA implementation of Viola-Jones Object Detection Framework
//
// The algorithm and code are explained in the upcoming GPU Computing Gems
// chapter in detail:
//
//   Anton Obukhov, "Haar Classifiers for Object Detection with CUDA"
//   PDF URL placeholder
//   email: aobukhov@nvidia.com, devsupport@nvidia.com
//
// Credits for help with the code to:
// Alexey Mendelenko, Cyril Crassin, and Mikhail Smirnov.
//
////////////////////////////////////////////////////////////////////////////////

#include <algorithm>
#include <cstdio>

#include "NCV.hpp"
#include "NCVAlg.hpp"
#include "NPP_staging.hpp"
#include "NCVRuntimeTemplates.hpp"
#include "NCVHaarObjectDetection.hpp"


//==============================================================================
//
// BlockScan file
//
//==============================================================================


NCV_CT_ASSERT(K_WARP_SIZE == 32); //this is required for the manual unroll of the loop in warpScanInclusive


//Almost the same as naive scan1Inclusive, but doesn't need __syncthreads()
//assuming size <= WARP_SIZE and size is power of 2
__device__ Ncv32u warpScanInclusive(Ncv32u idata, volatile Ncv32u *s_Data)
{
    Ncv32u pos = 2 * threadIdx.x - (threadIdx.x & (K_WARP_SIZE - 1));
    s_Data[pos] = 0;
    pos += K_WARP_SIZE;
    s_Data[pos] = idata;

    s_Data[pos] += s_Data[pos - 1];
    s_Data[pos] += s_Data[pos - 2];
    s_Data[pos] += s_Data[pos - 4];
    s_Data[pos] += s_Data[pos - 8];
    s_Data[pos] += s_Data[pos - 16];

    return s_Data[pos];
}

__device__ __forceinline__ Ncv32u warpScanExclusive(Ncv32u idata, volatile Ncv32u *s_Data)
{
    return warpScanInclusive(idata, s_Data) - idata;
}

template <Ncv32u tiNumScanThreads>
__device__ Ncv32u scan1Inclusive(Ncv32u idata, volatile Ncv32u *s_Data)
{
    if (tiNumScanThreads > K_WARP_SIZE)
    {
        //Bottom-level inclusive warp scan
        Ncv32u warpResult = warpScanInclusive(idata, s_Data);

        //Save top elements of each warp for exclusive warp scan
        //sync to wait for warp scans to complete (because s_Data is being overwritten)
        __syncthreads();
        if( (threadIdx.x & (K_WARP_SIZE - 1)) == (K_WARP_SIZE - 1) )
        {
            s_Data[threadIdx.x >> K_LOG2_WARP_SIZE] = warpResult;
        }

        //wait for warp scans to complete
        __syncthreads();

        if( threadIdx.x < (tiNumScanThreads / K_WARP_SIZE) )
        {
            //grab top warp elements
            Ncv32u val = s_Data[threadIdx.x];
            //calculate exclusive scan and write back to shared memory
            s_Data[threadIdx.x] = warpScanExclusive(val, s_Data);
        }

        //return updated warp scans with exclusive scan results
        __syncthreads();
        return warpResult + s_Data[threadIdx.x >> K_LOG2_WARP_SIZE];
    }
    else
    {
        return warpScanInclusive(idata, s_Data);
    }
}


//==============================================================================
//
// HaarClassifierCascade file
//
//==============================================================================


const Ncv32u MAX_GRID_DIM = 65535;


const Ncv32u NUM_THREADS_ANCHORSPARALLEL = 64;


#define NUM_THREADS_CLASSIFIERPARALLEL_LOG2     6
#define NUM_THREADS_CLASSIFIERPARALLEL          (1 << NUM_THREADS_CLASSIFIERPARALLEL_LOG2)


/** \internal
* Haar features solid array.
*/
texture<uint2, 1, hipReadModeElementType> texHaarFeatures;


/** \internal
* Haar classifiers flattened trees container.
* Two parts: first contains root nodes, second - nodes that are referred by root nodes.
* Drawback: breaks tree locality (might cause more cache misses
* Advantage: No need to introduce additional 32-bit field to index root nodes offsets
*/
texture<uint4, 1, hipReadModeElementType> texHaarClassifierNodes;


texture<Ncv32u, 1, hipReadModeElementType> texIImage;


__device__ HaarStage64 getStage(Ncv32u iStage, HaarStage64 *d_Stages)
{
    return d_Stages[iStage];
}


template <NcvBool tbCacheTextureCascade>
__device__ HaarClassifierNode128 getClassifierNode(Ncv32u iNode, HaarClassifierNode128 *d_ClassifierNodes)
{
    HaarClassifierNode128 tmpNode;
    if (tbCacheTextureCascade)
    {
        tmpNode._ui4 = tex1Dfetch(texHaarClassifierNodes, iNode);
    }
    else
    {
        tmpNode = d_ClassifierNodes[iNode];
    }
    return tmpNode;
}


template <NcvBool tbCacheTextureCascade>
__device__ void getFeature(Ncv32u iFeature, HaarFeature64 *d_Features,
                           Ncv32f *weight,
                           Ncv32u *rectX, Ncv32u *rectY, Ncv32u *rectWidth, Ncv32u *rectHeight)
{
    HaarFeature64 feature;
    if (tbCacheTextureCascade)
    {
        feature._ui2 = tex1Dfetch(texHaarFeatures, iFeature);
    }
    else
    {
        feature = d_Features[iFeature];
    }
    feature.getRect(rectX, rectY, rectWidth, rectHeight);
    *weight = feature.getWeight();
}


template <NcvBool tbCacheTextureIImg>
__device__ Ncv32u getElemIImg(Ncv32u x, Ncv32u *d_IImg)
{
    if (tbCacheTextureIImg)
    {
        return tex1Dfetch(texIImage, x);
    }
    else
    {
        return d_IImg[x];
    }
}


__device__ Ncv32u d_outMaskPosition;


__device__ void compactBlockWriteOutAnchorParallel(Ncv32u threadPassFlag, Ncv32u threadElem, Ncv32u *vectorOut)
{
    __shared__ Ncv32u shmem[NUM_THREADS_ANCHORSPARALLEL * 2];
    __shared__ Ncv32u numPassed;
    __shared__ Ncv32u outMaskOffset;

    Ncv32u incScan = scan1Inclusive<NUM_THREADS_ANCHORSPARALLEL>(threadPassFlag, shmem);
    __syncthreads();

    if (threadIdx.x == NUM_THREADS_ANCHORSPARALLEL-1)
    {
        numPassed = incScan;
        outMaskOffset = atomicAdd(&d_outMaskPosition, incScan);
    }

    if (threadPassFlag)
    {
        Ncv32u excScan = incScan - threadPassFlag;
        shmem[excScan] = threadElem;
    }

    __syncthreads();

    if (threadIdx.x < numPassed)
    {
        vectorOut[outMaskOffset + threadIdx.x] = shmem[threadIdx.x];
    }
}


template <NcvBool tbInitMaskPositively,
          NcvBool tbCacheTextureIImg,
          NcvBool tbCacheTextureCascade,
          NcvBool tbReadPixelIndexFromVector,
          NcvBool tbDoAtomicCompaction>
__global__ void applyHaarClassifierAnchorParallel(Ncv32u *d_IImg, Ncv32u IImgStride,
                                                  Ncv32f *d_weights, Ncv32u weightsStride,
                                                  HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages,
                                                  Ncv32u *d_inMask, Ncv32u *d_outMask,
                                                  Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                                  NcvSize32u anchorsRoi, Ncv32u startStageInc, Ncv32u endStageExc, Ncv32f scaleArea)
{
    Ncv32u y_offs;
    Ncv32u x_offs;
    Ncv32u maskOffset;
    Ncv32u outMaskVal;

    NcvBool bInactiveThread = false;

    if (tbReadPixelIndexFromVector)
    {
        maskOffset = (MAX_GRID_DIM * blockIdx.y + blockIdx.x) * NUM_THREADS_ANCHORSPARALLEL + threadIdx.x;

        if (maskOffset >= mask1Dlen)
        {
            if (tbDoAtomicCompaction) bInactiveThread = true; else return;
        }

        if (!tbDoAtomicCompaction || tbDoAtomicCompaction && !bInactiveThread)
        {
            outMaskVal = d_inMask[maskOffset];
            y_offs = outMaskVal >> 16;
            x_offs = outMaskVal & 0xFFFF;
        }
    }
    else
    {
        y_offs = blockIdx.y;
        x_offs = blockIdx.x * NUM_THREADS_ANCHORSPARALLEL + threadIdx.x;

        if (x_offs >= mask2Dstride)
        {
            if (tbDoAtomicCompaction) bInactiveThread = true; else return;
        }

        if (!tbDoAtomicCompaction || tbDoAtomicCompaction && !bInactiveThread)
        {
            maskOffset = y_offs * mask2Dstride + x_offs;

            if ((x_offs >= anchorsRoi.width) ||
                (!tbInitMaskPositively &&
                 d_inMask != d_outMask &&
                 d_inMask[maskOffset] == OBJDET_MASK_ELEMENT_INVALID_32U))
            {
                if (tbDoAtomicCompaction)
                {
                    bInactiveThread = true;
                }
                else
                {
                    d_outMask[maskOffset] = OBJDET_MASK_ELEMENT_INVALID_32U;
                    return;
                }
            }

            outMaskVal = (y_offs << 16) | x_offs;
        }
    }

    NcvBool bPass = true;

    if (!tbDoAtomicCompaction || tbDoAtomicCompaction)
    {
        Ncv32f pixelStdDev = 0.0f;

        if (!bInactiveThread)
            pixelStdDev = d_weights[y_offs * weightsStride + x_offs];

        for (Ncv32u iStage = startStageInc; iStage < endStageExc; iStage++)
        {
            Ncv32f curStageSum = 0.0f;

            HaarStage64 curStage = getStage(iStage, d_Stages);
            Ncv32u numRootNodesInStage = curStage.getNumClassifierRootNodes();
            Ncv32u curRootNodeOffset = curStage.getStartClassifierRootNodeOffset();
            Ncv32f stageThreshold = curStage.getStageThreshold();

            while (numRootNodesInStage--)
            {
                NcvBool bMoreNodesToTraverse = true;
                Ncv32u iNode = curRootNodeOffset;

                if (bPass && !bInactiveThread)
                {
                    while (bMoreNodesToTraverse)
                    {
                        HaarClassifierNode128 curNode = getClassifierNode<tbCacheTextureCascade>(iNode, d_ClassifierNodes);
                        HaarFeatureDescriptor32 featuresDesc = curNode.getFeatureDesc();
                        Ncv32u curNodeFeaturesNum = featuresDesc.getNumFeatures();
                        Ncv32u iFeature = featuresDesc.getFeaturesOffset();

                        Ncv32f curNodeVal = 0.0f;

                        for (Ncv32u iRect=0; iRect<curNodeFeaturesNum; iRect++)
                        {
                            Ncv32f rectWeight;
                            Ncv32u rectX, rectY, rectWidth, rectHeight;
                            getFeature<tbCacheTextureCascade>
                                (iFeature + iRect, d_Features,
                                &rectWeight, &rectX, &rectY, &rectWidth, &rectHeight);

                            Ncv32u iioffsTL = (y_offs + rectY) * IImgStride + (x_offs + rectX);
                            Ncv32u iioffsTR = iioffsTL + rectWidth;
                            Ncv32u iioffsBL = iioffsTL + rectHeight * IImgStride;
                            Ncv32u iioffsBR = iioffsBL + rectWidth;

                            Ncv32u rectSum = getElemIImg<tbCacheTextureIImg>(iioffsBR, d_IImg) -
                                             getElemIImg<tbCacheTextureIImg>(iioffsBL, d_IImg) +
                                             getElemIImg<tbCacheTextureIImg>(iioffsTL, d_IImg) -
                                             getElemIImg<tbCacheTextureIImg>(iioffsTR, d_IImg);

    #if defined CPU_FP_COMPLIANCE || defined DISABLE_MAD_SELECTIVELY
                        curNodeVal += __fmul_rn((Ncv32f)rectSum, rectWeight);
    #else
                        curNodeVal += (Ncv32f)rectSum * rectWeight;
    #endif
                        }

                        HaarClassifierNodeDescriptor32 nodeLeft = curNode.getLeftNodeDesc();
                        HaarClassifierNodeDescriptor32 nodeRight = curNode.getRightNodeDesc();
                        Ncv32f nodeThreshold = curNode.getThreshold();

                        HaarClassifierNodeDescriptor32 nextNodeDescriptor;
                        NcvBool nextNodeIsLeaf;

                        if (curNodeVal < scaleArea * pixelStdDev * nodeThreshold)
                        {
                            nextNodeDescriptor = nodeLeft;
                            nextNodeIsLeaf = featuresDesc.isLeftNodeLeaf();
                        }
                        else
                        {
                            nextNodeDescriptor = nodeRight;
                            nextNodeIsLeaf = featuresDesc.isRightNodeLeaf();
                        }

                        if (nextNodeIsLeaf)
                        {
                            Ncv32f tmpLeafValue = nextNodeDescriptor.getLeafValue();
                            curStageSum += tmpLeafValue;
                            bMoreNodesToTraverse = false;
                        }
                        else
                        {
                            iNode = nextNodeDescriptor.getNextNodeOffset();
                        }
                    }
                }

                __syncthreads();
                curRootNodeOffset++;
            }

            if (curStageSum < stageThreshold)
            {
                bPass = false;
                outMaskVal = OBJDET_MASK_ELEMENT_INVALID_32U;
            }
        }
    }

    __syncthreads();

    if (!tbDoAtomicCompaction)
    {
        if (!tbReadPixelIndexFromVector ||
            (tbReadPixelIndexFromVector && (!bPass || d_inMask != d_outMask)))
        {
            d_outMask[maskOffset] = outMaskVal;
        }
    }
    else
    {
        compactBlockWriteOutAnchorParallel(bPass && !bInactiveThread,
                                           outMaskVal,
                                           d_outMask);
    }
}


template <NcvBool tbCacheTextureIImg,
          NcvBool tbCacheTextureCascade,
          NcvBool tbDoAtomicCompaction>
__global__ void applyHaarClassifierClassifierParallel(Ncv32u *d_IImg, Ncv32u IImgStride,
                                                      Ncv32f *d_weights, Ncv32u weightsStride,
                                                      HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages,
                                                      Ncv32u *d_inMask, Ncv32u *d_outMask,
                                                      Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                                      NcvSize32u anchorsRoi, Ncv32u startStageInc, Ncv32u endStageExc, Ncv32f scaleArea)
{
    Ncv32u maskOffset = MAX_GRID_DIM * blockIdx.y + blockIdx.x;

    if (maskOffset >= mask1Dlen)
    {
        return;
    }

    Ncv32u outMaskVal = d_inMask[maskOffset];
    Ncv32u y_offs = outMaskVal >> 16;
    Ncv32u x_offs = outMaskVal & 0xFFFF;

    Ncv32f pixelStdDev = d_weights[y_offs * weightsStride + x_offs];
    NcvBool bPass = true;

    for (Ncv32u iStage = startStageInc; iStage<endStageExc; iStage++)
    {
        //this variable is subject to reduction
        Ncv32f curStageSum = 0.0f;

        HaarStage64 curStage = getStage(iStage, d_Stages);
        Ncv32s numRootNodesInStage = curStage.getNumClassifierRootNodes();
        Ncv32u curRootNodeOffset = curStage.getStartClassifierRootNodeOffset() + threadIdx.x;
        Ncv32f stageThreshold = curStage.getStageThreshold();

        Ncv32u numRootChunks = (numRootNodesInStage + NUM_THREADS_CLASSIFIERPARALLEL - 1) >> NUM_THREADS_CLASSIFIERPARALLEL_LOG2;

        for (Ncv32u chunkId=0; chunkId<numRootChunks; chunkId++)
        {
            NcvBool bMoreNodesToTraverse = true;

            if (chunkId * NUM_THREADS_CLASSIFIERPARALLEL + threadIdx.x < numRootNodesInStage)
            {
                Ncv32u iNode = curRootNodeOffset;

                while (bMoreNodesToTraverse)
                {
                    HaarClassifierNode128 curNode = getClassifierNode<tbCacheTextureCascade>(iNode, d_ClassifierNodes);
                    HaarFeatureDescriptor32 featuresDesc = curNode.getFeatureDesc();
                    Ncv32u curNodeFeaturesNum = featuresDesc.getNumFeatures();
                    Ncv32u iFeature = featuresDesc.getFeaturesOffset();

                    Ncv32f curNodeVal = 0.0f;
                    //TODO: fetch into shmem if size suffices. Shmem can be shared with reduce
                    for (Ncv32u iRect=0; iRect<curNodeFeaturesNum; iRect++)
                    {
                        Ncv32f rectWeight;
                        Ncv32u rectX, rectY, rectWidth, rectHeight;
                        getFeature<tbCacheTextureCascade>
                            (iFeature + iRect, d_Features,
                            &rectWeight, &rectX, &rectY, &rectWidth, &rectHeight);

                        Ncv32u iioffsTL = (y_offs + rectY) * IImgStride + (x_offs + rectX);
                        Ncv32u iioffsTR = iioffsTL + rectWidth;
                        Ncv32u iioffsBL = iioffsTL + rectHeight * IImgStride;
                        Ncv32u iioffsBR = iioffsBL + rectWidth;

                        Ncv32u rectSum = getElemIImg<tbCacheTextureIImg>(iioffsBR, d_IImg) -
                                         getElemIImg<tbCacheTextureIImg>(iioffsBL, d_IImg) +
                                         getElemIImg<tbCacheTextureIImg>(iioffsTL, d_IImg) -
                                         getElemIImg<tbCacheTextureIImg>(iioffsTR, d_IImg);

#if defined CPU_FP_COMPLIANCE || defined DISABLE_MAD_SELECTIVELY
                        curNodeVal += __fmul_rn((Ncv32f)rectSum, rectWeight);
#else
                        curNodeVal += (Ncv32f)rectSum * rectWeight;
#endif
                    }

                    HaarClassifierNodeDescriptor32 nodeLeft = curNode.getLeftNodeDesc();
                    HaarClassifierNodeDescriptor32 nodeRight = curNode.getRightNodeDesc();
                    Ncv32f nodeThreshold = curNode.getThreshold();

                    HaarClassifierNodeDescriptor32 nextNodeDescriptor;
                    NcvBool nextNodeIsLeaf;

                    if (curNodeVal < scaleArea * pixelStdDev * nodeThreshold)
                    {
                        nextNodeDescriptor = nodeLeft;
                        nextNodeIsLeaf = featuresDesc.isLeftNodeLeaf();
                    }
                    else
                    {
                        nextNodeDescriptor = nodeRight;
                        nextNodeIsLeaf = featuresDesc.isRightNodeLeaf();
                    }

                    if (nextNodeIsLeaf)
                    {
                        Ncv32f tmpLeafValue = nextNodeDescriptor.getLeafValue();
                        curStageSum += tmpLeafValue;
                        bMoreNodesToTraverse = false;
                    }
                    else
                    {
                        iNode = nextNodeDescriptor.getNextNodeOffset();
                    }
                }
            }
            __syncthreads();

            curRootNodeOffset += NUM_THREADS_CLASSIFIERPARALLEL;
        }

        Ncv32f finalStageSum = subReduce<Ncv32f, functorAddValues<Ncv32f>, NUM_THREADS_CLASSIFIERPARALLEL>(curStageSum);

        if (finalStageSum < stageThreshold)
        {
            bPass = false;
            outMaskVal = OBJDET_MASK_ELEMENT_INVALID_32U;
            break;
        }
    }

    if (!tbDoAtomicCompaction)
    {
        if (!bPass || d_inMask != d_outMask)
        {
            if (!threadIdx.x)
            {
                d_outMask[maskOffset] = outMaskVal;
            }
        }
    }
    else
    {
        if (bPass && !threadIdx.x)
        {
            Ncv32u outMaskOffset = atomicAdd(&d_outMaskPosition, 1);
            d_outMask[outMaskOffset] = outMaskVal;
        }
    }
}


template <NcvBool tbMaskByInmask,
          NcvBool tbDoAtomicCompaction>
__global__ void initializeMaskVector(Ncv32u *d_inMask, Ncv32u *d_outMask,
                                     Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                     NcvSize32u anchorsRoi, Ncv32u step)
{
    Ncv32u y_offs = blockIdx.y;
    Ncv32u x_offs = blockIdx.x * NUM_THREADS_ANCHORSPARALLEL + threadIdx.x;
    Ncv32u outMaskOffset = y_offs * gridDim.x * blockDim.x + x_offs;

    Ncv32u y_offs_upsc = step * y_offs;
    Ncv32u x_offs_upsc = step * x_offs;
    Ncv32u inMaskOffset = y_offs_upsc * mask2Dstride + x_offs_upsc;

    Ncv32u outElem = OBJDET_MASK_ELEMENT_INVALID_32U;

    if (x_offs_upsc < anchorsRoi.width &&
        (!tbMaskByInmask || d_inMask[inMaskOffset] != OBJDET_MASK_ELEMENT_INVALID_32U))
    {
        outElem = (y_offs_upsc << 16) | x_offs_upsc;
    }

    if (!tbDoAtomicCompaction)
    {
        d_outMask[outMaskOffset] = outElem;
    }
    else
    {
        compactBlockWriteOutAnchorParallel(outElem != OBJDET_MASK_ELEMENT_INVALID_32U,
                                           outElem,
                                           d_outMask);
    }
}


struct applyHaarClassifierAnchorParallelFunctor
{
    dim3 gridConf, blockConf;
    hipStream_t cuStream;

    //Kernel arguments are stored as members;
    Ncv32u *d_IImg;
    Ncv32u IImgStride;
    Ncv32f *d_weights;
    Ncv32u weightsStride;
    HaarFeature64 *d_Features;
    HaarClassifierNode128 *d_ClassifierNodes;
    HaarStage64 *d_Stages;
    Ncv32u *d_inMask;
    Ncv32u *d_outMask;
    Ncv32u mask1Dlen;
    Ncv32u mask2Dstride;
    NcvSize32u anchorsRoi;
    Ncv32u startStageInc;
    Ncv32u endStageExc;
    Ncv32f scaleArea;

    //Arguments are passed through the constructor
    applyHaarClassifierAnchorParallelFunctor(dim3 _gridConf, dim3 _blockConf, hipStream_t _cuStream,
                                             Ncv32u *_d_IImg, Ncv32u _IImgStride,
                                             Ncv32f *_d_weights, Ncv32u _weightsStride,
                                             HaarFeature64 *_d_Features, HaarClassifierNode128 *_d_ClassifierNodes, HaarStage64 *_d_Stages,
                                             Ncv32u *_d_inMask, Ncv32u *_d_outMask,
                                             Ncv32u _mask1Dlen, Ncv32u _mask2Dstride,
                                             NcvSize32u _anchorsRoi, Ncv32u _startStageInc,
                                             Ncv32u _endStageExc, Ncv32f _scaleArea) :
    gridConf(_gridConf),
    blockConf(_blockConf),
    cuStream(_cuStream),
    d_IImg(_d_IImg),
    IImgStride(_IImgStride),
    d_weights(_d_weights),
    weightsStride(_weightsStride),
    d_Features(_d_Features),
    d_ClassifierNodes(_d_ClassifierNodes),
    d_Stages(_d_Stages),
    d_inMask(_d_inMask),
    d_outMask(_d_outMask),
    mask1Dlen(_mask1Dlen),
    mask2Dstride(_mask2Dstride),
    anchorsRoi(_anchorsRoi),
    startStageInc(_startStageInc),
    endStageExc(_endStageExc),
    scaleArea(_scaleArea)
    {}

    template<class TList>
    void call(TList tl)
    {
        applyHaarClassifierAnchorParallel <
            Loki::TL::TypeAt<TList, 0>::Result::value,
            Loki::TL::TypeAt<TList, 1>::Result::value,
            Loki::TL::TypeAt<TList, 2>::Result::value,
            Loki::TL::TypeAt<TList, 3>::Result::value,
            Loki::TL::TypeAt<TList, 4>::Result::value >
            <<<gridConf, blockConf, 0, cuStream>>>
            (d_IImg, IImgStride,
            d_weights, weightsStride,
            d_Features, d_ClassifierNodes, d_Stages,
            d_inMask, d_outMask,
            mask1Dlen, mask2Dstride,
            anchorsRoi, startStageInc,
            endStageExc, scaleArea);
    }
};


void applyHaarClassifierAnchorParallelDynTemplate(NcvBool tbInitMaskPositively,
                                                  NcvBool tbCacheTextureIImg,
                                                  NcvBool tbCacheTextureCascade,
                                                  NcvBool tbReadPixelIndexFromVector,
                                                  NcvBool tbDoAtomicCompaction,

                                                  dim3 gridConf, dim3 blockConf, hipStream_t cuStream,

                                                  Ncv32u *d_IImg, Ncv32u IImgStride,
                                                  Ncv32f *d_weights, Ncv32u weightsStride,
                                                  HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages,
                                                  Ncv32u *d_inMask, Ncv32u *d_outMask,
                                                  Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                                  NcvSize32u anchorsRoi, Ncv32u startStageInc,
                                                  Ncv32u endStageExc, Ncv32f scaleArea)
{

    applyHaarClassifierAnchorParallelFunctor functor(gridConf, blockConf, cuStream,
                                                     d_IImg, IImgStride,
                                                     d_weights, weightsStride,
                                                     d_Features, d_ClassifierNodes, d_Stages,
                                                     d_inMask, d_outMask,
                                                     mask1Dlen, mask2Dstride,
                                                     anchorsRoi, startStageInc,
                                                     endStageExc, scaleArea);

    //Second parameter is the number of "dynamic" template parameters
    NCVRuntimeTemplateBool::KernelCaller<Loki::NullType, 5, applyHaarClassifierAnchorParallelFunctor>
        ::call( &functor,
                tbInitMaskPositively,
                tbCacheTextureIImg,
                tbCacheTextureCascade,
                tbReadPixelIndexFromVector,
                tbDoAtomicCompaction);
}


struct applyHaarClassifierClassifierParallelFunctor
{
    dim3 gridConf, blockConf;
    hipStream_t cuStream;

    //Kernel arguments are stored as members;
    Ncv32u *d_IImg;
    Ncv32u IImgStride;
    Ncv32f *d_weights;
    Ncv32u weightsStride;
    HaarFeature64 *d_Features;
    HaarClassifierNode128 *d_ClassifierNodes;
    HaarStage64 *d_Stages;
    Ncv32u *d_inMask;
    Ncv32u *d_outMask;
    Ncv32u mask1Dlen;
    Ncv32u mask2Dstride;
    NcvSize32u anchorsRoi;
    Ncv32u startStageInc;
    Ncv32u endStageExc;
    Ncv32f scaleArea;

    //Arguments are passed through the constructor
    applyHaarClassifierClassifierParallelFunctor(dim3 _gridConf, dim3 _blockConf, hipStream_t _cuStream,
                                                 Ncv32u *_d_IImg, Ncv32u _IImgStride,
                                                 Ncv32f *_d_weights, Ncv32u _weightsStride,
                                                 HaarFeature64 *_d_Features, HaarClassifierNode128 *_d_ClassifierNodes, HaarStage64 *_d_Stages,
                                                 Ncv32u *_d_inMask, Ncv32u *_d_outMask,
                                                 Ncv32u _mask1Dlen, Ncv32u _mask2Dstride,
                                                 NcvSize32u _anchorsRoi, Ncv32u _startStageInc,
                                                 Ncv32u _endStageExc, Ncv32f _scaleArea) :
    gridConf(_gridConf),
    blockConf(_blockConf),
    cuStream(_cuStream),
    d_IImg(_d_IImg),
    IImgStride(_IImgStride),
    d_weights(_d_weights),
    weightsStride(_weightsStride),
    d_Features(_d_Features),
    d_ClassifierNodes(_d_ClassifierNodes),
    d_Stages(_d_Stages),
    d_inMask(_d_inMask),
    d_outMask(_d_outMask),
    mask1Dlen(_mask1Dlen),
    mask2Dstride(_mask2Dstride),
    anchorsRoi(_anchorsRoi),
    startStageInc(_startStageInc),
    endStageExc(_endStageExc),
    scaleArea(_scaleArea)
    {}

    template<class TList>
    void call(TList tl)
    {
        applyHaarClassifierClassifierParallel <
            Loki::TL::TypeAt<TList, 0>::Result::value,
            Loki::TL::TypeAt<TList, 1>::Result::value,
            Loki::TL::TypeAt<TList, 2>::Result::value >
            <<<gridConf, blockConf, 0, cuStream>>>
            (d_IImg, IImgStride,
            d_weights, weightsStride,
            d_Features, d_ClassifierNodes, d_Stages,
            d_inMask, d_outMask,
            mask1Dlen, mask2Dstride,
            anchorsRoi, startStageInc,
            endStageExc, scaleArea);
    }
};


void applyHaarClassifierClassifierParallelDynTemplate(NcvBool tbCacheTextureIImg,
                                                      NcvBool tbCacheTextureCascade,
                                                      NcvBool tbDoAtomicCompaction,

                                                      dim3 gridConf, dim3 blockConf, hipStream_t cuStream,

                                                      Ncv32u *d_IImg, Ncv32u IImgStride,
                                                      Ncv32f *d_weights, Ncv32u weightsStride,
                                                      HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages,
                                                      Ncv32u *d_inMask, Ncv32u *d_outMask,
                                                      Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                                      NcvSize32u anchorsRoi, Ncv32u startStageInc,
                                                      Ncv32u endStageExc, Ncv32f scaleArea)
{
    applyHaarClassifierClassifierParallelFunctor functor(gridConf, blockConf, cuStream,
                                                         d_IImg, IImgStride,
                                                         d_weights, weightsStride,
                                                         d_Features, d_ClassifierNodes, d_Stages,
                                                         d_inMask, d_outMask,
                                                         mask1Dlen, mask2Dstride,
                                                         anchorsRoi, startStageInc,
                                                         endStageExc, scaleArea);

    //Second parameter is the number of "dynamic" template parameters
    NCVRuntimeTemplateBool::KernelCaller<Loki::NullType, 3, applyHaarClassifierClassifierParallelFunctor>
        ::call( &functor,
                tbCacheTextureIImg,
                tbCacheTextureCascade,
                tbDoAtomicCompaction);
}


struct initializeMaskVectorFunctor
{
    dim3 gridConf, blockConf;
    hipStream_t cuStream;

    //Kernel arguments are stored as members;
    Ncv32u *d_inMask;
    Ncv32u *d_outMask;
    Ncv32u mask1Dlen;
    Ncv32u mask2Dstride;
    NcvSize32u anchorsRoi;
    Ncv32u step;

    //Arguments are passed through the constructor
    initializeMaskVectorFunctor(dim3 _gridConf, dim3 _blockConf, hipStream_t _cuStream,
                                Ncv32u *_d_inMask, Ncv32u *_d_outMask,
                                Ncv32u _mask1Dlen, Ncv32u _mask2Dstride,
                                NcvSize32u _anchorsRoi, Ncv32u _step) :
    gridConf(_gridConf),
    blockConf(_blockConf),
    cuStream(_cuStream),
    d_inMask(_d_inMask),
    d_outMask(_d_outMask),
    mask1Dlen(_mask1Dlen),
    mask2Dstride(_mask2Dstride),
    anchorsRoi(_anchorsRoi),
    step(_step)
    {}

    template<class TList>
    void call(TList tl)
    {
        initializeMaskVector <
            Loki::TL::TypeAt<TList, 0>::Result::value,
            Loki::TL::TypeAt<TList, 1>::Result::value >
            <<<gridConf, blockConf, 0, cuStream>>>
            (d_inMask, d_outMask,
             mask1Dlen, mask2Dstride,
             anchorsRoi, step);
    }
};


void initializeMaskVectorDynTemplate(NcvBool tbMaskByInmask,
                                     NcvBool tbDoAtomicCompaction,

                                     dim3 gridConf, dim3 blockConf, hipStream_t cuStream,

                                     Ncv32u *d_inMask, Ncv32u *d_outMask,
                                     Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                     NcvSize32u anchorsRoi, Ncv32u step)
{
    initializeMaskVectorFunctor functor(gridConf, blockConf, cuStream,
                                        d_inMask, d_outMask,
                                        mask1Dlen, mask2Dstride,
                                        anchorsRoi, step);

    //Second parameter is the number of "dynamic" template parameters
    NCVRuntimeTemplateBool::KernelCaller<Loki::NullType, 2, initializeMaskVectorFunctor>
        ::call( &functor,
                tbMaskByInmask,
                tbDoAtomicCompaction);
}


Ncv32u getStageNumWithNotLessThanNclassifiers(Ncv32u N, HaarClassifierCascadeDescriptor &haar,
                                              NCVVector<HaarStage64> &h_HaarStages)
{
    for (Ncv32u i = 0; i<haar.NumStages; i++)
    {
        if (h_HaarStages.ptr()[i].getNumClassifierRootNodes() >= N)
        {
            return i;
        }
    }
    return haar.NumStages;
}


NCVStatus ncvApplyHaarClassifierCascade_device(NCVMatrix<Ncv32u> &d_integralImage,
                                               NCVMatrix<Ncv32f> &d_weights,
                                               NCVMatrixAlloc<Ncv32u> &d_pixelMask,
                                               Ncv32u &numDetections,
                                               HaarClassifierCascadeDescriptor &haar,
                                               NCVVector<HaarStage64> &h_HaarStages,
                                               NCVVector<HaarStage64> &d_HaarStages,
                                               NCVVector<HaarClassifierNode128> &d_HaarNodes,
                                               NCVVector<HaarFeature64> &d_HaarFeatures,
                                               NcvBool bMaskElements,
                                               NcvSize32u anchorsRoi,
                                               Ncv32u pixelStep,
                                               Ncv32f scaleArea,
                                               INCVMemAllocator &gpuAllocator,
                                               INCVMemAllocator &cpuAllocator,
                                               hipDeviceProp_t &devProp,
                                               hipStream_t cuStream)
{
    ncvAssertReturn(d_integralImage.memType() == d_weights.memType() &&
                    d_integralImage.memType() == d_pixelMask.memType() &&
                    d_integralImage.memType() == gpuAllocator.memType() &&
                     (d_integralImage.memType() == NCVMemoryTypeDevice ||
                      d_integralImage.memType() == NCVMemoryTypeNone), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(d_HaarStages.memType() == d_HaarNodes.memType() &&
                    d_HaarStages.memType() == d_HaarFeatures.memType() &&
                     (d_HaarStages.memType() == NCVMemoryTypeDevice ||
                      d_HaarStages.memType() == NCVMemoryTypeNone), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(h_HaarStages.memType() != NCVMemoryTypeDevice, NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(gpuAllocator.isInitialized() && cpuAllocator.isInitialized(), NCV_ALLOCATOR_NOT_INITIALIZED);
    ncvAssertReturn((d_integralImage.ptr() != NULL && d_weights.ptr() != NULL && d_pixelMask.ptr() != NULL &&
                     h_HaarStages.ptr() != NULL && d_HaarStages.ptr() != NULL && d_HaarNodes.ptr() != NULL &&
                     d_HaarFeatures.ptr() != NULL) || gpuAllocator.isCounting(), NCV_NULL_PTR);
    ncvAssertReturn(anchorsRoi.width > 0 && anchorsRoi.height > 0 &&
                    d_pixelMask.width() >= anchorsRoi.width && d_pixelMask.height() >= anchorsRoi.height &&
                    d_weights.width() >= anchorsRoi.width && d_weights.height() >= anchorsRoi.height &&
                    d_integralImage.width() >= anchorsRoi.width + haar.ClassifierSize.width &&
                    d_integralImage.height() >= anchorsRoi.height + haar.ClassifierSize.height, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(scaleArea > 0, NCV_INVALID_SCALE);
    ncvAssertReturn(d_HaarStages.length() >= haar.NumStages &&
                    d_HaarNodes.length() >= haar.NumClassifierTotalNodes &&
                    d_HaarFeatures.length() >= haar.NumFeatures &&
                    d_HaarStages.length() == h_HaarStages.length() &&
                    haar.NumClassifierRootNodes <= haar.NumClassifierTotalNodes, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(haar.bNeedsTiltedII == false || gpuAllocator.isCounting(), NCV_NOIMPL_HAAR_TILTED_FEATURES);
    ncvAssertReturn(pixelStep == 1 || pixelStep == 2, NCV_HAAR_INVALID_PIXEL_STEP);

    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

#if defined _SELF_TEST_

    NCVStatus ncvStat;

    NCVMatrixAlloc<Ncv32u> h_integralImage(cpuAllocator, d_integralImage.width, d_integralImage.height, d_integralImage.pitch);
    ncvAssertReturn(h_integralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv32f> h_weights(cpuAllocator, d_weights.width, d_weights.height, d_weights.pitch);
    ncvAssertReturn(h_weights.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv32u> h_pixelMask(cpuAllocator, d_pixelMask.width, d_pixelMask.height, d_pixelMask.pitch);
    ncvAssertReturn(h_pixelMask.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVVectorAlloc<HaarClassifierNode128> h_HaarNodes(cpuAllocator, d_HaarNodes.length);
    ncvAssertReturn(h_HaarNodes.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVVectorAlloc<HaarFeature64> h_HaarFeatures(cpuAllocator, d_HaarFeatures.length);
    ncvAssertReturn(h_HaarFeatures.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVMatrixAlloc<Ncv32u> h_pixelMask_d(cpuAllocator, d_pixelMask.width, d_pixelMask.height, d_pixelMask.pitch);
    ncvAssertReturn(h_pixelMask_d.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCV_SKIP_COND_BEGIN

    ncvStat = d_pixelMask.copySolid(h_pixelMask, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvStat = d_integralImage.copySolid(h_integralImage, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvStat = d_weights.copySolid(h_weights, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvStat = d_HaarNodes.copySolid(h_HaarNodes, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvStat = d_HaarFeatures.copySolid(h_HaarFeatures, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvAssertCUDAReturn(hipStreamSynchronize(0), NCV_CUDA_ERROR);

    for (Ncv32u i=0; i<(Ncv32u)anchorsRoi.height; i++)
    {
        for (Ncv32u j=0; j<d_pixelMask.stride(); j++)
        {
            if ((i%pixelStep==0) && (j%pixelStep==0) && (j<(Ncv32u)anchorsRoi.width))
            {
                if (!bMaskElements || h_pixelMask.ptr[i*d_pixelMask.stride()+j] != OBJDET_MASK_ELEMENT_INVALID_32U)
                {
                    h_pixelMask.ptr[i*d_pixelMask.stride()+j] = (i << 16) | j;
                }
            }
            else
            {
                h_pixelMask.ptr[i*d_pixelMask.stride()+j] = OBJDET_MASK_ELEMENT_INVALID_32U;
            }
        }
    }

    NCV_SKIP_COND_END

#endif

    NCVVectorReuse<Ncv32u> d_vecPixelMask(d_pixelMask.getSegment(), anchorsRoi.height * d_pixelMask.stride());
    ncvAssertReturn(d_vecPixelMask.isMemReused(), NCV_ALLOCATOR_BAD_REUSE);

    NCVVectorAlloc<Ncv32u> d_vecPixelMaskTmp(gpuAllocator, static_cast<Ncv32u>(d_vecPixelMask.length()));
    ncvAssertReturn(d_vecPixelMaskTmp.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVVectorAlloc<Ncv32u> hp_pool32u(cpuAllocator, 2);
    ncvAssertReturn(hp_pool32u.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    Ncv32u *hp_zero = &hp_pool32u.ptr()[0];
    Ncv32u *hp_numDet = &hp_pool32u.ptr()[1];

    NCV_SKIP_COND_BEGIN
    *hp_zero = 0;
    *hp_numDet = 0;
    NCV_SKIP_COND_END

    Ncv32f scaleAreaPixels = scaleArea * ((haar.ClassifierSize.width - 2*HAAR_STDDEV_BORDER) *
                                          (haar.ClassifierSize.height - 2*HAAR_STDDEV_BORDER));

    NcvBool bTexCacheCascade = devProp.major < 2;
    NcvBool bTexCacheIImg = true; //this works better even on Fermi so far
    NcvBool bDoAtomicCompaction = devProp.major >= 2 || (devProp.major == 1 && devProp.minor >= 3);

    NCVVector<Ncv32u> *d_ptrNowData = &d_vecPixelMask;
    NCVVector<Ncv32u> *d_ptrNowTmp = &d_vecPixelMaskTmp;

    Ncv32u szNppCompactTmpBuf;
    nppsStCompactGetSize_32u(static_cast<Ncv32u>(d_vecPixelMask.length()), &szNppCompactTmpBuf, devProp);
    if (bDoAtomicCompaction)
    {
        szNppCompactTmpBuf = 0;
    }
    NCVVectorAlloc<Ncv8u> d_tmpBufCompact(gpuAllocator, szNppCompactTmpBuf);

    NCV_SKIP_COND_BEGIN

    if (bTexCacheIImg)
    {
        hipChannelFormatDesc cfdTexIImage;
        cfdTexIImage = hipCreateChannelDesc<Ncv32u>();

        size_t alignmentOffset;
        ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, texIImage, d_integralImage.ptr(), cfdTexIImage,
            (anchorsRoi.height + haar.ClassifierSize.height) * d_integralImage.pitch()), NCV_CUDA_ERROR);
        ncvAssertReturn(alignmentOffset==0, NCV_TEXTURE_BIND_ERROR);
    }

    if (bTexCacheCascade)
    {
        hipChannelFormatDesc cfdTexHaarFeatures;
        hipChannelFormatDesc cfdTexHaarClassifierNodes;
        cfdTexHaarFeatures = hipCreateChannelDesc<uint2>();
        cfdTexHaarClassifierNodes = hipCreateChannelDesc<uint4>();

        size_t alignmentOffset;
        ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, texHaarFeatures,
            d_HaarFeatures.ptr(), cfdTexHaarFeatures,sizeof(HaarFeature64) * haar.NumFeatures), NCV_CUDA_ERROR);
        ncvAssertReturn(alignmentOffset==0, NCV_TEXTURE_BIND_ERROR);
        ncvAssertCUDAReturn(hipBindTexture(&alignmentOffset, texHaarClassifierNodes,
            d_HaarNodes.ptr(), cfdTexHaarClassifierNodes, sizeof(HaarClassifierNode128) * haar.NumClassifierTotalNodes), NCV_CUDA_ERROR);
        ncvAssertReturn(alignmentOffset==0, NCV_TEXTURE_BIND_ERROR);
    }

    Ncv32u stageStartAnchorParallel = 0;
    Ncv32u stageMiddleSwitch = getStageNumWithNotLessThanNclassifiers(NUM_THREADS_CLASSIFIERPARALLEL,
        haar, h_HaarStages);
    Ncv32u stageEndClassifierParallel = haar.NumStages;
    if (stageMiddleSwitch == 0)
    {
        stageMiddleSwitch = 1;
    }

    //create stages subdivision for pixel-parallel processing
    const Ncv32u compactEveryNstage = bDoAtomicCompaction ? 7 : 1;
    Ncv32u curStop = stageStartAnchorParallel;
    std::vector<Ncv32u> pixParallelStageStops;
    while (curStop < stageMiddleSwitch)
    {
        pixParallelStageStops.push_back(curStop);
        curStop += compactEveryNstage;
    }
    if (curStop > compactEveryNstage && curStop - stageMiddleSwitch > compactEveryNstage / 2)
    {
        pixParallelStageStops[pixParallelStageStops.size()-1] =
            (stageMiddleSwitch - (curStop - 2 * compactEveryNstage)) / 2;
    }
    pixParallelStageStops.push_back(stageMiddleSwitch);
    Ncv32u pixParallelStageStopsIndex = 0;

    if (pixelStep != 1 || bMaskElements)
    {
        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_outMaskPosition), hp_zero, sizeof(Ncv32u),
                                                        0, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        dim3 gridInit((((anchorsRoi.width + pixelStep - 1) / pixelStep + NUM_THREADS_ANCHORSPARALLEL - 1) / NUM_THREADS_ANCHORSPARALLEL),
                        (anchorsRoi.height + pixelStep - 1) / pixelStep);
        dim3 blockInit(NUM_THREADS_ANCHORSPARALLEL);

        if (gridInit.x == 0 || gridInit.y == 0)
        {
            numDetections = 0;
            return NCV_SUCCESS;
        }

        initializeMaskVectorDynTemplate(bMaskElements,
                                        bDoAtomicCompaction,
                                        gridInit, blockInit, cuStream,
                                        d_ptrNowData->ptr(),
                                        d_ptrNowTmp->ptr(),
                                        static_cast<Ncv32u>(d_vecPixelMask.length()), d_pixelMask.stride(),
                                        anchorsRoi, pixelStep);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemcpyFromSymbolAsync(hp_numDet, HIP_SYMBOL(d_outMaskPosition), sizeof(Ncv32u),
                                                          0, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            swap(d_ptrNowData, d_ptrNowTmp);
        }
        else
        {
            NCVStatus nppSt;
            nppSt = nppsStCompact_32u(d_ptrNowTmp->ptr(), static_cast<Ncv32u>(d_vecPixelMask.length()),
                                      d_ptrNowData->ptr(), hp_numDet, OBJDET_MASK_ELEMENT_INVALID_32U,
                                      d_tmpBufCompact.ptr(), szNppCompactTmpBuf, devProp);
            ncvAssertReturn(nppSt == NPPST_SUCCESS, NCV_NPP_ERROR);
        }
        numDetections = *hp_numDet;
    }
    else
    {
        //
        // 1. Run the first pixel-input pixel-parallel classifier for few stages
        //

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_outMaskPosition), hp_zero, sizeof(Ncv32u),
                                                        0, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        dim3 grid1(((d_pixelMask.stride() + NUM_THREADS_ANCHORSPARALLEL - 1) / NUM_THREADS_ANCHORSPARALLEL),
                   anchorsRoi.height);
        dim3 block1(NUM_THREADS_ANCHORSPARALLEL);
        applyHaarClassifierAnchorParallelDynTemplate(
            true,                         //tbInitMaskPositively
            bTexCacheIImg,                //tbCacheTextureIImg
            bTexCacheCascade,             //tbCacheTextureCascade
            pixParallelStageStops[pixParallelStageStopsIndex] != 0,//tbReadPixelIndexFromVector
            bDoAtomicCompaction,          //tbDoAtomicCompaction
            grid1,
            block1,
            cuStream,
            d_integralImage.ptr(), d_integralImage.stride(),
            d_weights.ptr(), d_weights.stride(),
            d_HaarFeatures.ptr(), d_HaarNodes.ptr(), d_HaarStages.ptr(),
            d_ptrNowData->ptr(),
            bDoAtomicCompaction ? d_ptrNowTmp->ptr() : d_ptrNowData->ptr(),
            0,
            d_pixelMask.stride(),
            anchorsRoi,
            pixParallelStageStops[pixParallelStageStopsIndex],
            pixParallelStageStops[pixParallelStageStopsIndex+1],
            scaleAreaPixels);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemcpyFromSymbolAsync(hp_numDet, HIP_SYMBOL(d_outMaskPosition), sizeof(Ncv32u),
                                                          0, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }
        else
        {
            NCVStatus nppSt;
            nppSt = nppsStCompact_32u(d_ptrNowData->ptr(), static_cast<Ncv32u>(d_vecPixelMask.length()),
                                      d_ptrNowTmp->ptr(), hp_numDet, OBJDET_MASK_ELEMENT_INVALID_32U,
                                      d_tmpBufCompact.ptr(), szNppCompactTmpBuf, devProp);
            ncvAssertReturnNcvStat(nppSt);
        }

        swap(d_ptrNowData, d_ptrNowTmp);
        numDetections = *hp_numDet;

        pixParallelStageStopsIndex++;
    }

    //
    // 2. Run pixel-parallel stages
    //

    for (; pixParallelStageStopsIndex < pixParallelStageStops.size()-1; pixParallelStageStopsIndex++)
    {
        if (numDetections == 0)
        {
            break;
        }

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_outMaskPosition), hp_zero, sizeof(Ncv32u),
                                                        0, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        dim3 grid2((numDetections + NUM_THREADS_ANCHORSPARALLEL - 1) / NUM_THREADS_ANCHORSPARALLEL);
        if (numDetections > MAX_GRID_DIM)
        {
            grid2.x = MAX_GRID_DIM;
            grid2.y = (numDetections + MAX_GRID_DIM - 1) / MAX_GRID_DIM;
        }
        dim3 block2(NUM_THREADS_ANCHORSPARALLEL);

        applyHaarClassifierAnchorParallelDynTemplate(
            false,                        //tbInitMaskPositively
            bTexCacheIImg,                //tbCacheTextureIImg
            bTexCacheCascade,             //tbCacheTextureCascade
            pixParallelStageStops[pixParallelStageStopsIndex] != 0 || pixelStep != 1 || bMaskElements,//tbReadPixelIndexFromVector
            bDoAtomicCompaction,          //tbDoAtomicCompaction
            grid2,
            block2,
            cuStream,
            d_integralImage.ptr(), d_integralImage.stride(),
            d_weights.ptr(), d_weights.stride(),
            d_HaarFeatures.ptr(), d_HaarNodes.ptr(), d_HaarStages.ptr(),
            d_ptrNowData->ptr(),
            bDoAtomicCompaction ? d_ptrNowTmp->ptr() : d_ptrNowData->ptr(),
            numDetections,
            d_pixelMask.stride(),
            anchorsRoi,
            pixParallelStageStops[pixParallelStageStopsIndex],
            pixParallelStageStops[pixParallelStageStopsIndex+1],
            scaleAreaPixels);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemcpyFromSymbolAsync(hp_numDet, HIP_SYMBOL(d_outMaskPosition), sizeof(Ncv32u),
                                                          0, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }
        else
        {
            NCVStatus nppSt;
            nppSt = nppsStCompact_32u(d_ptrNowData->ptr(), numDetections,
                                      d_ptrNowTmp->ptr(), hp_numDet, OBJDET_MASK_ELEMENT_INVALID_32U,
                                      d_tmpBufCompact.ptr(), szNppCompactTmpBuf, devProp);
            ncvAssertReturnNcvStat(nppSt);
        }

        swap(d_ptrNowData, d_ptrNowTmp);
        numDetections = *hp_numDet;
    }

    //
    // 3. Run all left stages in one stage-parallel kernel
    //

    if (numDetections > 0 && stageMiddleSwitch < stageEndClassifierParallel)
    {
        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_outMaskPosition), hp_zero, sizeof(Ncv32u),
                                                        0, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        dim3 grid3(numDetections);
        if (numDetections > MAX_GRID_DIM)
        {
            grid3.x = MAX_GRID_DIM;
            grid3.y = (numDetections + MAX_GRID_DIM - 1) / MAX_GRID_DIM;
        }
        dim3 block3(NUM_THREADS_CLASSIFIERPARALLEL);

        applyHaarClassifierClassifierParallelDynTemplate(
            bTexCacheIImg,                //tbCacheTextureIImg
            bTexCacheCascade,             //tbCacheTextureCascade
            bDoAtomicCompaction,          //tbDoAtomicCompaction
            grid3,
            block3,
            cuStream,
            d_integralImage.ptr(), d_integralImage.stride(),
            d_weights.ptr(), d_weights.stride(),
            d_HaarFeatures.ptr(), d_HaarNodes.ptr(), d_HaarStages.ptr(),
            d_ptrNowData->ptr(),
            bDoAtomicCompaction ? d_ptrNowTmp->ptr() : d_ptrNowData->ptr(),
            numDetections,
            d_pixelMask.stride(),
            anchorsRoi,
            stageMiddleSwitch,
            stageEndClassifierParallel,
            scaleAreaPixels);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemcpyFromSymbolAsync(hp_numDet, HIP_SYMBOL(d_outMaskPosition), sizeof(Ncv32u),
                                                          0, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }
        else
        {
            NCVStatus nppSt;
            nppSt = nppsStCompact_32u(d_ptrNowData->ptr(), numDetections,
                                      d_ptrNowTmp->ptr(), hp_numDet, OBJDET_MASK_ELEMENT_INVALID_32U,
                                      d_tmpBufCompact.ptr(), szNppCompactTmpBuf, devProp);
            ncvAssertReturnNcvStat(nppSt);
        }

        swap(d_ptrNowData, d_ptrNowTmp);
        numDetections = *hp_numDet;
    }

    if (d_ptrNowData != &d_vecPixelMask)
    {
        d_vecPixelMaskTmp.copySolid(d_vecPixelMask, cuStream);
        ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
    }

#if defined _SELF_TEST_

    ncvStat = d_pixelMask.copySolid(h_pixelMask_d, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);

    if (bDoAtomicCompaction)
    {
        std::sort(h_pixelMask_d.ptr, h_pixelMask_d.ptr + numDetections);
    }

    Ncv32u fpu_oldcw, fpu_cw;
    _controlfp_s(&fpu_cw, 0, 0);
    fpu_oldcw = fpu_cw;
    _controlfp_s(&fpu_cw, _PC_24, _MCW_PC);
    Ncv32u numDetGold;
    ncvStat = ncvApplyHaarClassifierCascade_host(h_integralImage, h_weights, h_pixelMask, numDetGold, haar,
                                                 h_HaarStages, h_HaarNodes, h_HaarFeatures,
                                                 bMaskElements, anchorsRoi, pixelStep, scaleArea);
    ncvAssertReturnNcvStat(ncvStat);
    _controlfp_s(&fpu_cw, fpu_oldcw, _MCW_PC);

    bool bPass = true;

    if (numDetGold != numDetections)
    {
        printf("NCVHaarClassifierCascade::applyHaarClassifierCascade numdetections don't match: cpu=%d, gpu=%d\n", numDetGold, numDetections);
        bPass = false;
    }
    else
    {
        for (Ncv32u i=0; i<std::max(numDetGold, numDetections) && bPass; i++)
        {
            if (h_pixelMask.ptr[i] != h_pixelMask_d.ptr[i])
            {
                printf("NCVHaarClassifierCascade::applyHaarClassifierCascade self test failed: i=%d, cpu=%d, gpu=%d\n", i, h_pixelMask.ptr[i], h_pixelMask_d.ptr[i]);
                bPass = false;
            }
        }
    }

    printf("NCVHaarClassifierCascade::applyHaarClassifierCascade %s\n", bPass?"PASSED":"FAILED");
#endif

    NCV_SKIP_COND_END

    return NCV_SUCCESS;
}


//==============================================================================
//
// HypothesesOperations file
//
//==============================================================================


const Ncv32u NUM_GROW_THREADS = 128;


__device__ __host__ NcvRect32u pixelToRect(Ncv32u pixel, Ncv32u width, Ncv32u height, Ncv32f scale)
{
    NcvRect32u res;
    res.x = (Ncv32u)(scale * (pixel & 0xFFFF));
    res.y = (Ncv32u)(scale * (pixel >> 16));
    res.width = (Ncv32u)(scale * width);
    res.height = (Ncv32u)(scale * height);
    return res;
}


__global__ void growDetectionsKernel(Ncv32u *pixelMask, Ncv32u numElements,
                                     NcvRect32u *hypotheses,
                                     Ncv32u rectWidth, Ncv32u rectHeight, Ncv32f curScale)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    Ncv32u elemAddr = blockId * NUM_GROW_THREADS + threadIdx.x;
    if (elemAddr >= numElements)
    {
        return;
    }
    hypotheses[elemAddr] = pixelToRect(pixelMask[elemAddr], rectWidth, rectHeight, curScale);
}


NCVStatus ncvGrowDetectionsVector_device(NCVVector<Ncv32u> &pixelMask,
                                         Ncv32u numPixelMaskDetections,
                                         NCVVector<NcvRect32u> &hypotheses,
                                         Ncv32u &totalDetections,
                                         Ncv32u totalMaxDetections,
                                         Ncv32u rectWidth,
                                         Ncv32u rectHeight,
                                         Ncv32f curScale,
                                         hipStream_t cuStream)
{
    ncvAssertReturn(pixelMask.ptr() != NULL && hypotheses.ptr() != NULL, NCV_NULL_PTR);
    ncvAssertReturn(pixelMask.memType() == hypotheses.memType() &&
                    pixelMask.memType() == NCVMemoryTypeDevice, NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(rectWidth > 0 && rectHeight > 0 && curScale > 0, NCV_INVALID_ROI);
    ncvAssertReturn(curScale > 0, NCV_INVALID_SCALE);
    ncvAssertReturn(totalMaxDetections <= hypotheses.length() &&
                    numPixelMaskDetections <= pixelMask.length(), NCV_INCONSISTENT_INPUT);

    NCVStatus ncvStat = NCV_SUCCESS;
    Ncv32u numDetsToCopy = numPixelMaskDetections;

    if (numDetsToCopy == 0)
    {
        return ncvStat;
    }

    if (totalDetections + numPixelMaskDetections > totalMaxDetections)
    {
        ncvStat = NCV_WARNING_HAAR_DETECTIONS_VECTOR_OVERFLOW;
        numDetsToCopy = totalMaxDetections - totalDetections;
    }

    dim3 block(NUM_GROW_THREADS);
    dim3 grid((numDetsToCopy + NUM_GROW_THREADS - 1) / NUM_GROW_THREADS);
    if (grid.x > 65535)
    {
        grid.y = (grid.x + 65534) / 65535;
        grid.x = 65535;
    }
    growDetectionsKernel<<<grid, block, 0, cuStream>>>(pixelMask.ptr(), numDetsToCopy,
                                                       hypotheses.ptr() + totalDetections,
                                                       rectWidth, rectHeight, curScale);
    ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

    totalDetections += numDetsToCopy;
    return ncvStat;
}


//==============================================================================
//
// Pipeline file
//
//==============================================================================


NCVStatus ncvDetectObjectsMultiScale_device(NCVMatrix<Ncv8u> &d_srcImg,
                                            NcvSize32u srcRoi,
                                            NCVVector<NcvRect32u> &d_dstRects,
                                            Ncv32u &dstNumRects,

                                            HaarClassifierCascadeDescriptor &haar,
                                            NCVVector<HaarStage64> &h_HaarStages,
                                            NCVVector<HaarStage64> &d_HaarStages,
                                            NCVVector<HaarClassifierNode128> &d_HaarNodes,
                                            NCVVector<HaarFeature64> &d_HaarFeatures,

                                            NcvSize32u minObjSize,
                                            Ncv32u minNeighbors,      //default 4
                                            Ncv32f scaleStep,         //default 1.2f
                                            Ncv32u pixelStep,         //default 1
                                            Ncv32u flags,             //default NCVPipeObjDet_Default

                                            INCVMemAllocator &gpuAllocator,
                                            INCVMemAllocator &cpuAllocator,
                                            hipDeviceProp_t &devProp,
                                            hipStream_t cuStream)
{
    ncvAssertReturn(d_srcImg.memType() == d_dstRects.memType() &&
                    d_srcImg.memType() == gpuAllocator.memType() &&
                     (d_srcImg.memType() == NCVMemoryTypeDevice ||
                      d_srcImg.memType() == NCVMemoryTypeNone), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(d_HaarStages.memType() == d_HaarNodes.memType() &&
                    d_HaarStages.memType() == d_HaarFeatures.memType() &&
                     (d_HaarStages.memType() == NCVMemoryTypeDevice ||
                      d_HaarStages.memType() == NCVMemoryTypeNone), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(h_HaarStages.memType() != NCVMemoryTypeDevice, NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(gpuAllocator.isInitialized() && cpuAllocator.isInitialized(), NCV_ALLOCATOR_NOT_INITIALIZED);
    ncvAssertReturn((d_srcImg.ptr() != NULL && d_dstRects.ptr() != NULL &&
                     h_HaarStages.ptr() != NULL && d_HaarStages.ptr() != NULL && d_HaarNodes.ptr() != NULL &&
                     d_HaarFeatures.ptr() != NULL) || gpuAllocator.isCounting(), NCV_NULL_PTR);
    ncvAssertReturn(srcRoi.width > 0 && srcRoi.height > 0 &&
                    d_srcImg.width() >= srcRoi.width && d_srcImg.height() >= srcRoi.height &&
                    srcRoi.width >= minObjSize.width && srcRoi.height >= minObjSize.height &&
                    d_dstRects.length() >= 1, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(scaleStep > 1.0f, NCV_INVALID_SCALE);
    ncvAssertReturn(d_HaarStages.length() >= haar.NumStages &&
                    d_HaarNodes.length() >= haar.NumClassifierTotalNodes &&
                    d_HaarFeatures.length() >= haar.NumFeatures &&
                    d_HaarStages.length() == h_HaarStages.length() &&
                    haar.NumClassifierRootNodes <= haar.NumClassifierTotalNodes, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(haar.bNeedsTiltedII == false, NCV_NOIMPL_HAAR_TILTED_FEATURES);
    ncvAssertReturn(pixelStep == 1 || pixelStep == 2, NCV_HAAR_INVALID_PIXEL_STEP);

    //TODO: set NPP active stream to cuStream

    NCVStatus ncvStat;
    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

    Ncv32u integralWidth = d_srcImg.width() + 1;
    Ncv32u integralHeight = d_srcImg.height() + 1;

    NCVMatrixAlloc<Ncv32u> d_integralImage(gpuAllocator, integralWidth, integralHeight);
    ncvAssertReturn(d_integralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv64u> d_sqIntegralImage(gpuAllocator, integralWidth, integralHeight);
    ncvAssertReturn(d_sqIntegralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVMatrixAlloc<Ncv32f> d_rectStdDev(gpuAllocator, d_srcImg.width(), d_srcImg.height());
    ncvAssertReturn(d_rectStdDev.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv32u> d_pixelMask(gpuAllocator, d_srcImg.width(), d_srcImg.height());
    ncvAssertReturn(d_pixelMask.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVMatrixAlloc<Ncv32u> d_scaledIntegralImage(gpuAllocator, integralWidth, integralHeight);
    ncvAssertReturn(d_scaledIntegralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv64u> d_scaledSqIntegralImage(gpuAllocator, integralWidth, integralHeight);
    ncvAssertReturn(d_scaledSqIntegralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVVectorAlloc<NcvRect32u> d_hypothesesIntermediate(gpuAllocator, d_srcImg.width() * d_srcImg.height());
    ncvAssertReturn(d_hypothesesIntermediate.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVVectorAlloc<NcvRect32u> h_hypothesesIntermediate(cpuAllocator, d_srcImg.width() * d_srcImg.height());
    ncvAssertReturn(h_hypothesesIntermediate.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVStatus nppStat;
    Ncv32u szTmpBufIntegral, szTmpBufSqIntegral;
    nppStat = nppiStIntegralGetSize_8u32u(NcvSize32u(d_srcImg.width(), d_srcImg.height()), &szTmpBufIntegral, devProp);
    ncvAssertReturnNcvStat(nppStat);
    nppStat = nppiStSqrIntegralGetSize_8u64u(NcvSize32u(d_srcImg.width(), d_srcImg.height()), &szTmpBufSqIntegral, devProp);
    ncvAssertReturnNcvStat(nppStat);
    NCVVectorAlloc<Ncv8u> d_tmpIIbuf(gpuAllocator, std::max(szTmpBufIntegral, szTmpBufSqIntegral));
    ncvAssertReturn(d_tmpIIbuf.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCV_SKIP_COND_BEGIN

    nppStat = nppiStIntegral_8u32u_C1R(d_srcImg.ptr(), d_srcImg.pitch(),
                                       d_integralImage.ptr(), d_integralImage.pitch(),
                                       NcvSize32u(d_srcImg.width(), d_srcImg.height()),
                                       d_tmpIIbuf.ptr(), szTmpBufIntegral, devProp);
    ncvAssertReturnNcvStat(nppStat);

    nppStat = nppiStSqrIntegral_8u64u_C1R(d_srcImg.ptr(), d_srcImg.pitch(),
                                          d_sqIntegralImage.ptr(), d_sqIntegralImage.pitch(),
                                          NcvSize32u(d_srcImg.width(), d_srcImg.height()),
                                          d_tmpIIbuf.ptr(), szTmpBufSqIntegral, devProp);
    ncvAssertReturnNcvStat(nppStat);

    NCV_SKIP_COND_END

    dstNumRects = 0;

    Ncv32u lastCheckedScale = 0;
    NcvBool bReverseTraverseScale = ((flags & NCVPipeObjDet_FindLargestObject) != 0);
    std::vector<Ncv32u> scalesVector;

    NcvBool bFoundLargestFace = false;

    for (Ncv32f scaleIter = 1.0f; ; scaleIter *= scaleStep)
    {
        Ncv32u scale = (Ncv32u)scaleIter;
        if (lastCheckedScale == scale)
        {
            continue;
        }
        lastCheckedScale = scale;

        if (haar.ClassifierSize.width * (Ncv32s)scale < minObjSize.width ||
            haar.ClassifierSize.height * (Ncv32s)scale < minObjSize.height)
        {
            continue;
        }

        NcvSize32s srcRoi, srcIIRoi, scaledIIRoi, searchRoi;

        srcRoi.width = d_srcImg.width();
        srcRoi.height = d_srcImg.height();

        srcIIRoi.width = srcRoi.width + 1;
        srcIIRoi.height = srcRoi.height + 1;

        scaledIIRoi.width = srcIIRoi.width / scale;
        scaledIIRoi.height = srcIIRoi.height / scale;

        searchRoi.width = scaledIIRoi.width - haar.ClassifierSize.width;
        searchRoi.height = scaledIIRoi.height - haar.ClassifierSize.height;

        if (searchRoi.width <= 0 || searchRoi.height <= 0)
        {
            break;
        }

        scalesVector.push_back(scale);

        if (gpuAllocator.isCounting())
        {
            break;
        }
    }

    if (bReverseTraverseScale)
    {
        std::reverse(scalesVector.begin(), scalesVector.end());
    }

    //TODO: handle _fair_scale_ flag
    for (Ncv32u i=0; i<scalesVector.size(); i++)
    {
        Ncv32u scale = scalesVector[i];

        NcvSize32u srcRoi, scaledIIRoi, searchRoi;
        NcvSize32u srcIIRoi;
        srcRoi.width = d_srcImg.width();
        srcRoi.height = d_srcImg.height();
        srcIIRoi.width = srcRoi.width + 1;
        srcIIRoi.height = srcRoi.height + 1;
        scaledIIRoi.width = srcIIRoi.width / scale;
        scaledIIRoi.height = srcIIRoi.height / scale;
        searchRoi.width = scaledIIRoi.width - haar.ClassifierSize.width;
        searchRoi.height = scaledIIRoi.height - haar.ClassifierSize.height;

        NCV_SKIP_COND_BEGIN

        nppStat = nppiStDecimate_32u_C1R(
            d_integralImage.ptr(), d_integralImage.pitch(),
            d_scaledIntegralImage.ptr(), d_scaledIntegralImage.pitch(),
            srcIIRoi, scale, true);
        ncvAssertReturnNcvStat(nppStat);

        nppStat = nppiStDecimate_64u_C1R(
            d_sqIntegralImage.ptr(), d_sqIntegralImage.pitch(),
            d_scaledSqIntegralImage.ptr(), d_scaledSqIntegralImage.pitch(),
            srcIIRoi, scale, true);
        ncvAssertReturnNcvStat(nppStat);

        const NcvRect32u rect(
            HAAR_STDDEV_BORDER,
            HAAR_STDDEV_BORDER,
            haar.ClassifierSize.width - 2*HAAR_STDDEV_BORDER,
            haar.ClassifierSize.height - 2*HAAR_STDDEV_BORDER);
        nppStat = nppiStRectStdDev_32f_C1R(
            d_scaledIntegralImage.ptr(), d_scaledIntegralImage.pitch(),
            d_scaledSqIntegralImage.ptr(), d_scaledSqIntegralImage.pitch(),
            d_rectStdDev.ptr(), d_rectStdDev.pitch(),
            NcvSize32u(searchRoi.width, searchRoi.height), rect,
            (Ncv32f)scale*scale, true);
        ncvAssertReturnNcvStat(nppStat);

        NCV_SKIP_COND_END

        Ncv32u detectionsOnThisScale;
        ncvStat = ncvApplyHaarClassifierCascade_device(
            d_scaledIntegralImage, d_rectStdDev, d_pixelMask,
            detectionsOnThisScale,
            haar, h_HaarStages, d_HaarStages, d_HaarNodes, d_HaarFeatures, false,
            searchRoi, pixelStep, (Ncv32f)scale*scale,
            gpuAllocator, cpuAllocator, devProp, cuStream);
        ncvAssertReturnNcvStat(nppStat);

        NCV_SKIP_COND_BEGIN

        NCVVectorReuse<Ncv32u> d_vecPixelMask(d_pixelMask.getSegment());
        ncvStat = ncvGrowDetectionsVector_device(
            d_vecPixelMask,
            detectionsOnThisScale,
            d_hypothesesIntermediate,
            dstNumRects,
            static_cast<Ncv32u>(d_hypothesesIntermediate.length()),
            haar.ClassifierSize.width,
            haar.ClassifierSize.height,
            (Ncv32f)scale,
            cuStream);
        ncvAssertReturn(ncvStat == NCV_SUCCESS, ncvStat);

        if (flags & NCVPipeObjDet_FindLargestObject)
        {
            if (dstNumRects == 0)
            {
                continue;
            }

            if (dstNumRects != 0)
            {
                ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
                ncvStat = d_hypothesesIntermediate.copySolid(h_hypothesesIntermediate, cuStream,
                                                             dstNumRects * sizeof(NcvRect32u));
                ncvAssertReturnNcvStat(ncvStat);
                ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            }

            Ncv32u numStrongHypothesesNow = dstNumRects;
            // TODO Fix this to be back operational
            /*
            ncvStat = ncvGroupRectangles_host(
                h_hypothesesIntermediate,
                numStrongHypothesesNow,
                minNeighbors,
                RECT_SIMILARITY_PROPORTION,
                NULL);
            ncvAssertReturnNcvStat(ncvStat);
            */
            if (numStrongHypothesesNow > 0)
            {
                NcvRect32u maxRect = h_hypothesesIntermediate.ptr()[0];
                for (Ncv32u j=1; j<numStrongHypothesesNow; j++)
                {
                    if (maxRect.width < h_hypothesesIntermediate.ptr()[j].width)
                    {
                        maxRect = h_hypothesesIntermediate.ptr()[j];
                    }
                }

                h_hypothesesIntermediate.ptr()[0] = maxRect;
                dstNumRects = 1;

                ncvStat = h_hypothesesIntermediate.copySolid(d_dstRects, cuStream, sizeof(NcvRect32u));
                ncvAssertReturnNcvStat(ncvStat);

                bFoundLargestFace = true;

                break;
            }
        }

        NCV_SKIP_COND_END

        if (gpuAllocator.isCounting())
        {
            break;
        }
    }

    NCVStatus ncvRetCode = NCV_SUCCESS;

    NCV_SKIP_COND_BEGIN

    if (flags & NCVPipeObjDet_FindLargestObject)
    {
        if (!bFoundLargestFace)
        {
            dstNumRects = 0;
        }
    }
    else
    {
        //TODO: move hypotheses filtration to GPU pipeline (the only CPU-resident element of the pipeline left)
        if (dstNumRects != 0)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvStat = d_hypothesesIntermediate.copySolid(h_hypothesesIntermediate, cuStream,
                                                         dstNumRects * sizeof(NcvRect32u));
            ncvAssertReturnNcvStat(ncvStat);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }
        // Todo fix this to be back operational
        /*
        ncvStat = ncvGroupRectangles_host(
            h_hypothesesIntermediate,
            dstNumRects,
            minNeighbors,
            RECT_SIMILARITY_PROPORTION,
            NULL);
        ncvAssertReturnNcvStat(ncvStat);
        */
        if (dstNumRects > d_dstRects.length())
        {
            ncvRetCode = NCV_WARNING_HAAR_DETECTIONS_VECTOR_OVERFLOW;
            dstNumRects = static_cast<Ncv32u>(d_dstRects.length());
        }

        if (dstNumRects != 0)
        {
            ncvStat = h_hypothesesIntermediate.copySolid(d_dstRects, cuStream,
                                                         dstNumRects * sizeof(NcvRect32u));
            ncvAssertReturnNcvStat(ncvStat);
        }
    }

    if (flags & NCVPipeObjDet_VisualizeInPlace)
    {
        ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        ncvDrawRects_8u_device(d_srcImg.ptr(), d_srcImg.stride(),
                               d_srcImg.width(), d_srcImg.height(),
                               d_dstRects.ptr(), dstNumRects, 255, cuStream);
    }

    NCV_SKIP_COND_END

    return ncvRetCode;
}


//==============================================================================
//
// Purely Host code: classifier IO, mock-ups
//
//==============================================================================


#ifdef _SELF_TEST_
#include <float.h>
#endif

#define NVBIN_HAAR_SIZERESERVED     16
#define NVBIN_HAAR_VERSION          0x1

NCVStatus ncvApplyHaarClassifierCascade_host(NCVMatrix<Ncv32u> &h_integralImage,
                                             NCVMatrix<Ncv32f> &h_weights,
                                             NCVMatrixAlloc<Ncv32u> &h_pixelMask,
                                             Ncv32u &numDetections,
                                             HaarClassifierCascadeDescriptor &haar,
                                             NCVVector<HaarStage64> &h_HaarStages,
                                             NCVVector<HaarClassifierNode128> &h_HaarNodes,
                                             NCVVector<HaarFeature64> &h_HaarFeatures,
                                             NcvBool bMaskElements,
                                             NcvSize32u anchorsRoi,
                                             Ncv32u pixelStep,
                                             Ncv32f scaleArea)
{
    ncvAssertReturn(h_integralImage.memType() == h_weights.memType() &&
                    h_integralImage.memType() == h_pixelMask.memType() &&
                     (h_integralImage.memType() == NCVMemoryTypeHostPageable ||
                      h_integralImage.memType() == NCVMemoryTypeHostPinned), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(h_HaarStages.memType() == h_HaarNodes.memType() &&
                    h_HaarStages.memType() == h_HaarFeatures.memType() &&
                     (h_HaarStages.memType() == NCVMemoryTypeHostPageable ||
                      h_HaarStages.memType() == NCVMemoryTypeHostPinned), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(h_integralImage.ptr() != NULL && h_weights.ptr() != NULL && h_pixelMask.ptr() != NULL &&
                    h_HaarStages.ptr() != NULL && h_HaarNodes.ptr() != NULL && h_HaarFeatures.ptr() != NULL, NCV_NULL_PTR);
    ncvAssertReturn(anchorsRoi.width > 0 && anchorsRoi.height > 0 &&
                    h_pixelMask.width() >= anchorsRoi.width && h_pixelMask.height() >= anchorsRoi.height &&
                    h_weights.width() >= anchorsRoi.width && h_weights.height() >= anchorsRoi.height &&
                    h_integralImage.width() >= anchorsRoi.width + haar.ClassifierSize.width &&
                    h_integralImage.height() >= anchorsRoi.height + haar.ClassifierSize.height, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(scaleArea > 0, NCV_INVALID_SCALE);
    ncvAssertReturn(h_HaarStages.length() >= haar.NumStages &&
                    h_HaarNodes.length() >= haar.NumClassifierTotalNodes &&
                    h_HaarFeatures.length() >= haar.NumFeatures &&
                    h_HaarStages.length() == h_HaarStages.length() &&
                    haar.NumClassifierRootNodes <= haar.NumClassifierTotalNodes, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(haar.bNeedsTiltedII == false, NCV_NOIMPL_HAAR_TILTED_FEATURES);
    ncvAssertReturn(pixelStep == 1 || pixelStep == 2, NCV_HAAR_INVALID_PIXEL_STEP);

    Ncv32f scaleAreaPixels = scaleArea * ((haar.ClassifierSize.width - 2*HAAR_STDDEV_BORDER) *
                                          (haar.ClassifierSize.height - 2*HAAR_STDDEV_BORDER));

    for (Ncv32u i=0; i<anchorsRoi.height; i++)
    {
        for (Ncv32u j=0; j<h_pixelMask.stride(); j++)
        {
            if (i % pixelStep != 0 || j % pixelStep != 0 || j >= anchorsRoi.width)
            {
                h_pixelMask.ptr()[i * h_pixelMask.stride() + j] = OBJDET_MASK_ELEMENT_INVALID_32U;
            }
            else
            {
                for (Ncv32u iStage = 0; iStage < haar.NumStages; iStage++)
                {
                    Ncv32f curStageSum = 0.0f;
                    Ncv32u numRootNodesInStage = h_HaarStages.ptr()[iStage].getNumClassifierRootNodes();
                    Ncv32u curRootNodeOffset = h_HaarStages.ptr()[iStage].getStartClassifierRootNodeOffset();

                    if (iStage == 0)
                    {
                        if (bMaskElements && h_pixelMask.ptr()[i * h_pixelMask.stride() + j] == OBJDET_MASK_ELEMENT_INVALID_32U)
                        {
                            break;
                        }
                        else
                        {
                            h_pixelMask.ptr()[i * h_pixelMask.stride() + j] = ((i << 16) | j);
                        }
                    }
                    else if (h_pixelMask.ptr()[i * h_pixelMask.stride() + j] == OBJDET_MASK_ELEMENT_INVALID_32U)
                    {
                        break;
                    }

                    while (numRootNodesInStage--)
                    {
                        NcvBool bMoreNodesToTraverse = true;
                        Ncv32u curNodeOffset = curRootNodeOffset;

                        while (bMoreNodesToTraverse)
                        {
                            HaarClassifierNode128 curNode = h_HaarNodes.ptr()[curNodeOffset];
                            HaarFeatureDescriptor32 curFeatDesc = curNode.getFeatureDesc();
                            Ncv32u curNodeFeaturesNum = curFeatDesc.getNumFeatures();
                            Ncv32u curNodeFeaturesOffs = curFeatDesc.getFeaturesOffset();

                            Ncv32f curNodeVal = 0.f;
                            for (Ncv32u iRect=0; iRect<curNodeFeaturesNum; iRect++)
                            {
                                HaarFeature64 feature = h_HaarFeatures.ptr()[curNodeFeaturesOffs + iRect];
                                Ncv32u rectX, rectY, rectWidth, rectHeight;
                                feature.getRect(&rectX, &rectY, &rectWidth, &rectHeight);
                                Ncv32f rectWeight = feature.getWeight();
                                Ncv32u iioffsTL = (i + rectY) * h_integralImage.stride() + (j + rectX);
                                Ncv32u iioffsTR = iioffsTL + rectWidth;
                                Ncv32u iioffsBL = iioffsTL + rectHeight * h_integralImage.stride();
                                Ncv32u iioffsBR = iioffsBL + rectWidth;

                                Ncv32u iivalTL = h_integralImage.ptr()[iioffsTL];
                                Ncv32u iivalTR = h_integralImage.ptr()[iioffsTR];
                                Ncv32u iivalBL = h_integralImage.ptr()[iioffsBL];
                                Ncv32u iivalBR = h_integralImage.ptr()[iioffsBR];
                                Ncv32u rectSum = iivalBR - iivalBL + iivalTL - iivalTR;
                                curNodeVal += (Ncv32f)rectSum * rectWeight;
                            }

                            HaarClassifierNodeDescriptor32 nodeLeft = curNode.getLeftNodeDesc();
                            HaarClassifierNodeDescriptor32 nodeRight = curNode.getRightNodeDesc();
                            Ncv32f nodeThreshold = curNode.getThreshold();

                            HaarClassifierNodeDescriptor32 nextNodeDescriptor;
                            NcvBool nextNodeIsLeaf;

                            if (curNodeVal < scaleAreaPixels * h_weights.ptr()[i * h_weights.stride() + j] * nodeThreshold)
                            {
                                nextNodeDescriptor = nodeLeft;
                                nextNodeIsLeaf = curFeatDesc.isLeftNodeLeaf();
                            }
                            else
                            {
                                nextNodeDescriptor = nodeRight;
                                nextNodeIsLeaf = curFeatDesc.isRightNodeLeaf();
                            }

                            if (nextNodeIsLeaf)
                            {
                                Ncv32f tmpLeafValue = nextNodeDescriptor.getLeafValueHost();
                                curStageSum += tmpLeafValue;
                                bMoreNodesToTraverse = false;
                            }
                            else
                            {
                                curNodeOffset = nextNodeDescriptor.getNextNodeOffset();
                            }
                        }

                        curRootNodeOffset++;
                    }

                    Ncv32f tmpStageThreshold = h_HaarStages.ptr()[iStage].getStageThreshold();
                    if (curStageSum < tmpStageThreshold)
                    {
                        //drop
                        h_pixelMask.ptr()[i * h_pixelMask.stride() + j] = OBJDET_MASK_ELEMENT_INVALID_32U;
                        break;
                    }
                }
            }
        }
    }

    std::sort(h_pixelMask.ptr(), h_pixelMask.ptr() + anchorsRoi.height * h_pixelMask.stride());
    Ncv32u i = 0;
    for (; i<anchorsRoi.height * h_pixelMask.stride(); i++)
    {
        if (h_pixelMask.ptr()[i] == OBJDET_MASK_ELEMENT_INVALID_32U)
        {
            break;
        }
    }
    numDetections = i;

    return NCV_SUCCESS;
}

NCVStatus ncvGrowDetectionsVector_host(NCVVector<Ncv32u> &pixelMask,
                                       Ncv32u numPixelMaskDetections,
                                       NCVVector<NcvRect32u> &hypotheses,
                                       Ncv32u &totalDetections,
                                       Ncv32u totalMaxDetections,
                                       Ncv32u rectWidth,
                                       Ncv32u rectHeight,
                                       Ncv32f curScale)
{
    ncvAssertReturn(pixelMask.ptr() != NULL && hypotheses.ptr() != NULL, NCV_NULL_PTR);
    ncvAssertReturn(pixelMask.memType() == hypotheses.memType() &&
                    pixelMask.memType() != NCVMemoryTypeDevice, NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(rectWidth > 0 && rectHeight > 0 && curScale > 0, NCV_INVALID_ROI);
    ncvAssertReturn(curScale > 0, NCV_INVALID_SCALE);
    ncvAssertReturn(totalMaxDetections <= hypotheses.length() &&
                    numPixelMaskDetections <= pixelMask.length(), NCV_INCONSISTENT_INPUT);

    NCVStatus ncvStat = NCV_SUCCESS;
    Ncv32u numDetsToCopy = numPixelMaskDetections;

    if (numDetsToCopy == 0)
    {
        return ncvStat;
    }

    if (totalDetections + numPixelMaskDetections > totalMaxDetections)
    {
        ncvStat = NCV_WARNING_HAAR_DETECTIONS_VECTOR_OVERFLOW;
        numDetsToCopy = totalMaxDetections - totalDetections;
    }

    for (Ncv32u i=0; i<numDetsToCopy; i++)
    {
        hypotheses.ptr()[totalDetections + i] = pixelToRect(pixelMask.ptr()[i], rectWidth, rectHeight, curScale);
    }

    totalDetections += numDetsToCopy;
    return ncvStat;
}

NCVStatus ncvHaarStoreNVBIN_host(const std::string &filename,
                                 HaarClassifierCascadeDescriptor haar,
                                 NCVVector<HaarStage64> &h_HaarStages,
                                 NCVVector<HaarClassifierNode128> &h_HaarNodes,
                                 NCVVector<HaarFeature64> &h_HaarFeatures)
{
    ncvAssertReturn(h_HaarStages.length() >= haar.NumStages, NCV_INCONSISTENT_INPUT);
    ncvAssertReturn(h_HaarNodes.length() >= haar.NumClassifierTotalNodes, NCV_INCONSISTENT_INPUT);
    ncvAssertReturn(h_HaarFeatures.length() >= haar.NumFeatures, NCV_INCONSISTENT_INPUT);
    ncvAssertReturn(h_HaarStages.memType() == NCVMemoryTypeHostPinned &&
                    h_HaarNodes.memType() == NCVMemoryTypeHostPinned &&
                    h_HaarFeatures.memType() == NCVMemoryTypeHostPinned, NCV_MEM_RESIDENCE_ERROR);

    Ncv32u szStages = haar.NumStages * sizeof(HaarStage64);
    Ncv32u szClassifiers = haar.NumClassifierTotalNodes * sizeof(HaarClassifierNode128);
    Ncv32u szFeatures = haar.NumFeatures * sizeof(HaarFeature64);

    Ncv32u dataOffset = 0;
    std::vector<unsigned char> fdata;
    fdata.resize(szStages+szClassifiers+szFeatures+1024, 0);

    //header
    *(Ncv32u *)(&fdata[0]+dataOffset) = NVBIN_HAAR_VERSION;

    //data
    dataOffset = NVBIN_HAAR_SIZERESERVED;
    *(Ncv32u *)(&fdata[0]+dataOffset) = haar.NumStages;
    dataOffset += sizeof(Ncv32u);
    *(Ncv32u *)(&fdata[0]+dataOffset) = haar.NumClassifierRootNodes;
    dataOffset += sizeof(Ncv32u);
    *(Ncv32u *)(&fdata[0]+dataOffset) = haar.NumClassifierTotalNodes;
    dataOffset += sizeof(Ncv32u);
    *(Ncv32u *)(&fdata[0]+dataOffset) = haar.NumFeatures;
    dataOffset += sizeof(Ncv32u);
    *(NcvSize32u *)(&fdata[0]+dataOffset) = haar.ClassifierSize;
    dataOffset += sizeof(NcvSize32u);
    *(NcvBool *)(&fdata[0]+dataOffset) = haar.bNeedsTiltedII;
    dataOffset += sizeof(NcvBool);
    *(NcvBool *)(&fdata[0]+dataOffset) = haar.bHasStumpsOnly;
    dataOffset += sizeof(NcvBool);

    memcpy(&fdata[0]+dataOffset, h_HaarStages.ptr(), szStages);
    dataOffset += szStages;
    memcpy(&fdata[0]+dataOffset, h_HaarNodes.ptr(), szClassifiers);
    dataOffset += szClassifiers;
    memcpy(&fdata[0]+dataOffset, h_HaarFeatures.ptr(), szFeatures);
    dataOffset += szFeatures;
    Ncv32u fsize = dataOffset;

    //TODO: CRC32 here

    //update header
    dataOffset = sizeof(Ncv32u);
    *(Ncv32u *)(&fdata[0]+dataOffset) = fsize;

    FILE *fp = fopen(filename.c_str(), "wb");
    ncvAssertReturn(fp != NULL, NCV_FILE_ERROR);
    fwrite(&fdata[0], fsize, 1, fp);
    fclose(fp);
    return NCV_SUCCESS;
}
