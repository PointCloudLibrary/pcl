#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (C) 2009-2010, NVIDIA Corporation, all rights reserved.
 *  Third party copyrights are property of their respective owners.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 * $Id:  $
 * Ported to PCL by Koen Buys : Attention Work in progress!
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "NCV.hpp"
#include "NCVAlg.hpp"
#include "NCVPyramid.hpp"
#include "NCVPixelOperations.hpp"
//#include "opencv2/gpu/device/common.hpp"

template<typename T, Ncv32u CN> struct __average4_CN {static __host__ __device__ T _average4_CN(const T &p00, const T &p01, const T &p10, const T &p11);};

template<typename T> struct __average4_CN<T, 1> {
static __host__ __device__ T _average4_CN(const T &p00, const T &p01, const T &p10, const T &p11)
{
    T out;
    out.x = ((Ncv32s)p00.x + p01.x + p10.x + p11.x + 2) / 4;
    return out;
}};

template<> struct __average4_CN<float1, 1> {
static __host__ __device__ float1 _average4_CN(const float1 &p00, const float1 &p01, const float1 &p10, const float1 &p11)
{
    float1 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    return out;
}};

template<> struct __average4_CN<double1, 1> {
static __host__ __device__ double1 _average4_CN(const double1 &p00, const double1 &p01, const double1 &p10, const double1 &p11)
{
    double1 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    return out;
}};

template<typename T> struct __average4_CN<T, 3> {
static __host__ __device__ T _average4_CN(const T &p00, const T &p01, const T &p10, const T &p11)
{
    T out;
    out.x = ((Ncv32s)p00.x + p01.x + p10.x + p11.x + 2) / 4;
    out.y = ((Ncv32s)p00.y + p01.y + p10.y + p11.y + 2) / 4;
    out.z = ((Ncv32s)p00.z + p01.z + p10.z + p11.z + 2) / 4;
    return out;
}};

template<> struct __average4_CN<float3, 3> {
static __host__ __device__ float3 _average4_CN(const float3 &p00, const float3 &p01, const float3 &p10, const float3 &p11)
{
    float3 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    out.y = (p00.y + p01.y + p10.y + p11.y) / 4;
    out.z = (p00.z + p01.z + p10.z + p11.z) / 4;
    return out;
}};

template<> struct __average4_CN<double3, 3> {
static __host__ __device__ double3 _average4_CN(const double3 &p00, const double3 &p01, const double3 &p10, const double3 &p11)
{
    double3 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    out.y = (p00.y + p01.y + p10.y + p11.y) / 4;
    out.z = (p00.z + p01.z + p10.z + p11.z) / 4;
    return out;
}};

template<typename T> struct __average4_CN<T, 4> {
static __host__ __device__ T _average4_CN(const T &p00, const T &p01, const T &p10, const T &p11)
{
    T out;
    out.x = ((Ncv32s)p00.x + p01.x + p10.x + p11.x + 2) / 4;
    out.y = ((Ncv32s)p00.y + p01.y + p10.y + p11.y + 2) / 4;
    out.z = ((Ncv32s)p00.z + p01.z + p10.z + p11.z + 2) / 4;
    out.w = ((Ncv32s)p00.w + p01.w + p10.w + p11.w + 2) / 4;
    return out;
}};

template<> struct __average4_CN<float4, 4> {
static __host__ __device__ float4 _average4_CN(const float4 &p00, const float4 &p01, const float4 &p10, const float4 &p11)
{
    float4 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    out.y = (p00.y + p01.y + p10.y + p11.y) / 4;
    out.z = (p00.z + p01.z + p10.z + p11.z) / 4;
    out.w = (p00.w + p01.w + p10.w + p11.w) / 4;
    return out;
}};

template<> struct __average4_CN<double4, 4> {
static __host__ __device__ double4 _average4_CN(const double4 &p00, const double4 &p01, const double4 &p10, const double4 &p11)
{
    double4 out;
    out.x = (p00.x + p01.x + p10.x + p11.x) / 4;
    out.y = (p00.y + p01.y + p10.y + p11.y) / 4;
    out.z = (p00.z + p01.z + p10.z + p11.z) / 4;
    out.w = (p00.w + p01.w + p10.w + p11.w) / 4;
    return out;
}};

template<typename T> static __host__ __device__ T _average4(const T &p00, const T &p01, const T &p10, const T &p11)
{
    return __average4_CN<T, NC(T)>::_average4_CN(p00, p01, p10, p11);
}

template<typename Tin, typename Tout, Ncv32u CN> struct __lerp_CN {static __host__ __device__ Tout _lerp_CN(const Tin &a, const Tin &b, Ncv32f d);};

template<typename Tin, typename Tout> struct __lerp_CN<Tin, Tout, 1> {
static __host__ __device__ Tout _lerp_CN(const Tin &a, const Tin &b, Ncv32f d)
{
    using TB = typename TConvVec2Base<Tout>::TBase;
    return _pixMake(TB(b.x * d + a.x * (1 - d)));
}};

template<typename Tin, typename Tout> struct __lerp_CN<Tin, Tout, 3> {
static __host__ __device__ Tout _lerp_CN(const Tin &a, const Tin &b, Ncv32f d)
{
    using TB = typename TConvVec2Base<Tout>::TBase;
    return _pixMake(TB(b.x * d + a.x * (1 - d)),
                    TB(b.y * d + a.y * (1 - d)),
                    TB(b.z * d + a.z * (1 - d)));
}};

template<typename Tin, typename Tout> struct __lerp_CN<Tin, Tout, 4> {
static __host__ __device__ Tout _lerp_CN(const Tin &a, const Tin &b, Ncv32f d)
{
    using TB = typename TConvVec2Base<Tout>::TBase;
    return _pixMake(TB(b.x * d + a.x * (1 - d)),
                    TB(b.y * d + a.y * (1 - d)),
                    TB(b.z * d + a.z * (1 - d)),
                    TB(b.w * d + a.w * (1 - d)));
}};

template<typename Tin, typename Tout> static __host__ __device__ Tout _lerp(const Tin &a, const Tin &b, Ncv32f d)
{
    return __lerp_CN<Tin, Tout, NC(Tin)>::_lerp_CN(a, b, d);
}

template<typename T>
__global__ void kernelDownsampleX2(T *d_src,
                                   Ncv32u srcPitch,
                                   T *d_dst,
                                   Ncv32u dstPitch,
                                   NcvSize32u dstRoi)
{
    Ncv32u i = blockIdx.y * blockDim.y + threadIdx.y;
    Ncv32u j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < dstRoi.height && j < dstRoi.width)
    {
        T *d_src_line1 = (T *)((Ncv8u *)d_src + (2 * i + 0) * srcPitch);
        T *d_src_line2 = (T *)((Ncv8u *)d_src + (2 * i + 1) * srcPitch);
        T *d_dst_line = (T *)((Ncv8u *)d_dst + i * dstPitch);

        T p00 = d_src_line1[2*j+0];
        T p01 = d_src_line1[2*j+1];
        T p10 = d_src_line2[2*j+0];
        T p11 = d_src_line2[2*j+1];

        d_dst_line[j] = _average4(p00, p01, p10, p11);
    }
}

/*
namespace cv { namespace gpu { namespace device 
{
    namespace pyramid
    {
        template <typename T> void kernelDownsampleX2_gpu(DevMem2Db src, DevMem2Db dst, hipStream_t stream)
        {
            dim3 bDim(16, 8);
            dim3 gDim(divUp(src.cols, bDim.x), divUp(src.rows, bDim.y));

            kernelDownsampleX2<<<gDim, bDim, 0, stream>>>((T*)src.data, src.step, (T*)dst.data, dst.step, NcvSize32u(dst.cols, dst.rows));

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void kernelDownsampleX2_gpu<uchar1>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelDownsampleX2_gpu<uchar3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelDownsampleX2_gpu<uchar4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        template void kernelDownsampleX2_gpu<ushort1>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelDownsampleX2_gpu<ushort3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelDownsampleX2_gpu<ushort4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        template void kernelDownsampleX2_gpu<float1>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelDownsampleX2_gpu<float3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelDownsampleX2_gpu<float4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
    }
}}} */

template<typename T>
__global__ void kernelInterpolateFrom1(T *d_srcTop,
                                       Ncv32u srcTopPitch,
                                       NcvSize32u szTopRoi,
                                       T *d_dst,
                                       Ncv32u dstPitch,
                                       NcvSize32u dstRoi)
{
    Ncv32u i = blockIdx.y * blockDim.y + threadIdx.y;
    Ncv32u j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < dstRoi.height && j < dstRoi.width)
    {
        Ncv32f ptTopX = 1.0f * (szTopRoi.width - 1) * j / (dstRoi.width - 1);
        Ncv32f ptTopY = 1.0f * (szTopRoi.height - 1) * i / (dstRoi.height - 1);
        Ncv32u xl = (Ncv32u)ptTopX;
        Ncv32u xh = xl+1;
        Ncv32f dx = ptTopX - xl;
        Ncv32u yl = (Ncv32u)ptTopY;
        Ncv32u yh = yl+1;
        Ncv32f dy = ptTopY - yl;

        T *d_src_line1 = (T *)((Ncv8u *)d_srcTop + yl * srcTopPitch);
        T *d_src_line2 = (T *)((Ncv8u *)d_srcTop + yh * srcTopPitch);
        T *d_dst_line = (T *)((Ncv8u *)d_dst + i * dstPitch);

        T p00, p01, p10, p11;
        p00 = d_src_line1[xl];
        p01 = xh < szTopRoi.width ? d_src_line1[xh] : p00;
        p10 = yh < szTopRoi.height ? d_src_line2[xl] : p00;
        p11 = (xh < szTopRoi.width && yh < szTopRoi.height) ? d_src_line2[xh] : p00;
        using TVFlt = typename TConvBase2Vec<Ncv32f, NC(T)>::TVec;
        TVFlt m_00_01 = _lerp<T, TVFlt>(p00, p01, dx);
        TVFlt m_10_11 = _lerp<T, TVFlt>(p10, p11, dx);
        TVFlt mixture = _lerp<TVFlt, TVFlt>(m_00_01, m_10_11, dy);
        T outPix = _pixDemoteClampZ<TVFlt, T>(mixture);

        d_dst_line[j] = outPix;
    }
}

/*
namespace cv { namespace gpu { namespace device 
{
    namespace pyramid
    {
        template <typename T> void kernelInterpolateFrom1_gpu(DevMem2Db src, DevMem2Db dst, hipStream_t stream)
        {
            dim3 bDim(16, 8);
            dim3 gDim(divUp(dst.cols, bDim.x), divUp(dst.rows, bDim.y));

            kernelInterpolateFrom1<<<gDim, bDim, 0, stream>>>((T*) src.data, src.step, NcvSize32u(src.cols, src.rows), 
                (T*) dst.data, dst.step, NcvSize32u(dst.cols, dst.rows));

            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void kernelInterpolateFrom1_gpu<uchar1>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelInterpolateFrom1_gpu<uchar3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelInterpolateFrom1_gpu<uchar4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        template void kernelInterpolateFrom1_gpu<ushort1>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelInterpolateFrom1_gpu<ushort3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelInterpolateFrom1_gpu<ushort4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);

        template void kernelInterpolateFrom1_gpu<float1>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelInterpolateFrom1_gpu<float3>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
        template void kernelInterpolateFrom1_gpu<float4>(DevMem2Db src, DevMem2Db dst, hipStream_t stream);
    }
}}} */


#if 0 //def _WIN32

template<typename T>
static T _interpLinear(const T &a, const T &b, Ncv32f d)
{
    using TVFlt = typename TConvBase2Vec<Ncv32f, NC(T)>::TVec;
    TVFlt tmp = _lerp<T, TVFlt>(a, b, d);
    return _pixDemoteClampZ<TVFlt, T>(tmp);
}

template<typename T>
static T _interpBilinear(const NCVMatrix<T> &refLayer, Ncv32f x, Ncv32f y)
{
    Ncv32u xl = (Ncv32u)x;
    Ncv32u xh = xl+1;
    Ncv32f dx = x - xl;
    Ncv32u yl = (Ncv32u)y;
    Ncv32u yh = yl+1;
    Ncv32f dy = y - yl;
    T p00, p01, p10, p11;
    p00 = refLayer.at(xl, yl);
    p01 = xh < refLayer.width() ? refLayer.at(xh, yl) : p00;
    p10 = yh < refLayer.height() ? refLayer.at(xl, yh) : p00;
    p11 = (xh < refLayer.width() && yh < refLayer.height()) ? refLayer.at(xh, yh) : p00;
    using TVFlt = typename TConvBase2Vec<Ncv32f, NC(T)>::TVec;
    TVFlt m_00_01 = _lerp<T, TVFlt>(p00, p01, dx);
    TVFlt m_10_11 = _lerp<T, TVFlt>(p10, p11, dx);
    TVFlt mixture = _lerp<TVFlt, TVFlt>(m_00_01, m_10_11, dy);
    return _pixDemoteClampZ<TVFlt, T>(mixture);
}

template <class T>
NCVImagePyramid<T>::NCVImagePyramid(const NCVMatrix<T> &img,
                                    Ncv8u numLayers,
                                    INCVMemAllocator &alloc,
                                    hipStream_t cuStream)
{
    this->_isInitialized = false;
    ncvAssertPrintReturn(img.memType() == alloc.memType(), "NCVImagePyramid::ctor error", );

    this->layer0 = &img;
    NcvSize32u szLastLayer(img.width(), img.height());
    this->nLayers = 1;

    NCV_SET_SKIP_COND(alloc.isCounting());
    NcvBool bDeviceCode = alloc.memType() == NCVMemoryTypeDevice;

    if (numLayers == 0)
    {
        numLayers = 255; //it will cut-off when any of the dimensions goes 1
    }

#ifdef SELF_CHECK_GPU
    NCVMemNativeAllocator allocCPU(NCVMemoryTypeHostPinned, 512);
#endif

    for (Ncv32u i=0; i<(Ncv32u)numLayers-1; i++)
    {
        NcvSize32u szCurLayer(szLastLayer.width / 2, szLastLayer.height / 2);
        if (szCurLayer.width == 0 || szCurLayer.height == 0)
        {
            break;
        }

        this->pyramid.push_back(new NCVMatrixAlloc<T>(alloc, szCurLayer.width, szCurLayer.height));
        ncvAssertPrintReturn(((NCVMatrixAlloc<T> *)(this->pyramid[i]))->isMemAllocated(), "NCVImagePyramid::ctor error", );
        this->nLayers++;

        //fill in the layer
        NCV_SKIP_COND_BEGIN

        const NCVMatrix<T> *prevLayer = i == 0 ? this->layer0 : this->pyramid[i-1];
        NCVMatrix<T> *curLayer = this->pyramid[i];

        if (bDeviceCode)
        {
            dim3 bDim(16, 8);
            dim3 gDim(divUp(szCurLayer.width, bDim.x), divUp(szCurLayer.height, bDim.y));
            kernelDownsampleX2<<<gDim, bDim, 0, cuStream>>>(prevLayer->ptr(),
                                                            prevLayer->pitch(),
                                                            curLayer->ptr(),
                                                            curLayer->pitch(),
                                                            szCurLayer);
            ncvAssertPrintReturn(hipSuccess == hipGetLastError(), "NCVImagePyramid::ctor error", );

#ifdef SELF_CHECK_GPU
            NCVMatrixAlloc<T> h_prevLayer(allocCPU, prevLayer->width(), prevLayer->height());
            ncvAssertPrintReturn(h_prevLayer.isMemAllocated(), "Validation failure in NCVImagePyramid::ctor", );
            NCVMatrixAlloc<T> h_curLayer(allocCPU, curLayer->width(), curLayer->height());
            ncvAssertPrintReturn(h_curLayer.isMemAllocated(), "Validation failure in NCVImagePyramid::ctor", );
            ncvAssertPrintReturn(NCV_SUCCESS == prevLayer->copy2D(h_prevLayer, prevLayer->size(), cuStream), "Validation failure in NCVImagePyramid::ctor", );
            ncvAssertPrintReturn(NCV_SUCCESS == curLayer->copy2D(h_curLayer, curLayer->size(), cuStream), "Validation failure in NCVImagePyramid::ctor", );
            ncvAssertPrintReturn(hipSuccess == hipStreamSynchronize(cuStream), "Validation failure in NCVImagePyramid::ctor", );
            for (Ncv32u i=0; i<szCurLayer.height; i++)
            {
                for (Ncv32u j=0; j<szCurLayer.width; j++)
                {
                    T p00 = h_prevLayer.at(2*j+0, 2*i+0);
                    T p01 = h_prevLayer.at(2*j+1, 2*i+0);
                    T p10 = h_prevLayer.at(2*j+0, 2*i+1);
                    T p11 = h_prevLayer.at(2*j+1, 2*i+1);
                    T outGold = _average4(p00, p01, p10, p11);
                    T outGPU = h_curLayer.at(j, i);
                    ncvAssertPrintReturn(0 == memcmp(&outGold, &outGPU, sizeof(T)), "Validation failure in NCVImagePyramid::ctor with kernelDownsampleX2", );
                }
            }
#endif
        }
        else
        {
            for (Ncv32u i=0; i<szCurLayer.height; i++)
            {
                for (Ncv32u j=0; j<szCurLayer.width; j++)
                {
                    T p00 = prevLayer->at(2*j+0, 2*i+0);
                    T p01 = prevLayer->at(2*j+1, 2*i+0);
                    T p10 = prevLayer->at(2*j+0, 2*i+1);
                    T p11 = prevLayer->at(2*j+1, 2*i+1);
                    curLayer->at(j, i) = _average4(p00, p01, p10, p11);
                }
            }
        }

        NCV_SKIP_COND_END

        szLastLayer = szCurLayer;
    }

    this->_isInitialized = true;
}

template <class T>
NCVImagePyramid<T>::~NCVImagePyramid()
{
}

template <class T>
NcvBool NCVImagePyramid<T>::isInitialized() const
{
    return this->_isInitialized;
}

template <class T>
NCVStatus NCVImagePyramid<T>::getLayer(NCVMatrix<T> &outImg,
                                       NcvSize32u outRoi,
                                       NcvBool bTrilinear,
                                       hipStream_t cuStream) const
{
    ncvAssertReturn(this->isInitialized(), NCV_UNKNOWN_ERROR);
    ncvAssertReturn(outImg.memType() == this->layer0->memType(), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(outRoi.width <= this->layer0->width() && outRoi.height <= this->layer0->height() &&
                    outRoi.width > 0 && outRoi.height > 0, NCV_DIMENSIONS_INVALID);

    if (outRoi.width == this->layer0->width() && outRoi.height == this->layer0->height())
    {
        ncvAssertReturnNcvStat(this->layer0->copy2D(outImg, NcvSize32u(this->layer0->width(), this->layer0->height()), cuStream));
        return NCV_SUCCESS;
    }

    Ncv32f lastScale = 1.0f;
    Ncv32f curScale;
    const NCVMatrix<T> *lastLayer = this->layer0;
    const NCVMatrix<T> *curLayer = NULL;
    NcvBool bUse2Refs = false;

    for (Ncv32u i=0; i<this->nLayers-1; i++)
    {
        curScale = lastScale * 0.5f;
        curLayer = this->pyramid[i];

        if (outRoi.width == curLayer->width() && outRoi.height == curLayer->height())
        {
            ncvAssertReturnNcvStat(this->pyramid[i]->copy2D(outImg, NcvSize32u(this->pyramid[i]->width(), this->pyramid[i]->height()), cuStream));
            return NCV_SUCCESS;
        }

        if (outRoi.width >= curLayer->width() && outRoi.height >= curLayer->height())
        {
            if (outRoi.width < lastLayer->width() && outRoi.height < lastLayer->height())
            {
                bUse2Refs = true;
            }
            break;
        }

        lastScale = curScale;
        lastLayer = curLayer;
    }

    bUse2Refs = bUse2Refs && bTrilinear;

    NCV_SET_SKIP_COND(outImg.memType() == NCVMemoryTypeNone);
    NcvBool bDeviceCode = this->layer0->memType() == NCVMemoryTypeDevice;

#ifdef SELF_CHECK_GPU
    NCVMemNativeAllocator allocCPU(NCVMemoryTypeHostPinned, 512);
#endif

    NCV_SKIP_COND_BEGIN

    if (bDeviceCode)
    {
        ncvAssertReturn(bUse2Refs == false, NCV_NOT_IMPLEMENTED);

        dim3 bDim(16, 8);
        dim3 gDim(divUp(outRoi.width, bDim.x), divUp(outRoi.height, bDim.y));
        kernelInterpolateFrom1<<<gDim, bDim, 0, cuStream>>>(lastLayer->ptr(),
                                                            lastLayer->pitch(),
                                                            lastLayer->size(),
                                                            outImg.ptr(),
                                                            outImg.pitch(),
                                                            outRoi);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

#ifdef SELF_CHECK_GPU
        ncvSafeMatAlloc(h_lastLayer, T, allocCPU, lastLayer->width(), lastLayer->height(), NCV_ALLOCATOR_BAD_ALLOC);
        ncvSafeMatAlloc(h_outImg, T, allocCPU, outImg.width(), outImg.height(), NCV_ALLOCATOR_BAD_ALLOC);
        ncvAssertReturnNcvStat(lastLayer->copy2D(h_lastLayer, lastLayer->size(), cuStream));
        ncvAssertReturnNcvStat(outImg.copy2D(h_outImg, outRoi, cuStream));
        ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);

        for (Ncv32u i=0; i<outRoi.height; i++)
        {
            for (Ncv32u j=0; j<outRoi.width; j++)
            {
                NcvSize32u szTopLayer(lastLayer->width(), lastLayer->height());
                Ncv32f ptTopX = 1.0f * (szTopLayer.width - 1) * j / (outRoi.width - 1);
                Ncv32f ptTopY = 1.0f * (szTopLayer.height - 1) * i / (outRoi.height - 1);
                T outGold = _interpBilinear(h_lastLayer, ptTopX, ptTopY);
                ncvAssertPrintReturn(0 == memcmp(&outGold, &h_outImg.at(j,i), sizeof(T)), "Validation failure in NCVImagePyramid::ctor with kernelInterpolateFrom1", NCV_UNKNOWN_ERROR);
            }
        }
#endif
    }
    else
    {
        for (Ncv32u i=0; i<outRoi.height; i++)
        {
            for (Ncv32u j=0; j<outRoi.width; j++)
            {
                //top layer pixel (always exists)
                NcvSize32u szTopLayer(lastLayer->width(), lastLayer->height());
                Ncv32f ptTopX = 1.0f * (szTopLayer.width - 1) * j / (outRoi.width - 1);
                Ncv32f ptTopY = 1.0f * (szTopLayer.height - 1) * i / (outRoi.height - 1);
                T topPix = _interpBilinear(*lastLayer, ptTopX, ptTopY);
                T trilinearPix = topPix;

                if (bUse2Refs)
                {
                    //bottom layer pixel (exists only if the requested scale is greater than the smallest layer scale)
                    NcvSize32u szBottomLayer(curLayer->width(), curLayer->height());
                    Ncv32f ptBottomX = 1.0f * (szBottomLayer.width - 1) * j / (outRoi.width - 1);
                    Ncv32f ptBottomY = 1.0f * (szBottomLayer.height - 1) * i / (outRoi.height - 1);
                    T bottomPix = _interpBilinear(*curLayer, ptBottomX, ptBottomY);

                    Ncv32f scale = (1.0f * outRoi.width / layer0->width() + 1.0f * outRoi.height / layer0->height()) / 2;
                    Ncv32f dl = (scale - curScale) / (lastScale - curScale);
                    dl = CLAMP(dl, 0.0f, 1.0f);
                    trilinearPix = _interpLinear(bottomPix, topPix, dl);
                }

                outImg.at(j, i) = trilinearPix;
            }
        }
    }

    NCV_SKIP_COND_END

    return NCV_SUCCESS;
}

template class NCVImagePyramid<uchar1>;
template class NCVImagePyramid<uchar3>;
template class NCVImagePyramid<uchar4>;
template class NCVImagePyramid<ushort1>;
template class NCVImagePyramid<ushort3>;
template class NCVImagePyramid<ushort4>;
template class NCVImagePyramid<uint1>;
template class NCVImagePyramid<uint3>;
template class NCVImagePyramid<uint4>;
template class NCVImagePyramid<float1>;
template class NCVImagePyramid<float3>;
template class NCVImagePyramid<float4>;

#endif //_WIN32
