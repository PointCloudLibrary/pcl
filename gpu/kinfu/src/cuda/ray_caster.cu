#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */
#include "device.hpp"

namespace pcl
{
  namespace device
  {
    __device__ __forceinline__ float
    getMinTime (const float3& volume_max, const float3& origin, const float3& dir)
    {
      float txmin = ( (dir.x > 0 ? 0.f : volume_max.x) - origin.x) / dir.x;
      float tymin = ( (dir.y > 0 ? 0.f : volume_max.y) - origin.y) / dir.y;
      float tzmin = ( (dir.z > 0 ? 0.f : volume_max.z) - origin.z) / dir.z;

      return fmax ( fmax (txmin, tymin), tzmin);
    }

    __device__ __forceinline__ float
    getMaxTime (const float3& volume_max, const float3& origin, const float3& dir)
    {
      float txmax = ( (dir.x > 0 ? volume_max.x : 0.f) - origin.x) / dir.x;
      float tymax = ( (dir.y > 0 ? volume_max.y : 0.f) - origin.y) / dir.y;
      float tzmax = ( (dir.z > 0 ? volume_max.z : 0.f) - origin.z) / dir.z;

      return fmin (fmin (txmax, tymax), tzmax);
    }

    struct RayCaster
    {
      enum { CTA_SIZE_X = 32, CTA_SIZE_Y = 8 };

      Mat33 Rcurr;
      float3 tcurr;

      float time_step;
      float3 volume_size;

      float3 cell_size;
      int cols, rows;

      PtrStep<short2> volume;

      Intr intr;

      mutable PtrStep<float> nmap;
      mutable PtrStep<float> vmap;

      __device__ __forceinline__ float3
      get_ray_next (int x, int y) const
      {
        float3 ray_next;
        ray_next.x = (x - intr.cx) / intr.fx;
        ray_next.y = (y - intr.cy) / intr.fy;
        ray_next.z = 1;
        return ray_next;
      }

      __device__ __forceinline__ bool
      checkInds (const int3& g) const
      {
        return (g.x >= 0 && g.y >= 0 && g.z >= 0 && g.x < VOLUME_X && g.y < VOLUME_Y && g.z < VOLUME_Z);
      }

      __device__ __forceinline__ float
      readTsdf (int x, int y, int z) const
      {
        return unpack_tsdf (volume.ptr (VOLUME_Y * z + y)[x]);
      }

      __device__ __forceinline__ int3
      getVoxel (float3 point) const
      {
        int vx = __float2int_rd (point.x / cell_size.x);        // round to negative infinity
        int vy = __float2int_rd (point.y / cell_size.y);
        int vz = __float2int_rd (point.z / cell_size.z);

        return make_int3 (vx, vy, vz);
      }

      __device__ __forceinline__ float
      interpolateTrilineary (const float3& origin, const float3& dir, float time) const
      {
        return interpolateTrilineary (origin + dir * time);
      }

      __device__ __forceinline__ float
      interpolateTrilineary (const float3& point) const
      {
        int3 g = getVoxel (point);

        if (g.x <= 0 || g.x >= VOLUME_X - 1)
          return std::numeric_limits<float>::quiet_NaN ();

        if (g.y <= 0 || g.y >= VOLUME_Y - 1)
          return std::numeric_limits<float>::quiet_NaN ();

        if (g.z <= 0 || g.z >= VOLUME_Z - 1)
          return std::numeric_limits<float>::quiet_NaN ();

        float vx = (g.x + 0.5f) * cell_size.x;
        float vy = (g.y + 0.5f) * cell_size.y;
        float vz = (g.z + 0.5f) * cell_size.z;

        g.x = (point.x < vx) ? (g.x - 1) : g.x;
        g.y = (point.y < vy) ? (g.y - 1) : g.y;
        g.z = (point.z < vz) ? (g.z - 1) : g.z;

        float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
        float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
        float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

        float res = readTsdf (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                    readTsdf (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                    readTsdf (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                    readTsdf (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                    readTsdf (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                    readTsdf (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                    readTsdf (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                    readTsdf (g.x + 1, g.y + 1, g.z + 1) * a * b * c;
        return res;
      }
      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= cols || y >= rows)
          return;

        vmap.ptr (y)[x] = std::numeric_limits<float>::quiet_NaN ();
        nmap.ptr (y)[x] = std::numeric_limits<float>::quiet_NaN ();

        float3 ray_start = tcurr;
        float3 ray_next = Rcurr * get_ray_next (x, y) + tcurr;

        float3 ray_dir = normalized (ray_next - ray_start);

        //ensure that it isn't a degenerate case
        ray_dir.x = (ray_dir.x == 0.f) ? 1e-15 : ray_dir.x;
        ray_dir.y = (ray_dir.y == 0.f) ? 1e-15 : ray_dir.y;
        ray_dir.z = (ray_dir.z == 0.f) ? 1e-15 : ray_dir.z;

        // computer time when entry and exit volume
        float time_start_volume = getMinTime (volume_size, ray_start, ray_dir);
        float time_exit_volume = getMaxTime (volume_size, ray_start, ray_dir);

        const float min_dist = 0.f;         //in meters
        time_start_volume = fmax (time_start_volume, min_dist);
        if (time_start_volume >= time_exit_volume)
          return;

        float time_curr = time_start_volume;
        int3 g = getVoxel (ray_start + ray_dir * time_curr);
        g.x = max (0, min (g.x, VOLUME_X - 1));
        g.y = max (0, min (g.y, VOLUME_Y - 1));
        g.z = max (0, min (g.z, VOLUME_Z - 1));

        float tsdf = readTsdf (g.x, g.y, g.z);

        //infinite loop guard
        const float max_time = 3 * (volume_size.x + volume_size.y + volume_size.z);

        for (; time_curr < max_time; time_curr += time_step)
        {
          float tsdf_prev = tsdf;

          int3 g = getVoxel (  ray_start + ray_dir * (time_curr + time_step)  );
          if (!checkInds (g))
            break;

          tsdf = readTsdf (g.x, g.y, g.z);

          if (tsdf_prev < 0.f && tsdf > 0.f)
            break;

          if (tsdf_prev > 0.f && tsdf < 0.f)           //zero crossing
          {
            float Ftdt = interpolateTrilineary (ray_start, ray_dir, time_curr + time_step);
            if (isnan (Ftdt))
              break;

            float Ft = interpolateTrilineary (ray_start, ray_dir, time_curr);
            if (isnan (Ft))
              break;

            //float Ts = time_curr - time_step * Ft/(Ftdt - Ft);
            float Ts = time_curr - time_step * Ft / (Ftdt - Ft);

            float3 vetex_found = ray_start + ray_dir * Ts;

            vmap.ptr (y       )[x] = vetex_found.x;
            vmap.ptr (y + rows)[x] = vetex_found.y;
            vmap.ptr (y + 2 * rows)[x] = vetex_found.z;

            int3 g = getVoxel ( ray_start + ray_dir * time_curr );
            if (g.x > 1 && g.y > 1 && g.z > 1 && g.x < VOLUME_X - 2 && g.y < VOLUME_Y - 2 && g.z < VOLUME_Z - 2)
            {
              float3 t;
              float3 n;

              t = vetex_found;
              t.x += cell_size.x;
              float Fx1 = interpolateTrilineary (t);

              t = vetex_found;
              t.x -= cell_size.x;
              float Fx2 = interpolateTrilineary (t);

              n.x = (Fx1 - Fx2);

              t = vetex_found;
              t.y += cell_size.y;
              float Fy1 = interpolateTrilineary (t);

              t = vetex_found;
              t.y -= cell_size.y;
              float Fy2 = interpolateTrilineary (t);

              n.y = (Fy1 - Fy2);

              t = vetex_found;
              t.z += cell_size.z;
              float Fz1 = interpolateTrilineary (t);

              t = vetex_found;
              t.z -= cell_size.z;
              float Fz2 = interpolateTrilineary (t);

              n.z = (Fz1 - Fz2);

              n = normalized (n);

              nmap.ptr (y       )[x] = n.x;
              nmap.ptr (y + rows)[x] = n.y;
              nmap.ptr (y + 2 * rows)[x] = n.z;
            }
            break;
          }

        }          /* for(;;)  */
      }
    };

    __global__ void
    rayCastKernel (const RayCaster rc) {
      rc ();
    }
  }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::raycast (const Intr& intr, const Mat33& Rcurr, const float3& tcurr, 
                      float tranc_dist, const float3& volume_size,
                      const PtrStep<short2>& volume, MapArr& vmap, MapArr& nmap)
{
  RayCaster rc;

  rc.Rcurr = Rcurr;
  rc.tcurr = tcurr;

  rc.time_step = tranc_dist * 0.8f;

  rc.volume_size = volume_size;

  rc.cell_size.x = volume_size.x / VOLUME_X;
  rc.cell_size.y = volume_size.y / VOLUME_Y;
  rc.cell_size.z = volume_size.z / VOLUME_Z;

  rc.cols = vmap.cols ();
  rc.rows = vmap.rows () / 3;

  rc.intr = intr;

  rc.volume = volume;
  rc.vmap = vmap;
  rc.nmap = nmap;

  dim3 block (RayCaster::CTA_SIZE_X, RayCaster::CTA_SIZE_Y);
  dim3 grid (divUp (rc.cols, block.x), divUp (rc.rows, block.y));

  rayCastKernel<<<grid, block>>>(rc);
  cudaSafeCall (hipGetLastError ());
  //cudaSafeCall(hipDeviceSynchronize());
}

