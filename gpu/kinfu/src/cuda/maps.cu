#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

using namespace pcl::device;
using namespace pcl::gpu;

namespace pcl
{
  namespace device
  {
    __global__ void
    computeVmapKernel (const PtrStepSz<unsigned short> depth, PtrStep<float> vmap, float fx_inv, float fy_inv, float cx, float cy)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u < depth.cols && v < depth.rows)
      {
        float z = depth.ptr (v)[u] / 1000.f; // load and convert: mm -> meters

        if (z != 0)
        {
          float vx = z * (u - cx) * fx_inv;
          float vy = z * (v - cy) * fy_inv;
          float vz = z;

          vmap.ptr (v                 )[u] = vx;
          vmap.ptr (v + depth.rows    )[u] = vy;
          vmap.ptr (v + depth.rows * 2)[u] = vz;
        }
        else
          vmap.ptr (v)[u] = std::numeric_limits<float>::quiet_NaN ();

      }
    }

    __global__ void
    computeNmapKernel (int rows, int cols, const PtrStep<float> vmap, PtrStep<float> nmap)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u >= cols || v >= rows)
        return;

      if (u == cols - 1 || v == rows - 1)
      {
        nmap.ptr (v)[u] = std::numeric_limits<float>::quiet_NaN ();
        return;
      }

      float3 v00, v01, v10;
      v00.x = vmap.ptr (v  )[u];
      v01.x = vmap.ptr (v  )[u + 1];
      v10.x = vmap.ptr (v + 1)[u];

      if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x))
      {
        v00.y = vmap.ptr (v + rows)[u];
        v01.y = vmap.ptr (v + rows)[u + 1];
        v10.y = vmap.ptr (v + 1 + rows)[u];

        v00.z = vmap.ptr (v + 2 * rows)[u];
        v01.z = vmap.ptr (v + 2 * rows)[u + 1];
        v10.z = vmap.ptr (v + 1 + 2 * rows)[u];

        float3 r = normalized (cross (v01 - v00, v10 - v00));

        nmap.ptr (v       )[u] = r.x;
        nmap.ptr (v + rows)[u] = r.y;
        nmap.ptr (v + 2 * rows)[u] = r.z;
      }
      else
        nmap.ptr (v)[u] = std::numeric_limits<float>::quiet_NaN ();
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::createVMap (const Intr& intr, const DepthMap& depth, MapArr& vmap)
{
  vmap.create (depth.rows () * 3, depth.cols ());

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (depth.cols (), block.x);
  grid.y = divUp (depth.rows (), block.y);

  float fx = intr.fx, cx = intr.cx;
  float fy = intr.fy, cy = intr.cy;

  computeVmapKernel<<<grid, block>>>(depth, vmap, 1.f / fx, 1.f / fy, cx, cy);
  cudaSafeCall (hipGetLastError ());
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::createNMap (const MapArr& vmap, MapArr& nmap)
{
  nmap.create (vmap.rows (), vmap.cols ());

  int rows = vmap.rows () / 3;
  int cols = vmap.cols ();

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  computeNmapKernel<<<grid, block>>>(rows, cols, vmap, nmap);
  cudaSafeCall (hipGetLastError ());
}

namespace pcl
{
  namespace device
  {
    __global__ void
    tranformMapsKernel (int rows, int cols, const PtrStep<float> vmap_src, const PtrStep<float> nmap_src,
                        const Mat33 Rmat, const float3 tvec, PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      const float qnan = std::numeric_limits<float>::quiet_NaN ();

      if (x < cols && y < rows)
      {
        //vertices
        float3 vsrc, vdst = make_float3 (qnan, qnan, qnan);
        vsrc.x = vmap_src.ptr (y)[x];

        if (!isnan (vsrc.x))
        {
          vsrc.y = vmap_src.ptr (y + rows)[x];
          vsrc.z = vmap_src.ptr (y + 2 * rows)[x];

          vdst = Rmat * vsrc + tvec;

          vmap_dst.ptr (y + rows)[x] = vdst.y;
          vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
        }

        vmap_dst.ptr (y)[x] = vdst.x;

        //normals
        float3 nsrc, ndst = make_float3 (qnan, qnan, qnan);
        nsrc.x = nmap_src.ptr (y)[x];

        if (!isnan (nsrc.x))
        {
          nsrc.y = nmap_src.ptr (y + rows)[x];
          nsrc.z = nmap_src.ptr (y + 2 * rows)[x];

          ndst = Rmat * nsrc;

          nmap_dst.ptr (y + rows)[x] = ndst.y;
          nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
      }
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::tranformMaps (const MapArr& vmap_src, const MapArr& nmap_src,
                           const Mat33& Rmat, const float3& tvec,
                           MapArr& vmap_dst, MapArr& nmap_dst)
{
  int cols = vmap_src.cols ();
  int rows = vmap_src.rows () / 3;

  vmap_dst.create (rows * 3, cols);
  nmap_dst.create (rows * 3, cols);

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  tranformMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst);
  cudaSafeCall (hipGetLastError ());

  cudaSafeCall (hipDeviceSynchronize ());
}

namespace pcl
{
  namespace device
  {
    template<bool normalize>
    __global__ void
    resizeMapKernel (int drows, int dcols, int srows, const PtrStep<float> input, PtrStep<float> output)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= dcols || y >= drows)
        return;

      const float qnan = std::numeric_limits<float>::quiet_NaN ();

      int xs = x * 2;
      int ys = y * 2;

      float x00 = input.ptr (ys + 0)[xs + 0];
      float x01 = input.ptr (ys + 0)[xs + 1];
      float x10 = input.ptr (ys + 1)[xs + 0];
      float x11 = input.ptr (ys + 1)[xs + 1];

      if (isnan (x00) || isnan (x01) || isnan (x10) || isnan (x11))
      {
        output.ptr (y)[x] = qnan;
        return;
      }
      else
      {
        float3 n;

        n.x = (x00 + x01 + x10 + x11) / 4;

        float y00 = input.ptr (ys + srows + 0)[xs + 0];
        float y01 = input.ptr (ys + srows + 0)[xs + 1];
        float y10 = input.ptr (ys + srows + 1)[xs + 0];
        float y11 = input.ptr (ys + srows + 1)[xs + 1];

        n.y = (y00 + y01 + y10 + y11) / 4;

        float z00 = input.ptr (ys + 2 * srows + 0)[xs + 0];
        float z01 = input.ptr (ys + 2 * srows + 0)[xs + 1];
        float z10 = input.ptr (ys + 2 * srows + 1)[xs + 0];
        float z11 = input.ptr (ys + 2 * srows + 1)[xs + 1];

        n.z = (z00 + z01 + z10 + z11) / 4;

        if (normalize)
          n = normalized (n);

        output.ptr (y        )[x] = n.x;
        output.ptr (y + drows)[x] = n.y;
        output.ptr (y + 2 * drows)[x] = n.z;
      }
    }

    template<bool normalize>
    void
    resizeMap (const MapArr& input, MapArr& output)
    {
      int in_cols = input.cols ();
      int in_rows = input.rows () / 3;

      int out_cols = in_cols / 2;
      int out_rows = in_rows / 2;

      output.create (out_rows * 3, out_cols);

      dim3 block (32, 8);
      dim3 grid (divUp (out_cols, block.x), divUp (out_rows, block.y));
      resizeMapKernel<normalize><< < grid, block>>>(out_rows, out_cols, in_rows, input, output);
      cudaSafeCall ( hipGetLastError () );
      cudaSafeCall (hipDeviceSynchronize ());
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::resizeVMap (const MapArr& input, MapArr& output)
{
  resizeMap<false>(input, output);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::resizeNMap (const MapArr& input, MapArr& output)
{
  resizeMap<true>(input, output);
}

namespace pcl
{
  namespace device
  {

    template<typename T>
    __global__ void
    convertMapKernel (int rows, int cols, const PtrStep<float> map, PtrStep<T> output)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= cols || y >= rows)
        return;

      const float qnan = std::numeric_limits<float>::quiet_NaN ();

      T t;
      t.x = map.ptr (y)[x];
      if (!isnan (t.x))
      {
        t.y = map.ptr (y + rows)[x];
        t.z = map.ptr (y + 2 * rows)[x];
      }
      else
        t.y = t.z = qnan;

      output.ptr (y)[x] = t;
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<typename T> void
pcl::device::convert (const MapArr& vmap, DeviceArray2D<T>& output)
{
  int cols = vmap.cols ();
  int rows = vmap.rows () / 3;

  output.create (rows, cols);

  dim3 block (32, 8);
  dim3 grid (divUp (cols, block.x), divUp (rows, block.y));

  convertMapKernel<T><< < grid, block>>>(rows, cols, vmap, output);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

template void pcl::device::convert (const MapArr& vmap, DeviceArray2D<float4>& output);
template void pcl::device::convert (const MapArr& vmap, DeviceArray2D<float8>& output);

namespace pcl
{
  namespace device
  {
    __global__ void
    mergePointNormalKernel (const float4* cloud, const float8* normals, PtrSz<float12> output)
    {
      int idx = threadIdx.x + blockIdx.x * blockDim.x;

      if (idx < output.size)
      {
        float4 p = cloud[idx];
        float8 n = normals[idx];

        float12 o;
        o.x = p.x;
        o.y = p.y;
        o.z = p.z;

        o.normal_x = n.x;
        o.normal_y = n.y;
        o.normal_z = n.z;

        output.data[idx] = o;
      }
    }
  }
}

void
pcl::device::mergePointNormal (const DeviceArray<float4>& cloud, const DeviceArray<float8>& normals, const DeviceArray<float12>& output)
{
  const int block = 256;
  int total = (int)output.size ();

  mergePointNormalKernel<<<divUp (total, block), block>>>(cloud, normals, output);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}
