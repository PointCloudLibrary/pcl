#include "hip/hip_runtime.h"
/*
* Software License Agreement (BSD License)
*
*  Copyright (c) 2011, Willow Garage, Inc.
*  All rights reserved.
*
*  Redistribution and use in source and binary forms, with or without
*  modification, are permitted provided that the following conditions
*  are met:
*
*   * Redistributions of source code must retain the above copyright
*     notice, this list of conditions and the following disclaimer.
*   * Redistributions in binary form must reproduce the above
*     copyright notice, this list of conditions and the following
*     disclaimer in the documentation and/or other materials provided
*     with the distribution.
*   * Neither the name of Willow Garage, Inc. nor the names of its
*     contributors may be used to endorse or promote products derived
*     from this software without specific prior written permission.
*
*  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
*  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
*  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
*  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
*  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
*  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
*  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
*  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
*  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
*  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
*  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
*  POSSIBILITY OF SUCH DAMAGE.
*
*  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
*/

#include "internal.hpp"

#include "pcl/gpu/utils/device/warp.hpp"
#include "utils/copygen.hpp"
#include "utils/boxutils.hpp"
#include "utils/scan_block.hpp"

#include "octree_iterator.hpp"

namespace pcl 
{
    namespace device
    {           
        using PointType = OctreeImpl::PointType;

        template<typename RadiusStrategy, typename FetchStrategy>
        struct Batch : public RadiusStrategy, public FetchStrategy
        {               
            const int *indices;
            PtrStep<float> points;            
            OctreeGlobalWithBox octree;

            int max_results;
            mutable int* output;
            mutable int* output_sizes;        
        };

        struct DirectQuery
        {
            PtrSz<PointType> queries;
            __device__ __forceinline__ float3 fetch(const int query_index) const
            {
                const PointType& q = queries.data[query_index];
                return make_float3(q.x, q.y, q.z);
            }
        };


        struct IndicesQuery : public DirectQuery
        {
            const int* queries_indices;
            __device__ __forceinline__ float3 fetch(const int query_index) const
            {
                const PointType& q = queries[queries_indices[query_index]];
                return make_float3(q.x, q.y, q.z);
            }
        };

        struct SharedRadius
        {
            float radius;
            __device__ __forceinline__ float getRadius(const int /*index*/) const { return radius; }
        };

        struct IndividualRadius
        {
            const float* radiuses;
            __device__ __forceinline__ float getRadius(const int index) const { return radiuses[index]; }
        };

        struct KernelPolicy
        {
            enum 
            {
                CTA_SIZE = 512,

                WARP_SIZE = 32,
                WARPS_COUNT = CTA_SIZE/WARP_SIZE,

                MAX_LEVELS_PLUS_ROOT = 11,

                CHECK_FLAG = 1 << 31
            };

            struct SmemStorage
            {                             
                volatile int per_warp_buffer[WARPS_COUNT];
                volatile int cta_buffer[CTA_SIZE];
            };                                
        };

        __shared__ KernelPolicy::SmemStorage storage;


        template<typename BatchType>
        struct Warp_radiusSearch
        {   
        public:                
            using OctreeIterator = OctreeIteratorDeviceNS;

            const BatchType& batch;
            OctreeIterator iterator;        

            int found_count;
            int query_index;        
            float3 query;
            float radius;

            __device__ __forceinline__ Warp_radiusSearch(const BatchType& batch_arg, const int query_index_arg)
                : batch(batch_arg), iterator(/**/batch.octree/*storage.paths*/), found_count(0), query_index(query_index_arg){}

            __device__ __forceinline__ void launch(bool active)
            {                                 
                if (active)
                {
                    query = batch.fetch(query_index);                    
                    radius = batch.getRadius(query_index);
                }
                else                
                    query_index = -1;

                while(__any_sync(0xFFFFFFFF, active))
                {                
                    int leaf = -1;                

                    if (active)
                        leaf = examineNode(iterator);             

                    processLeaf(leaf);                           

                    active = active && iterator.level >= 0 && found_count < batch.max_results;
                }            

                if (query_index != -1)
                    batch.output_sizes[query_index] = found_count;
            }    

        private:

            __device__ __forceinline__ int examineNode(OctreeIterator& iterator)
            {                        
                using namespace pcl::gpu;

                const int node_idx = *iterator;
                const int code = batch.octree.codes[node_idx];

                float3 node_minp = batch.octree.minp;
                float3 node_maxp = batch.octree.maxp;        
                calcBoundingBox(iterator.level, code, node_minp, node_maxp);

                //if true, take nothing, and go to next
                if (checkIfNodeOutsideSphere(node_minp, node_maxp, query, radius))
                {     
                    ++iterator;
                    return -1;                
                }

                if (checkIfNodeInsideSphere(node_minp, node_maxp, query, radius))
                {   
                    ++iterator;       
                    return node_idx; //return node to copy
                }                              

                //need to go to next level
                const int node = batch.octree.nodes[node_idx];
                const int children_mask = node & 0xFF;            
                const bool isLeaf = children_mask == 0;            

                if (isLeaf)
                {
                    ++iterator;
                    return (node_idx | KernelPolicy::CHECK_FLAG); // return node to check                                                              
                }

                //goto next level
                const int first = node >> 8;
                const int len   = __popc(children_mask);
                iterator.gotoNextLevel(first, len);                    
                return -1;
            };

            __device__ __forceinline__ void processLeaf(int leaf)
            {   
                int mask = __ballot_sync(0xFFFFFFFF, leaf != -1);            

                while(mask)
                {                
                    const unsigned int laneId = Warp::laneId();

                    int active_lane = __ffs(mask) - 1; //[0..31]

                    mask &= ~(1 << active_lane);              

                    //broadcast active_found_count                                
                    const int active_found_count = __shfl_sync(0xFFFFFFFF, found_count, active_lane);

                    const int node_idx = leaf & ~KernelPolicy::CHECK_FLAG;

                    //broadcast beg and end
                    int fbeg, fend;
                    if (active_lane == laneId)
                    {
                      fbeg = batch.octree.begs[node_idx];
                      fend = batch.octree.ends[node_idx];
                    }
                    const int beg = __shfl_sync(0xFFFFFFFF, fbeg, active_lane);
                    const int end = __shfl_sync(0xFFFFFFFF, fend, active_lane);

                    //broadcast active_query_index
                    const int active_query_index = __shfl_sync(0xFFFFFFFF, query_index, active_lane);

                    int length = end - beg;

                    int *out = batch.output + active_query_index * batch.max_results + active_found_count;                    
                    const int length_left = batch.max_results - active_found_count;

                    const int test = __any_sync(0xFFFFFFFF, active_lane == laneId && (leaf & KernelPolicy::CHECK_FLAG));

                    if (test)
                    {
                        //broadcast warp_radius
                        const float radius2 = __shfl_sync(0xFFFFFFFF, radius * radius, active_lane);

                        //broadcast warp_query
                        const float3 active_query = make_float3(
                            __shfl_sync(0xFFFFFFFF, query.x, active_lane),
                            __shfl_sync(0xFFFFFFFF, query.y, active_lane),
                            __shfl_sync(0xFFFFFFFF, query.z, active_lane)
                        );

                        length = TestWarpKernel(beg, active_query, radius2, length, out, length_left);
                    }
                    else
                    {                            
                        length = min(length, length_left);                        
                        Warp::copy(batch.indices + beg, batch.indices + beg + length, out);
                    }

                    if (active_lane == laneId)
                        found_count += length;
                }            
            }    

            __device__ __forceinline__ int TestWarpKernel(const int beg, const float3& active_query, const float radius2, const int length, int* out, const int length_left)
            {                        
                unsigned int idx = Warp::laneId();
                const int last_threadIdx = threadIdx.x - idx + 31;            

                int total_new = 0;

                for(;;)
                {                
                    int take = 0;

                    if (idx < length)
                    {                                                                                                            
                        const float dx = batch.points.ptr(0)[beg + idx] - active_query.x;
                        const float dy = batch.points.ptr(1)[beg + idx] - active_query.y;
                        const float dz = batch.points.ptr(2)[beg + idx] - active_query.z;

                        const float d2 = dx * dx + dy * dy + dz * dz;

                        if (d2 < radius2)
                            take = 1;
                    }

                    storage.cta_buffer[threadIdx.x] = take;

                    const int offset = scan_warp<exclusive>(storage.cta_buffer);

                    //ensure that we copy
                    const bool out_of_bounds = (offset + total_new) >= length_left;                              

                    if (take && !out_of_bounds)
                        out[offset] = batch.indices[beg + idx];

                    const int new_nodes = storage.cta_buffer[last_threadIdx];

                    idx += Warp::STRIDE;

                    total_new += new_nodes;
                    out += new_nodes;                

                    if (__all_sync(0xFFFFFFFF, idx >= length) || __any_sync(0xFFFFFFFF, out_of_bounds) || total_new == length_left)
                        break;
                }
                return min(total_new, length_left);
            }
        };

        template<typename BatchType>
        __global__ void KernelRS(const BatchType batch) 
        {         
            const int query_index = blockIdx.x * blockDim.x + threadIdx.x;

            const bool active = query_index < batch.queries.size;

            if (__all_sync(0xFFFFFFFF, active == false)) 
                return;

            Warp_radiusSearch<BatchType> search(batch, query_index);
            search.launch(active); 
        }
    }
}

template<typename BatchType>
void pcl::device::OctreeImpl::radiusSearchEx(BatchType& batch, const Queries& queries, NeighborIndices& results)
{
    batch.indices = indices;
    batch.octree = octreeGlobal;

    batch.max_results = results.max_elems;
    batch.output = results.data;                
    batch.output_sizes = results.sizes;

    batch.points = points_sorted;
    
    
    cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(KernelRS<BatchType>), hipFuncCachePreferL1) );

    int block = KernelPolicy::CTA_SIZE;
    int grid = divUp((int)batch.queries.size, block);

    KernelRS<<<grid, block>>>(batch);
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );
}


void pcl::device::OctreeImpl::radiusSearch(const Queries& queries, float radius, NeighborIndices& results)
{        
    using BatchType = Batch<SharedRadius, DirectQuery>;

    BatchType batch;
    batch.radius = radius;
    batch.queries = queries;
    radiusSearchEx(batch, queries, results);              
}

void pcl::device::OctreeImpl::radiusSearch(const Queries& queries, const Radiuses& radiuses, NeighborIndices& results)
{
    using BatchType = Batch<IndividualRadius, DirectQuery>;

    BatchType batch;
    batch.radiuses = radiuses;
    batch.queries = queries;
    radiusSearchEx(batch, queries, results);              
}

void pcl::device::OctreeImpl::radiusSearch(const Queries& queries, const Indices& indices, float radius, NeighborIndices& results)
{
    using BatchType = Batch<SharedRadius, IndicesQuery>;

    BatchType batch;
    batch.radius = radius;
    
    batch.queries = queries;
    batch.queries_indices = indices;
    batch.queries.size = indices.size();

    radiusSearchEx(batch, queries, results);        
}
