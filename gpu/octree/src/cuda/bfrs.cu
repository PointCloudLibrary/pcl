#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2011, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */


#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>

#include "internal.hpp"

#include "hip/hip_runtime.h"

using namespace std;
using namespace thrust;

namespace pcl
{    
    namespace device
    {
        struct InSphere
        {    
            float x_, y_, z_, radius2_;
            InSphere(float x, float y, float z, float radius) : x_(x), y_(y), z_(z), radius2_(radius * radius) {}

            __device__ __host__ __forceinline__ bool operator()(const float3& point) const
            {
                float dx = point.x - x_;
                float dy = point.y - y_;
                float dz = point.z - z_;

                return (dx * dx + dy * dy + dz * dz) < radius2_;
            }

            __device__ __host__ __forceinline__ bool operator()(const float4& point) const
            {
                return (*this)(make_float3(point.x, point.y, point.z));                
            }
        };
    }
}

void pcl::device::bruteForceRadiusSearch(const OctreeImpl::PointCloud& cloud, const OctreeImpl::PointType& query, float radius, DeviceArray<int>& result, DeviceArray<int>& buffer)
{   
    using PointType = OctreeImpl::PointType;

    if (buffer.size() < cloud.size())
        buffer.create(cloud.size());

    InSphere cond(query.x, query.y, query.z, radius);

    device_ptr<const PointType> cloud_ptr((const PointType*)cloud.ptr());
    device_ptr<int> res_ptr(buffer.ptr());
    
    counting_iterator<int> first(0);
    counting_iterator<int> last = first + cloud.size();
    
    //main bottle neck is a kernel call overhead/allocs
    //work time for 871k points ~0.8ms
    int count = (int)(thrust::copy_if(first, last, cloud_ptr, res_ptr, cond) - res_ptr);
    result = DeviceArray<int>(buffer.ptr(), count);
}
