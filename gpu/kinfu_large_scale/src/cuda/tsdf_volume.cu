#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"
//#include <boost/graph/buffer_concepts.hpp>

namespace pcl
{
  namespace device
  {
    namespace kinfuLS
    {
      template<typename T>
      __global__ void
      initializeVolume (PtrStep<T> volume)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
        
        
        if (x < VOLUME_X && y < VOLUME_Y)
        {
            T *pos = volume.ptr(y) + x;
            int z_step = VOLUME_Y * volume.step / sizeof(*pos);

  #pragma unroll
            for(int z = 0; z < VOLUME_Z; ++z, pos+=z_step)
              pack_tsdf (0.f, 0, *pos);
        }
      }
      
          template<typename T>
      __global__ void
      clearSliceKernel (PtrStep<T> volume, pcl::gpu::kinfuLS::tsdf_buffer buffer, int3 minBounds, int3 maxBounds)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
            
        //compute relative indices
        int idX, idY;
        
        if(x <= minBounds.x)
          idX = x + buffer.voxels_size.x;
        else
          idX = x;
        
        if(y <= minBounds.y)
          idY = y + buffer.voxels_size.y;
        else
          idY = y;	 
                
        
        if ( x < buffer.voxels_size.x && y < buffer.voxels_size.y)
        {
            if( (idX >= minBounds.x && idX <= maxBounds.x) || (idY >= minBounds.y && idY <= maxBounds.y) )
            {
                // BLACK ZONE => clear on all Z values
          
                ///Pointer to the first x,y,0			
                T *pos = volume.ptr(y) + x;
                
                ///Get the step on Z
                int z_step = buffer.voxels_size.y * volume.step / sizeof(*pos);
                                    
                ///Get the size of the whole TSDF memory
                int size = buffer.tsdf_memory_end - buffer.tsdf_memory_start + 1;
                                  
                ///Move along z axis
      #pragma unroll
                for(int z = 0; z < buffer.voxels_size.z; ++z, pos+=z_step)
                {
                  ///If we went outside of the memory, make sure we go back to the beginning of it
                  if(pos > buffer.tsdf_memory_end)
                    pos = pos - size;
                  
                  if (pos >= buffer.tsdf_memory_start && pos <= buffer.tsdf_memory_end) // quickfix for http://dev.pointclouds.org/issues/894
                    pack_tsdf (0.f, 0, *pos);
                }
            }
            else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
            {
              
                ///RED ZONE  => clear only appropriate Z
              
                ///Pointer to the first x,y,0
                T *pos = volume.ptr(y) + x;
                
                ///Get the step on Z
                int z_step = buffer.voxels_size.y * volume.step / sizeof(*pos);
                            
                ///Get the size of the whole TSDF memory 
                int size = buffer.tsdf_memory_end - buffer.tsdf_memory_start + 1;
                              
                ///Move pointer to the Z origin
                pos+= minBounds.z * z_step;
                
                ///If the Z offset is negative, we move the pointer back
                if(maxBounds.z < 0)
                  pos += maxBounds.z * z_step;
                  
                ///We make sure that we are not already before the start of the memory
                if(pos < buffer.tsdf_memory_start)
                    pos = pos + size;

                int nbSteps = abs(maxBounds.z);
                
            #pragma unroll				
                for(int z = 0; z < nbSteps; ++z, pos+=z_step)
                {
                  ///If we went outside of the memory, make sure we go back to the beginning of it
                  if(pos > buffer.tsdf_memory_end)
                    pos = pos - size;
                  
                  if (pos >= buffer.tsdf_memory_start && pos <= buffer.tsdf_memory_end) // quickfix for http://dev.pointclouds.org/issues/894
                    pack_tsdf (0.f, 0, *pos);
                }
            } //else /* if( idX > maxBounds.x && idY > maxBounds.y)*/
        } // if ( x < VOLUME_X && y < VOLUME_Y)
      } // clearSliceKernel
   
      void
      initVolume (PtrStep<short2> volume)
      {
        dim3 block (16, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (VOLUME_X, block.x);      
        grid.y = divUp (VOLUME_Y, block.y);

        initializeVolume<<<grid, block>>>(volume);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }
    }
  }
}


namespace pcl
{
  namespace device
  {
    namespace kinfuLS
    {
      struct Tsdf
      {
        enum
        {
          CTA_SIZE_X = 32, CTA_SIZE_Y = 8,
          MAX_WEIGHT = 1 << 7
        };

        mutable PtrStep<short2> volume;
        float3 cell_size;

        Intr intr;

        Mat33 Rcurr_inv;
        float3 tcurr;

        PtrStepSz<ushort> depth_raw; //depth in mm

        float tranc_dist_mm;

        __device__ __forceinline__ float3
        getVoxelGCoo (int x, int y, int z) const
        {
          float3 coo = make_float3 (x, y, z);
          coo += 0.5f;         //shift to cell center;

          coo.x *= cell_size.x;
          coo.y *= cell_size.y;
          coo.z *= cell_size.z;

          return coo;
        }

        __device__ __forceinline__ void
        operator () () const
        {
          int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
          int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

          if (x >= VOLUME_X || y >= VOLUME_Y)
            return;

          short2 *pos = volume.ptr (y) + x;
          int elem_step = volume.step * VOLUME_Y / sizeof(*pos);

          for (int z = 0; z < VOLUME_Z; ++z, pos += elem_step)
          {
            float3 v_g = getVoxelGCoo (x, y, z);            //3 // p

            //transform to curr cam coo space
            float3 v = Rcurr_inv * (v_g - tcurr);           //4

            int2 coo;           //project to current cam
            coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
            coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);

            if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)           //6
            {
              int Dp = depth_raw.ptr (coo.y)[coo.x];

              if (Dp != 0)
              {
                float xl = (coo.x - intr.cx) / intr.fx;
                float yl = (coo.y - intr.cy) / intr.fy;
                float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

                float sdf = 1000 * norm (tcurr - v_g) * lambda_inv - Dp; //mm

                sdf *= (-1);

                if (sdf >= -tranc_dist_mm)
                {
                  float tsdf = fmin (1.f, sdf / tranc_dist_mm);

                  int weight_prev;
                  float tsdf_prev;

                  //read and unpack
                  unpack_tsdf (*pos, tsdf_prev, weight_prev);

                  const int Wrk = 1;

                  float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                  int weight_new = min (weight_prev + Wrk, MAX_WEIGHT);

                  pack_tsdf (tsdf_new, weight_new, *pos);
                }
              }
            }
          }
        }
      };

      __global__ void
      integrateTsdfKernel (const Tsdf tsdf) {
        tsdf ();
      }

      __global__ void
      tsdf2 (PtrStep<short2> volume, const float tranc_dist_mm, const Mat33 Rcurr_inv, float3 tcurr,
            const Intr intr, const PtrStepSz<ushort> depth_raw, const float3 cell_size)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
          return;

        short2 *pos = volume.ptr (y) + x;
        int elem_step = volume.step * VOLUME_Y / sizeof(short2);

        float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
        float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
        float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

        float v_x = Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z;
        float v_y = Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z;
        float v_z = Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z;

  //#pragma unroll
        for (int z = 0; z < VOLUME_Z; ++z)
        {
          float3 vr;
          vr.x = v_g_x;
          vr.y = v_g_y;
          vr.z = (v_g_z + z * cell_size.z);

          float3 v;
          v.x = v_x + Rcurr_inv.data[0].z * z * cell_size.z;
          v.y = v_y + Rcurr_inv.data[1].z * z * cell_size.z;
          v.z = v_z + Rcurr_inv.data[2].z * z * cell_size.z;

          int2 coo;         //project to current cam
          coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
          coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);


          if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)         //6
          {
            int Dp = depth_raw.ptr (coo.y)[coo.x]; //mm

            if (Dp != 0)
            {
              float xl = (coo.x - intr.cx) / intr.fx;
              float yl = (coo.y - intr.cy) / intr.fy;
              float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

              float sdf = Dp - norm (vr) * lambda_inv * 1000; //mm


              if (sdf >= -tranc_dist_mm)
              {
                float tsdf = fmin (1.f, sdf / tranc_dist_mm);

                int weight_prev;
                float tsdf_prev;

                //read and unpack
                unpack_tsdf (*pos, tsdf_prev, weight_prev);

                const int Wrk = 1;

                float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

                pack_tsdf (tsdf_new, weight_new, *pos);
              }
            }
          }
          pos += elem_step;
        }       /* for(int z = 0; z < VOLUME_Z; ++z) */
      }      /* __global__ */

      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      void
      integrateTsdfVolume (const PtrStepSz<ushort>& depth_raw, const Intr& intr, const float3& volume_size,
                                        const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, 
                                        PtrStep<short2> volume)
      {
        Tsdf tsdf;

        tsdf.volume = volume;  
        tsdf.cell_size.x = volume_size.x / VOLUME_X;
        tsdf.cell_size.y = volume_size.y / VOLUME_Y;
        tsdf.cell_size.z = volume_size.z / VOLUME_Z;
        
        tsdf.intr = intr;

        tsdf.Rcurr_inv = Rcurr_inv;
        tsdf.tcurr = tcurr;
        tsdf.depth_raw = depth_raw;

        tsdf.tranc_dist_mm = tranc_dist*1000; //mm

        dim3 block (Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
        dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

      #if 0
        //tsdf2<<<grid, block>>>(volume, tranc_dist, Rcurr_inv, tcurr, intr, depth_raw, tsdf.cell_size);
        integrateTsdfKernel<<<grid, block>>>(tsdf);
      #endif
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }
    }
  }
}

namespace pcl
{
  namespace device
  {
    namespace kinfuLS
    {
      __global__ void
      scaleDepth (const PtrStepSz<ushort> depth, PtrStep<float> scaled, const Intr intr)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= depth.cols || y >= depth.rows)
          return;

        int Dp = depth.ptr (y)[x];

        float xl = (x - intr.cx) / intr.fx;
        float yl = (y - intr.cy) / intr.fy;
        float lambda = sqrtf (xl * xl + yl * yl + 1);

        scaled.ptr (y)[x] = Dp * lambda/1000.f; //meters
      }

      __global__ void
      tsdf23 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
              const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size, const pcl::gpu::kinfuLS::tsdf_buffer buffer)
      {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= buffer.voxels_size.x || y >= buffer.voxels_size.y)
          return;

        float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
        float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
        float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

        float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

        float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
        float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
        float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

        float z_scaled = 0;

        float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
        float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

        float tranc_dist_inv = 1.0f / tranc_dist;

        short2* pos = volume.ptr (y) + x;
        
        // shift the pointer to relative indices
        shift_tsdf_pointer(&pos, buffer);
        
        int elem_step = volume.step * buffer.voxels_size.y / sizeof(short2);

  //#pragma unroll
        for (int z = 0; z < buffer.voxels_size.z;
            ++z,
            v_g_z += cell_size.z,
            z_scaled += cell_size.z,
            v_x += Rcurr_inv_0_z_scaled,
            v_y += Rcurr_inv_1_z_scaled,
            pos += elem_step)
        {
          
          // As the pointer is incremented in the for loop, we have to make sure that the pointer is never outside the memory
          if(pos > buffer.tsdf_memory_end)
            pos -= (buffer.tsdf_memory_end - buffer.tsdf_memory_start + 1);
          
          float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
          if (inv_z < 0)
              continue;

          // project to current cam
          int2 coo =
          {
            __float2int_rn (v_x * inv_z + intr.cx),
            __float2int_rn (v_y * inv_z + intr.cy)
          };

          if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
          {
            float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

            float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

            if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
            {
              float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

              //read and unpack
              float tsdf_prev;
              int weight_prev;
              unpack_tsdf (*pos, tsdf_prev, weight_prev);

              const int Wrk = 1;

              float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
              int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

              pack_tsdf (tsdf_new, weight_new, *pos);
            }
          }
        }       // for(int z = 0; z < VOLUME_Z; ++z)
      }      // __global__

      __global__ void
      tsdf23normal_hack (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
                    const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
      {
          int x = threadIdx.x + blockIdx.x * blockDim.x;
          int y = threadIdx.y + blockIdx.y * blockDim.y;

          if (x >= VOLUME_X || y >= VOLUME_Y)
              return;

          const float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
          const float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
          float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

          float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

          float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
          float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
          float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

          float z_scaled = 0;

          float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
          float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

          float tranc_dist_inv = 1.0f / tranc_dist;

          short2* pos = volume.ptr (y) + x;
          int elem_step = volume.step * VOLUME_Y / sizeof(short2);

          //#pragma unroll
          for (int z = 0; z < VOLUME_Z;
              ++z,
              v_g_z += cell_size.z,
              z_scaled += cell_size.z,
              v_x += Rcurr_inv_0_z_scaled,
              v_y += Rcurr_inv_1_z_scaled,
              pos += elem_step)
          {
              float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
              if (inv_z < 0)
                  continue;

              // project to current cam
              int2 coo =
              {
                  __float2int_rn (v_x * inv_z + intr.cx),
                  __float2int_rn (v_y * inv_z + intr.cy)
              };

              if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
              {
                  float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

                  float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

                  if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
                  {
                      float tsdf = fmin (1.0f, sdf * tranc_dist_inv);                                              

                      bool integrate = true;
                      if ((x > 0 &&  x < VOLUME_X-2) && (y > 0 && y < VOLUME_Y-2) && (z > 0 && z < VOLUME_Z-2))
                      {
                          const float qnan = numeric_limits<float>::quiet_NaN();
                          float3 normal = make_float3(qnan, qnan, qnan);

                          float Fn, Fp;
                          int Wn = 0, Wp = 0;
                          unpack_tsdf (*(pos + elem_step), Fn, Wn);
                          unpack_tsdf (*(pos - elem_step), Fp, Wp);

                          if (Wn > 16 && Wp > 16) 
                              normal.z = (Fn - Fp)/cell_size.z;

                          unpack_tsdf (*(pos + volume.step/sizeof(short2) ), Fn, Wn);
                          unpack_tsdf (*(pos - volume.step/sizeof(short2) ), Fp, Wp);

                          if (Wn > 16 && Wp > 16) 
                              normal.y = (Fn - Fp)/cell_size.y;

                          unpack_tsdf (*(pos + 1), Fn, Wn);
                          unpack_tsdf (*(pos - 1), Fp, Wp);

                          if (Wn > 16 && Wp > 16) 
                              normal.x = (Fn - Fp)/cell_size.x;

                          if (normal.x != qnan && normal.y != qnan && normal.z != qnan)
                          {
                              float norm2 = dot(normal, normal);
                              if (norm2 >= 1e-10)
                              {
                                  normal *= rsqrt(norm2);

                                  float nt = v_g_x * normal.x + v_g_y * normal.y + v_g_z * normal.z;
                                  float cosine = nt * rsqrt(v_g_x * v_g_x + v_g_y * v_g_y + v_g_z * v_g_z);

                                  if (cosine < 0.5)
                                      integrate = false;
                              }
                          }
                      }

                      if (integrate)
                      {
                          //read and unpack
                          float tsdf_prev;
                          int weight_prev;
                          unpack_tsdf (*pos, tsdf_prev, weight_prev);

                          const int Wrk = 1;

                          float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                          int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

                          pack_tsdf (tsdf_new, weight_new, *pos);
                      }
                  }
              }
          }       // for(int z = 0; z < VOLUME_Z; ++z)
      }      // __global__

      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      void
      integrateTsdfVolume (const PtrStepSz<ushort>& depth, const Intr& intr,
                                        const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, 
                                        float tranc_dist,
                                        PtrStep<short2> volume, const pcl::gpu::kinfuLS::tsdf_buffer* buffer, DeviceArray2D<float>& depthScaled)
      {
        depthScaled.create (depth.rows, depth.cols);

        dim3 block_scale (32, 8);
        dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

        //scales depth along ray and converts mm -> meters. 
        scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
        cudaSafeCall ( hipGetLastError () );

        float3 cell_size;
        cell_size.x = volume_size.x / buffer->voxels_size.x;
        cell_size.y = volume_size.y / buffer->voxels_size.y;
        cell_size.z = volume_size.z / buffer->voxels_size.z;

        //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
        dim3 block (16, 16);
        dim3 grid (divUp (buffer->voxels_size.x, block.x), divUp (buffer->voxels_size.y, block.y));

        tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, *buffer);    
        //tsdf23normal_hack<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);

        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());
      }

      /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      void 
      clearTSDFSlice (PtrStep<short2> volume, pcl::gpu::kinfuLS::tsdf_buffer* buffer, int shiftX, int shiftY, int shiftZ)
      {
        int newX = buffer->origin_GRID.x + shiftX;
        int newY = buffer->origin_GRID.y + shiftY;

        int3 minBounds, maxBounds;
        
        //X
        if(newX >= 0)
        {
        minBounds.x = buffer->origin_GRID.x;
        maxBounds.x = newX;    
        }
        else
        {
        minBounds.x = newX + buffer->voxels_size.x; 
        maxBounds.x = buffer->origin_GRID.x + buffer->voxels_size.x;
        }
        
        if(minBounds.x > maxBounds.x)
        std::swap(minBounds.x, maxBounds.x);
          
      
        //Y
        if(newY >= 0)
        {
        minBounds.y = buffer->origin_GRID.y;
        maxBounds.y = newY;
        }
        else
        {
        minBounds.y = newY + buffer->voxels_size.y; 
        maxBounds.y = buffer->origin_GRID.y + buffer->voxels_size.y;
        }
        
        if(minBounds.y > maxBounds.y)
        std::swap(minBounds.y, maxBounds.y);
        
        //Z
        minBounds.z = buffer->origin_GRID.z;
        maxBounds.z = shiftZ;
      
        // call kernel
        dim3 block (32, 16);
        dim3 grid (1, 1, 1);
        grid.x = divUp (buffer->voxels_size.x, block.x);      
        grid.y = divUp (buffer->voxels_size.y, block.y);
        
        clearSliceKernel<<<grid, block>>>(volume, *buffer, minBounds, maxBounds);
        cudaSafeCall ( hipGetLastError () );
        cudaSafeCall (hipDeviceSynchronize ());        
      }
    }
  }
}
