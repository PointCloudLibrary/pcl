#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "internal.h"
#include "device.h"
#include <limits>

#include <pcl/gpu/utils/device/limits.hpp>
#include <pcl/gpu/utils/device/algorithm.hpp>
#include <pcl/gpu/utils/device/warp.hpp>
#include <pcl/gpu/utils/device/static_check.hpp>
//#include <pcl/gpu/utils/device/funcattrib.hpp>
#include <pcl/gpu/utils/safe_call.hpp>

#include <thrust/tuple.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include "thrust/device_ptr.h"
#include <thrust/transform.h>
#include <thrust/sort.h>
#include <thrust/transform_scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/unique.h>
#include <thrust/gather.h>

using namespace thrust;
using namespace std;

namespace pcl
{
  namespace device
  { 	  
	  __global__ void size_check() { Static<sizeof(uint64_type) == 8>::check(); };
	  
	  template<bool use_max>
	  struct IndOp
	  {
		  __device__ __forceinline__ thrust::tuple<float, int> operator()(const thrust::tuple<float, int>& e1, const thrust::tuple<float, int>& e2) const
		  {	
			  thrust::tuple<float, int> res;
			  
			  if (use_max)
			    res.get<0>() = fmax(e1.get<0>(), e2.get<0>());			  			  
			  else
				res.get<0>() = fmin(e1.get<0>(), e2.get<0>());			  			  

			  res.get<1>()  = (res.get<0>() == e1.get<0>()) ? e1.get<1>() : e2.get<1>();
			  return res;			  
		  }		 
	  };

	  struct X
	  {			  
		  __device__ __forceinline__ 
		  thrust::tuple<float, int> 
		  operator()(const thrust::tuple<PointType, int>& in) const
		  {
			return thrust::tuple<float, int>(in.get<0>().x, in.get<1>());			  
		  }
	  };

	  struct Y
	  {			  
		  __device__ __forceinline__  float operator()(const PointType& in) const { return in.y; }
	  };

	  struct Z
	  {			  
		  __device__ __forceinline__  float operator()(const PointType& in) const { return in.z; }
	  };
		  
	  struct LineDist
	  {
		  float3 x1, x2;
		  LineDist(const PointType& p1, const PointType& p2) : x1(tr(p1)), x2(tr(p2)) {}
		  
		  __device__ __forceinline__
		  thrust::tuple<float, int> operator()(const thrust::tuple<PointType, int>& in) const
		  {			  
			  float3 x0 = tr(in.get<0>());

			  float dist = norm(cross(x0 - x1, x0 - x2))/norm(x1 - x2);			  
			  return thrust::tuple<float, int>(dist, in.get<1>());
		  }	      
	  };

	  struct PlaneDist
	  {		  
		  float3 x1, n;
		  PlaneDist(const PointType& p1, const PointType& p2, const PointType& p3) : x1(tr(p1))
		  {
			  float3 x2 = tr(p2), x3 = tr(p3);
              n = normalized(cross(x2 - x1, x3 - x1));
		  }
		  
		  __device__ __forceinline__
		  thrust::tuple<float, int> operator()(const thrust::tuple<PointType, int>& in) const
		  {
			  float3 x0 = tr(in.get<0>());
              float dist = fabs(dot(n, x0 - x1));
			  return thrust::tuple<float, int>(dist, in.get<1>());
		  }
	  };
	  
	  template<typename It, typename Unary, typename Init, typename Binary>
      int transform_reduce_index(It beg, It end, Unary unop, Init init, Binary binary)
	  {
	    counting_iterator<int> cbeg(0);
		counting_iterator<int> cend = cbeg + thrust::distance(beg, end);
			 		
	    thrust::tuple<float, int> t = transform_reduce( 
		  make_zip_iterator(thrust::make_tuple(beg, cbeg)), 
		  make_zip_iterator(thrust::make_tuple(end, cend)), 
		  unop, init, binary);
		
		return t.get<1>();
	  }

	  template<typename It, typename Unary>
      int transform_reduce_min_index(It beg, It end, Unary unop)
	  {
		thrust::tuple<float, int> min_tuple(std::numeric_limits<float>::max(), 0);
		return transform_reduce_index(beg, end, unop, min_tuple, IndOp<false>());
	  }

	  template<typename It, typename Unary>
      int transform_reduce_max_index(It beg, It end, Unary unop)
	  {
		thrust::tuple<float, int> max_tuple(std::numeric_limits<float>::min(), 0);
		return transform_reduce_index(beg, end, unop, max_tuple, IndOp<true>());
	  }	 
  }
}

pcl::device::PointStream::PointStream(const Cloud& cloud_) : cloud(cloud_)
{				
  cloud_size = cloud.size();
  facets_dists.create(cloud_size);
  perm.create(cloud_size);

  device_ptr<int> pbeg(perm.ptr());  
  thrust::sequence(pbeg, pbeg + cloud_size);
}

void pcl::device::PointStream::computeInitalSimplex()
{
  device_ptr<const PointType> beg(cloud.ptr());  
  device_ptr<const PointType> end = beg + cloud_size;
     
  int minx = transform_reduce_min_index(beg, end, X());
  int maxx = transform_reduce_max_index(beg, end, X());

  PointType p1 = *(beg + minx);
  PointType p2 = *(beg + maxx);
  	        
  int maxl = transform_reduce_max_index(beg, end, LineDist(p1, p2));

  PointType p3 = *(beg + maxl);
    
  int maxp = transform_reduce_max_index(beg, end, PlaneDist(p1, p2, p3));

  PointType p4 = *(beg + maxp);

  simplex.x1 = tr(p1);  simplex.x2 = tr(p2);  simplex.x3 = tr(p3);  simplex.x4 = tr(p4);
  simplex.i1 = minx;    simplex.i2 = maxx;    simplex.i3 = maxl;    simplex.i4 = maxp;

  float maxy = transform_reduce(beg, end, Y(), std::numeric_limits<float>::min(), maximum<float>()); 
  float miny = transform_reduce(beg, end, Y(), std::numeric_limits<float>::max(), minimum<float>()); 

  float maxz = transform_reduce(beg, end, Z(), std::numeric_limits<float>::min(), maximum<float>()); 
  float minz = transform_reduce(beg, end, Z(), std::numeric_limits<float>::max(), minimum<float>()); 
		  
  float dx = (p2.x - p1.x);
  float dy = (maxy - miny);
  float dz = (maxz - minz);

  cloud_diag = sqrt(dx*dx + dy*dy + dz*dz);

  simplex.p1 = compute_plane(simplex.x4, simplex.x2, simplex.x3, simplex.x1);
  simplex.p2 = compute_plane(simplex.x3, simplex.x1, simplex.x4, simplex.x2);
  simplex.p3 = compute_plane(simplex.x2, simplex.x1, simplex.x4, simplex.x3);
  simplex.p4 = compute_plane(simplex.x1, simplex.x2, simplex.x3, simplex.x4);  
}

namespace pcl
{
  namespace device
  {
    __global__ void init_fs(int i1, int i2, int i3, int i4, PtrStep<int> verts_inds)
	{	  	  	  
      *(int4*)verts_inds.ptr(0) = make_int4(i2, i1, i1, i1);
      *(int4*)verts_inds.ptr(1) = make_int4(i3, i3, i2, i2);
      *(int4*)verts_inds.ptr(2) = make_int4(i4, i4, i4, i3);
	}

  }
}

void pcl::device::FacetStream::setInitialFacets(const InitalSimplex& s)
{  
  init_fs<<<1, 1>>>(s.i1, s.i2, s.i3, s.i4, verts_inds);  
  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );  
  facet_count = 4;
}

///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
  namespace device
  {
	struct InitalClassify
	{
      float diag;
      float4 pl1, pl2, pl3, pl4;
	  	  
      InitalClassify(const float4& p1, const float4& p2, const float4& p3, const float4& p4, float diagonal) 
          : diag(diagonal), pl1(p1), pl2(p2), pl3(p3), pl4(p4)
	  {				
        pl1 *= compue_inv_normal_norm(pl1);
        pl2 *= compue_inv_normal_norm(pl2);
        pl3 *= compue_inv_normal_norm(pl3);
        pl4 *= compue_inv_normal_norm(pl4);
	  }
	  	  
	  __device__ __forceinline__
	  uint64_type 
	  operator()(const PointType& p) const
	  {                   
		  float4 x = p;
		  x.w = 1;

          float d0 = dot(pl1, x);
          float d1 = dot(pl2, x);
          float d2 = dot(pl3, x);
          float d3 = dot(pl4, x);

          float dists[] = { d0, d1, d2, d3 };
          int negs_inds[4];
          int neg_count = 0;
          
          int idx = numeric_limits<int>::max();
          float dist = 0;

          #pragma unroll
          for(int i = 0; i < 4; ++i)
            if (dists[i] < 0)
              negs_inds[neg_count++] = i;

          if (neg_count == 3)
          {
             int i1 = negs_inds[1];
             int i2 = negs_inds[2];
             
             int ir = fabs(dists[i1]) < fabs(dists[i2]) ? i2 : i1;
             negs_inds[1] = ir;
             --neg_count;
          }

          if (neg_count == 2)
          {
             int i1 = negs_inds[0];
             int i2 = negs_inds[1];
             
             int ir = fabs(dists[i1]) < fabs(dists[i2]) ? i2 : i1;
             negs_inds[0] = ir;
             --neg_count;              
          }

          if (neg_count == 1)
          {
            idx = negs_inds[0];
            dist = diag - fabs(dists[idx]); // to ensure that sorting order is inverse, i.e. distant points go first
          }

          //if (neg_count == 0)
          //  then internal point ==>> idx = INT_MAX

		  uint64_type res = idx;
		  res <<= 32;
		  return res + *reinterpret_cast<unsigned int*>(&dist);
	  }		
	};		

    __global__ void initalClassifyKernel(const InitalClassify ic, const PointType* points, int cloud_size, uint64_type* output) 
    { 
        int index = threadIdx.x + blockIdx.x * blockDim.x;

        if (index < cloud_size)              
          output[index] = ic(points[index]); 
    }
  }
}

void pcl::device::PointStream::initalClassify()
{        
  //thrust::device_ptr<const PointType> beg(cloud.ptr());
  //thrust::device_ptr<const PointType> end = beg + cloud_size;
  thrust::device_ptr<uint64_type> out(facets_dists.ptr());
  
  InitalClassify ic(simplex.p1, simplex.p2, simplex.p3, simplex.p4, cloud_diag);
  //thrust::transform(beg, end, out, ic);
  
  //printFuncAttrib(initalClassifyKernel);

  initalClassifyKernel<<<divUp(cloud_size, 256), 256>>>(ic, cloud, cloud_size, facets_dists);
  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );

  thrust::device_ptr<int> pbeg(perm.ptr());
  thrust::sort_by_key(out, out + cloud_size, pbeg);
}

///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
  namespace device
  {
    __device__ int new_cloud_size;    
	struct SearchFacetHeads
	{		
	  uint64_type *facets_dists;
	  int cloud_size;
	  int facet_count;
	  int *perm;
	  const PointType* points;

	  mutable int* head_points;
      //bool logger;
	
	  __device__ __forceinline__
	  void operator()(int facet) const
	  {			
		const uint64_type* b = facets_dists;
		const uint64_type* e = b + cloud_size;

        bool last_thread = facet == facet_count;

        int search_value = !last_thread ? facet : numeric_limits<int>::max();		
		int index = lower_bound(b, e, search_value, LessThanByFacet()) - b;			
        
        if (last_thread)
            new_cloud_size = index;
        else
        {
          bool not_found = index == cloud_size || (facet != (facets_dists[index] >> 32));

          head_points[facet] = not_found ? -1 : perm[index];		
        }
	  }
	};

    __global__ void searchFacetHeadsKernel(const SearchFacetHeads sfh)
    {
        int facet = threadIdx.x + blockDim.x * blockIdx.x;

        if (facet <= sfh.facet_count)
          sfh(facet);
    }
  }
}

int pcl::device::PointStream::searchFacetHeads(size_t facet_count, DeviceArray<int>& head_points)
{
	SearchFacetHeads sfh;

	sfh.facets_dists = facets_dists;
	sfh.cloud_size = (int)cloud_size;
	sfh.facet_count = (int)facet_count;
	sfh.perm = perm;
	sfh.points = cloud.ptr();
	sfh.head_points = head_points;  
	
    //thrust::counting_iterator<int> b(0);
    //thrust::counting_iterator<int> e = b + facet_count + 1;  	
    //thrust::for_each(b, e, sfh);

    searchFacetHeadsKernel<<<divUp(facet_count+1, 256), 256>>>(sfh);
    cudaSafeCall( hipGetLastError() );
    cudaSafeCall( hipDeviceSynchronize() );        

	int new_size;
	cudaSafeCall( hipMemcpyFromSymbol(	(void*)&new_size,  pcl::device::new_cloud_size, sizeof(new_size)) );	
	return new_size;
}

///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
  namespace device
  {
    struct NotMinus1
	{
	  __device__ __forceinline__
	  int operator()(const int& v) const { return (v == -1) ?  0 : 1; }
	};


	struct Compaction
	{
		enum 
		{
			CTA_SIZE = 256,

			WARPS = CTA_SIZE/ Warp::WARP_SIZE
		};

		int* head_points_in;
		PtrStep<int>  verts_inds_in;
		

		int *scan_buffer;
		int facet_count;

		mutable int* head_points_out;
		mutable PtrStep<int>  verts_inds_out;
		

		mutable PtrStep<int> empty_facets;
		mutable int *empty_count;
		  
		__device__ __forceinline__
		void operator()() const
		{
			int idx = threadIdx.x + blockIdx.x * blockDim.x;

#if CUDA_VERSION >= 9000
      if (__all_sync (__activemask (), idx >= facet_count))
        return;
#else
			if (__all (idx >= facet_count))
				return;
#endif

			int empty = 0;

			if(idx < facet_count)
			{
				int head_idx = head_points_in[idx];
				if (head_idx != -1)
				{
					int offset = scan_buffer[idx];

					head_points_out[offset] = head_idx;
					
					verts_inds_out.ptr(0)[offset] = verts_inds_in.ptr(0)[idx];
					verts_inds_out.ptr(1)[offset] = verts_inds_in.ptr(1)[idx];
					verts_inds_out.ptr(2)[offset] = verts_inds_in.ptr(2)[idx];

                    
					
				}
				else                
				  empty = 1;                
			}

#if CUDA_VERSION >= 9000
      int total = __popc (__ballot_sync (__activemask (), empty));
#else
			int total = __popc (__ballot (empty));
#endif
			if (total > 0)
			{
#if CUDA_VERSION >= 9000
        int offset = Warp::binaryExclScan (__ballot_sync (__activemask (), empty));
#else
				int offset = Warp::binaryExclScan (__ballot (empty));
#endif

				volatile __shared__ int wapr_buffer[WARPS];

				int laneid = Warp::laneId();
				int warpid = Warp::id();
				if (laneid == 0)
				{
					int old = atomicAdd(empty_count, total);
					wapr_buffer[warpid] = old;                    
				}
				int old = wapr_buffer[warpid];

                if (empty)
                {
				  empty_facets.ptr(0)[old + offset] = verts_inds_in.ptr(0)[idx];
				  empty_facets.ptr(1)[old + offset] = verts_inds_in.ptr(1)[idx];
				  empty_facets.ptr(2)[old + offset] = verts_inds_in.ptr(2)[idx];		                  

                  int a1 = verts_inds_in.ptr(0)[idx], a2 = verts_inds_in.ptr(1)[idx], a3 = verts_inds_in.ptr(2)[idx];
                }
			}							
		}
	};

	__global__ void compactionKernel( const Compaction c )  { c(); }
  }
}


void pcl::device::FacetStream::compactFacets()
{
  int old_empty_count;  
  empty_count.download(&old_empty_count); 

  thrust::device_ptr<int> b(head_points.ptr());
  thrust::device_ptr<int> e = b + facet_count;
  thrust::device_ptr<int> o(scan_buffer.ptr());
  
  thrust::transform_exclusive_scan(b, e, o, NotMinus1(), 0, thrust::plus<int>());                                                                                    
  
  Compaction c;

  c.verts_inds_in   = verts_inds;
  c.head_points_in  = head_points;    

  c.scan_buffer = scan_buffer;
  c.facet_count = facet_count;

  c.head_points_out = head_points2;
  c.verts_inds_out = verts_inds2;

  c.empty_facets = empty_facets;
  c.empty_count = empty_count;
 
  int block = Compaction::CTA_SIZE;
  int grid = divUp(facet_count, block);

  compactionKernel<<<grid, block>>>(c);   
  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
    
  verts_inds.swap(verts_inds2);
  head_points.swap(head_points2);

  int new_empty_count;  
  empty_count.download(&new_empty_count); 
  
  facet_count -= new_empty_count - old_empty_count;
}


///////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
  namespace device
  {
	  struct Classify
	  {
		uint64_type* facets_dists;
		int* scan_buffer;

		int* head_points;
		int* perm;
		PtrStep<int>  verts_inds;

		const PointType *points;

		float diag;

		int facet_count;

		__device__ __forceinline__ 
		void operator()(int point_idx) const
		{
          int perm_index = perm[point_idx];
          
		  int facet = facets_dists[point_idx] >> 32;
		  facet = scan_buffer[facet];

		  int hi = head_points[facet];

          if (hi == perm_index)
          {
            uint64_type res = numeric_limits<int>::max();
		    res <<= 32;		                      
            facets_dists[point_idx] = res;
          }
          else            
          {

		    int i1 = verts_inds.ptr(0)[facet];
		    int i2 = verts_inds.ptr(1)[facet];
		    int i3 = verts_inds.ptr(2)[facet];

		    float3 hp = tr( points[ hi ] );
		    float3 v1 = tr( points[ i1 ] );
		    float3 v2 = tr( points[ i2 ] );
		    float3 v3 = tr( points[ i3 ] );
		
		    float4 p0 = compute_plane(hp, v1, v2, /*opposite*/v3); // j
		    float4 p1 = compute_plane(hp, v2, v3, /*opposite*/v1); // facet_count + j
		    float4 p2 = compute_plane(hp, v3, v1, /*opposite*/v2); // facet_count + j*2			

            p0 *= compue_inv_normal_norm(p0);
            p1 *= compue_inv_normal_norm(p1);
            p2 *= compue_inv_normal_norm(p2);

          
		    float4 p = points[perm_index];
		    p.w = 1;

		    float d0 = dot(p, p0);
		    float d1 = dot(p, p1);
		    float d2 = dot(p, p2);

            float dists[] = { d0, d1, d2 };
            int negs_inds[3];
            int neg_count = 0;

            int new_idx = numeric_limits<int>::max();
            float dist = 0;

            int indeces[] = { facet, facet + facet_count, facet + facet_count * 2 };

            #pragma unroll
            for(int i = 0; i < 3; ++i)
              if (dists[i] < 0)
               negs_inds[neg_count++] = i;
 
            if (neg_count == 3)
            {
              int i1 = negs_inds[1];
              int i2 = negs_inds[2];
           
              int ir = fabs(dists[i1]) < fabs(dists[i2]) ? i2 : i1;
              negs_inds[1] = ir;
              --neg_count;
            }

            if (neg_count == 2)
            {
              int i1 = negs_inds[0];
              int i2 = negs_inds[1];
           
              int ir = fabs(dists[i1]) < fabs(dists[i2]) ? i2 : i1;
              negs_inds[0] = ir;
              --neg_count;              
            }

            if (neg_count == 1)
            {
              new_idx = negs_inds[0];
              dist = diag - fabs(dists[new_idx]); // to ensure that sorting order is inverse, i.e. distant points go first
              new_idx = indeces[new_idx];
            }

            // if (neg_count == 0)
            // new_idx = INT_MAX ==>> internal point
                      	       	 	   
            uint64_type res = new_idx;
		    res <<= 32;
		    res += *reinterpret_cast<unsigned int*>(&dist);

		    facets_dists[point_idx] = res;

          } /* if (hi == perm_index) */            
        }
	  };    

      __global__ void classifyKernel(const Classify c, int cloud_size)
      {
        int point_idx = threadIdx.x + blockIdx.x * blockDim.x;

        if ( point_idx < cloud_size )
          c(point_idx);
      }
  }
}

void pcl::device::PointStream::classify(FacetStream& fs)
{   
  Classify c;

  c.facets_dists = facets_dists;
  c.scan_buffer = fs.scan_buffer;
  c.head_points = fs.head_points;
  c.perm = perm;

  c.verts_inds = fs.verts_inds;
  c.points = cloud;

  c.diag = cloud_diag;
  c.facet_count = fs.facet_count;

  //thrust::counting_iterator<int> b(0);    
  //thrust::for_each(b, b + cloud_size, c);

  classifyKernel<<<divUp(cloud_size, 256), 256>>>(c, cloud_size);
  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
  
  thrust::device_ptr<uint64_type> beg(facets_dists.ptr());
  thrust::device_ptr<uint64_type> end = beg + cloud_size;
  
  thrust::device_ptr<int> pbeg(perm.ptr());
  thrust::sort_by_key(beg, end, pbeg);
}

namespace pcl
{
  namespace device
  {
    struct SplitFacets
    {
      int* head_points;
      int facet_count;

      mutable PtrStep<int>  verts_inds;

      __device__ __forceinline__ 
      void operator()(int facet) const
      {
        int hi = head_points[facet];
        int i1 = verts_inds.ptr(0)[facet];
        int i2 = verts_inds.ptr(1)[facet];
        int i3 = verts_inds.ptr(2)[facet];
        
        make_facet(hi, i1, i2, facet);
        make_facet(hi, i2, i3, facet + facet_count);
        make_facet(hi, i3, i1, facet + facet_count * 2);
      }

      __device__ __forceinline__
      void make_facet(int i1, int i2, int i3, int out_idx) const
      {
        verts_inds.ptr(0)[out_idx] = i1;
        verts_inds.ptr(1)[out_idx] = i2;
        verts_inds.ptr(2)[out_idx] = i3;
      }
    };

    __global__ void splitFacetsKernel(const SplitFacets sf)
    {
      int facet = threadIdx.x + blockIdx.x * blockDim.x;

      if (facet < sf.facet_count)        
        sf(facet);        
    }
  }
}

void pcl::device::FacetStream::splitFacets()
{
  SplitFacets sf;
  sf.head_points = head_points;
  sf.verts_inds = verts_inds;
  sf.facet_count = facet_count;
    

  //thrust::counting_iterator<int> b(0);    
  //thrust::for_each(b, b + facet_count, sf);

  splitFacetsKernel<<<divUp(facet_count, 256), 256>>>(sf);
  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );

  facet_count *= 3;
}

size_t pcl::device::remove_duplicates(DeviceArray<int>& indeces)
{
  thrust::device_ptr<int> beg(indeces.ptr());
  thrust::device_ptr<int> end = beg + indeces.size();

  thrust::sort(beg, end);  
  return (size_t)(thrust::unique(beg, end) - beg);  
}


namespace pcl
{
  namespace device
  {
    __global__ void gatherKernel(const PtrSz<int> indeces, const PointType* src, PointType* dst)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < indeces.size)
          dst[idx] = src[indeces.data[idx]];
    }
  }
}


void pcl::device::pack_hull(const DeviceArray<PointType>& points, const DeviceArray<int>& indeces, DeviceArray<PointType>& output)
{
  output.create(indeces.size());

  //device_ptr<const PointType> in(points.ptr());  
  
  //thrust::device_ptr<const int> mb(indeces.ptr());
  //thrust::device_ptr<const int> me = mb + indeces.size();

  //device_ptr<PointType> out(output.ptr());  

  //thrust::gather(mb, me, in, out);
  
  gatherKernel<<<divUp(indeces.size(), 256), 256>>>(indeces, points, output);
  cudaSafeCall( hipGetLastError() );
  cudaSafeCall( hipDeviceSynchronize() );
}
